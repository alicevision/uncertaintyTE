#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zmergecg.cu, normal z -> d, Sun Nov 20 20:20:43 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512

#define PRECISION_d


// These routines merge multiple kernels from dmergecg into one
// for a description see 
// "Reformulated Conjugate Gradient for the Energy-Aware 
// Solution of Linear Systems on GPUs (ICPP '13)

// accelerated reduction for one vector
__global__ void
magma_dcgreduce_kernel_spmv1( 
    int Gs,
    int n, 
    double * vtmp,
    double * vtmp2 )
{
    extern __shared__ double temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    temp[Idx] = MAGMA_D_MAKE( 0.0, 0.0);
    int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
    while (i < Gs ) {
        temp[ Idx  ] += vtmp[ i ]; 
        temp[ Idx  ] += ( i + blockSize < Gs ) ? vtmp[ i + blockSize ] 
                                                : MAGMA_D_MAKE( 0.0, 0.0); 
        i += gridSize;
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ) {
        vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}


// accelerated reduction for two vectors
__global__ void
magma_dcgreduce_kernel_spmv2( 
    int Gs,
    int n, 
    double * vtmp,
    double * vtmp2 )
{
    extern __shared__ double temp[];     
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    int j;

    for( j=0; j<2; j++){
        int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
        temp[Idx+j*(blockSize)] = MAGMA_D_ZERO;
        while (i < Gs ) {
            temp[ Idx+j*(blockSize)  ] += vtmp[ i+j*n ]; 
            temp[ Idx+j*(blockSize)  ] += 
                ( i + (blockSize) < Gs ) ? vtmp[ i+j*n + (blockSize) ] 
                                                : MAGMA_D_ZERO;
            i += gridSize;
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp2[ blockIdx.x+j*n ] = temp[ j*(blockSize) ];
        }
    }
}



// computes the SpMV using CSR and the first step of the reduction
__global__ void
magma_dcgmerge_spmvcsr_kernel(  
    int n,
    double * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    double * d,
    double * z,
    double * vtmp )
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    temp[ Idx ] = MAGMA_D_MAKE( 0.0, 0.0);

    if( i<n ) {
        double dot = MAGMA_D_ZERO;
        int start = drowptr[ i ];
        int end = drowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * d[ dcolind[j] ];
        z[ i ] =  dot;
        temp[ Idx ] =  d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

// computes the SpMV using ELL and the first step of the reduction
__global__ void
magma_dcgmerge_spmvell_kernel(  
    int n,
    int num_cols_per_row,
    double * dval, 
    magma_index_t * dcolind,
    double * d,
    double * z,
    double * vtmp )
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = MAGMA_D_MAKE( 0.0, 0.0);

    if(i < n ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        for ( int k = 0; k < num_cols_per_row; k++ ) {
            int col = dcolind [ n * k + i ];
            double val = dval [ n * k + i ];
            if( val != 0)
                dot += val * d[ col ];
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// computes the SpMV using ELLPACK and the first step of the reduction
__global__ void
magma_dcgmerge_spmvellpack_kernel(  
    int n,
    int num_cols_per_row,
    double * dval, 
    magma_index_t * dcolind,
    double * d,
    double * z,
    double * vtmp )
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = MAGMA_D_MAKE( 0.0, 0.0);

    if(i < n ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        for ( int k = 0; k < num_cols_per_row; k++ ) {
            int col = dcolind [ num_cols_per_row * i + k ];
            double val = dval [ num_cols_per_row * i + k ];
            if( val != 0)
                dot += val * d[ col ];
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// computes the SpMV using SELL alignment 1 and the first step of the reduction
__global__ void
magma_dcgmerge_spmvell_kernelb1(  
    int n,
    int blocksize,
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    double * d,
    double * z,
    double * vtmp )
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = MAGMA_D_MAKE( 0.0, 0.0);
    
    int idx = threadIdx.x;      // local row
    int bdx = blockIdx.x; // global block index
    int row = bdx * 256 + idx;  // global row index
    // int lblocksize = ( row + blocksize < num_rows) ? blocksize : ( num_rows - blocksize * (row/blocksize) );
    int lrow = threadIdx.x%blocksize; // local row;
    
    if( row < n ) {
        int offset = drowptr[ row/blocksize ];
        int border = (drowptr[ row/blocksize+1 ]-offset)/blocksize;
    
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < border; n++) { 
            int col = dcolind [ offset+ blocksize * n + lrow ];
            double val = dval[ offset+ blocksize * n + lrow ];
            dot = dot + val * d [ col ];
        }
        z[ i ] = dot;
        temp[ Idx ] = d[ i ] * dot;
    }
    
/*
    if(i < n ) {
        int offset = drowptr[ blockIdx.x ];
        int border = (drowptr[ blockIdx.x+1 ]-offset)/blocksize;
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        for ( int k = 0; k < border; k++){ 
            int col = dcolind [ offset+ blocksize * k + threadIdx.x ];
            double val = dval[offset+ blocksize * k + threadIdx.x];
            if( val != 0){
                  dot += val*d[col];
            }
        }
        
        
        //double dot = MAGMA_D_MAKE(0.0, 0.0);
        //for ( int k = 0; k < num_cols_per_row; k++ ) {
        //    int col = dcolind [ n * k + i ];
        //    double val = dval [ n * k + i ];
        //    if( val != 0)
        //        dot += val * d[ col ];
        //}
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }*/

    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_dcgmerge_spmvellpackrt_kernel_8(  
    int n,
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    double * d,
    double * z,
    double * vtmp,
    magma_int_t T, 
    magma_int_t alignment  )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ double shared[];

    if(i < n ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ) {
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }
        }
    }
}

// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_dcgmerge_spmvellpackrt_kernel_16(  
    int n,
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    double * d,
    double * z,
    double * vtmp,
    magma_int_t T, 
    magma_int_t alignment  )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ double shared[];

    if(i < n ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ) {
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }
        }
    }
}

// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_dcgmerge_spmvellpackrt_kernel_32(  
    int n,
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    double * d,
    double * z,
    double * vtmp,
    magma_int_t T, 
    magma_int_t alignment  )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ double shared[];

    if(i < n ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ) {
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }
        }
    }
}


// additional kernel necessary to compute first reduction step
__global__ void
magma_dcgmerge_spmvellpackrt_kernel2(  
    int n,
    double * z,
    double * d,
    double * vtmp2 )
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    

    temp[ Idx ] = ( i < n ) ? z[i]*d[i] : MAGMA_D_MAKE(0.0, 0.0);
    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}



// computes the SpMV using SELLC
__global__ void
magma_dcgmerge_spmvsellc_kernel(   
    int num_rows, 
    int blocksize,
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    double * d,
    double * z,
    double * vtmp)
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int offset = drowptr[ blockIdx.x ];
    int border = (drowptr[ blockIdx.x+1 ]-offset)/blocksize;

    temp[ Idx ] = MAGMA_D_MAKE( 0.0, 0.0);


    if(i < num_rows ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < border; n ++) {
            int col = dcolind [offset+ blocksize * n + Idx ];
            double val = dval[offset+ blocksize * n + Idx];
            if( val != 0) {
                  dot=dot+val*d[col];
            }
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }
    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_dcgmerge_spmvsellpt_kernel_8( 
    int num_rows, 
    int blocksize,
    int T,
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    double * d,
    double * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y;      // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ double shared[];

    if(row < num_rows ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_; k++ ) {
            double val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 4 ) {
            shared[ldx]+=shared[ldx+blocksize*4];              
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }
        }
    }
}
// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_dcgmerge_spmvsellpt_kernel_16( 
    int num_rows, 
    int blocksize,
    int T,
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    double * d,
    double * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y;      // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ double shared[];

    if(row < num_rows ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_; k++ ) {
            double val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 8 ) {
            shared[ldx]+=shared[ldx+blocksize*8];              
            __syncthreads();
            if( idx < 4 ) shared[ldx]+=shared[ldx+blocksize*4];   
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }
        }
    }
}


// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_dcgmerge_spmvsellpt_kernel_32( 
    int num_rows, 
    int blocksize,
    int T,
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    double * d,
    double * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y;      // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ double shared[];

    if(row < num_rows ) {
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_; k++ ) {
            double val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 16 ) {
            shared[ldx]+=shared[ldx+blocksize*16];              
            __syncthreads();
            if( idx < 8 ) shared[ldx]+=shared[ldx+blocksize*8];  
            __syncthreads();
            if( idx < 4 ) shared[ldx]+=shared[ldx+blocksize*4];   
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }
        }
    }
}


// kernel to handle scalars
__global__ void // rho = beta/tmp; gamma = beta;
magma_dcg_rhokernel(  
    double * skp ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ) {
        double tmp = skp[1];
        skp[3] = tmp/skp[4];
        skp[2] = tmp;
    }
}

/**
    Purpose
    -------

    Merges the first SpmV using different formats with the dot product 
    and the computation of rho

    Arguments
    ---------

    @param[in]
    A           magma_d_matrix
                input matrix 

    @param[in]
    d1          magmaDouble_ptr 
                temporary vector

    @param[in]
    d2          magmaDouble_ptr 
                temporary vector

    @param[in]
    dd          magmaDouble_ptr 
                input vector d

    @param[out]
    dz          magmaDouble_ptr 
                input vector z

    @param[out]
    skp         magmaDouble_ptr 
                array for parameters ( skp[3]=rho )

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_dcgmerge_spmv1(
    magma_d_matrix A,
    magmaDouble_ptr d1,
    magmaDouble_ptr d2,
    magmaDouble_ptr dd,
    magmaDouble_ptr dz,
    magmaDouble_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( A.num_rows, local_block_size ) );
    dim3 Gs_next;
    int Ms =  local_block_size * sizeof( double ); 
    magmaDouble_ptr aux1 = d1, aux2 = d2;
    int b = 1;        

    if ( A.storage_type == Magma_CSR )
        magma_dcgmerge_spmvcsr_kernel<<< Gs, Bs, Ms, queue->hip_stream() >>>
        ( A.num_rows, A.dval, A.drow, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_ELLPACKT )
        magma_dcgmerge_spmvellpack_kernel<<< Gs, Bs, Ms, queue->hip_stream() >>>
        ( A.num_rows, A.max_nnz_row, A.dval, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_ELL )
        magma_dcgmerge_spmvell_kernel<<< Gs, Bs, Ms, queue->hip_stream() >>>
        ( A.num_rows, A.max_nnz_row, A.dval, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_CUCSR ) {
        hipsparseHandle_t cusparseHandle = 0;
        hipsparseMatDescr_t descr = 0;
        double c_one = MAGMA_D_ONE;
        double c_zero = MAGMA_D_ZERO;
        hipsparseCreate( &cusparseHandle );
        hipsparseSetStream( cusparseHandle, queue->hip_stream() );
        hipsparseCreateMatDescr( &descr );
        hipsparseSetMatType( descr, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descr, HIPSPARSE_INDEX_BASE_ZERO );
        hipsparseDcsrmv( cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
        A.num_rows, A.num_cols, A.nnz, &c_one, descr,
        A.dval, A.drow, A.dcol, dd, &c_zero, dz );
        hipsparseDestroyMatDescr( descr );
        hipsparseDestroy( cusparseHandle );
        cusparseHandle = 0;
        descr = 0;
        magma_dcgmerge_spmvellpackrt_kernel2<<< Gs, Bs, Ms, queue->hip_stream() >>>
                      ( A.num_rows, dz, dd, d1 );
    }
    else if ( A.storage_type == Magma_SELLP && A.alignment == 1 ) {
            magma_dcgmerge_spmvell_kernelb1<<< Gs, Bs, Ms, queue->hip_stream() >>>
            ( A.num_rows, A.blocksize, 
                A.dval, A.dcol, A.drow, dd, dz, d1 );
    }
    else if ( A.storage_type == Magma_SELLP && A.alignment > 1) {
            int num_threadssellp = A.blocksize*A.alignment;
            magma_int_t arch = magma_getdevice_arch();
            if ( arch < 200 && num_threadssellp > 256 )
                printf("error: too much shared memory requested.\n");

            dim3 block( A.blocksize, A.alignment, 1);
            int dimgrid1 = int( sqrt( double( A.numblocks )));
            int dimgrid2 = magma_ceildiv( A.numblocks, dimgrid1 );

            dim3 gridsellp( dimgrid1, dimgrid2, 1);
            int Mssellp = num_threadssellp * sizeof( double );

            if ( A.alignment == 8)
                magma_dcgmerge_spmvsellpt_kernel_8
                <<< gridsellp, block, Mssellp, queue->hip_stream() >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else if ( A.alignment == 16)
                magma_dcgmerge_spmvsellpt_kernel_16
                <<< gridsellp, block, Mssellp, queue->hip_stream() >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else if ( A.alignment == 32)
                magma_dcgmerge_spmvsellpt_kernel_32
                <<< gridsellp, block, Mssellp, queue->hip_stream() >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else
                printf("error: alignment not supported.\n");

        // in case of using SELLP, we can't efficiently merge the 
        // dot product and the first reduction loop into the SpMV kernel
        // as the SpMV grid would result in low occupancy.
        magma_dcgmerge_spmvellpackrt_kernel2<<< Gs, Bs, Ms, queue->hip_stream() >>>
                              ( A.num_rows, dz, dd, d1 );
    }
    else if ( A.storage_type == Magma_ELLRT ) {
        // in case of using ELLRT, we need a different grid, assigning
        // threads_per_row processors to each row
        // the block size is num_threads
        // fixed values


    int num_blocks = magma_ceildiv( A.num_rows, A.blocksize );

    int num_threads = A.alignment*A.blocksize;

    int real_row_length = magma_roundup( A.max_nnz_row, A.alignment );

    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = int( sqrt( double( num_blocks )));
    int dimgrid2 = magma_ceildiv( num_blocks, dimgrid1 );
    dim3 gridellrt( dimgrid1, dimgrid2, 1);

    int Mellrt = A.alignment * A.blocksize * sizeof( double );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if ( A.alignment == 32 ) {
        magma_dcgmerge_spmvellpackrt_kernel_32
                <<< gridellrt, num_threads , Mellrt, queue->hip_stream() >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else if ( A.alignment == 16 ) {
        magma_dcgmerge_spmvellpackrt_kernel_16
                <<< gridellrt, num_threads , Mellrt, queue->hip_stream() >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else if ( A.alignment == 8 ) {
        magma_dcgmerge_spmvellpackrt_kernel_8
                <<< gridellrt, num_threads , Mellrt, queue->hip_stream() >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", int(A.alignment) );
        return MAGMA_ERR_NOT_SUPPORTED;
    }
        // in case of using ELLRT, we can't efficiently merge the 
        // dot product and the first reduction loop into the SpMV kernel
        // as the SpMV grid would result in low occupancy.

        magma_dcgmerge_spmvellpackrt_kernel2<<< Gs, Bs, Ms, queue->hip_stream() >>>
                              ( A.num_rows, dz, dd, d1 );
    }

    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_dcgreduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                                        ( Gs.x,  A.num_rows, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_dcopyvector( 1, aux1, 1, skp+4, 1, queue );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_dcg_rhokernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

    return MAGMA_SUCCESS;
}


/* -------------------------------------------------------------------------- */

// updates x and r and computes the first part of the dot product r*r
__global__ void
magma_dcgmerge_xrbeta_kernel(  
    int n, 
    double * x, 
    double * r,
    double * d,
    double * z,
    double * skp,
    double * vtmp )
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    double rho = skp[3];
    double mrho = MAGMA_D_MAKE( -1.0, 0.0)*rho;

    temp[ Idx ] = MAGMA_D_MAKE( 0.0, 0.0);

    if( i<n ) {
        x[i] += rho * d[i];
        r[i] += mrho * z[i];
        temp[ Idx ] = r[i] * r[i];
    }
    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

// kernel to handle scalars
__global__ void //alpha = beta / gamma
magma_dcg_alphabetakernel(  
    double * skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ) {
        double tmp1 = skp[1];
        skp[0] =  tmp1/skp[2];
        //printf("beta=%e\n", MAGMA_D_REAL(tmp1));
    }
}

// update search Krylov vector d
__global__ void
magma_dcg_d_kernel(  
    int n, 
    double * skp,
    double * r,
    double * d )
{
    int i   = blockIdx.x * blockDim.x + threadIdx.x;

    double alpha = skp[0];

    if( i<n ) {
        d[i] = r[i] + alpha * d[i];
    }
}



/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    d1          magmaDouble_ptr 
                temporary vector

    @param[in]
    d2          magmaDouble_ptr 
                temporary vector

    @param[in,out]
    dx          magmaDouble_ptr
                input vector x

    @param[in,out]
    dr          magmaDouble_ptr 
                input/output vector r

    @param[in]
    dd          magmaDouble_ptr 
                input vector d

    @param[in]
    dz          magmaDouble_ptr 
                input vector z
    @param[in]
    skp         magmaDouble_ptr 
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dsygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_dcgmerge_xrbeta(
    magma_int_t n,
    magmaDouble_ptr d1,
    magmaDouble_ptr d2,
    magmaDouble_ptr dx,
    magmaDouble_ptr dr,
    magmaDouble_ptr dd,
    magmaDouble_ptr dz, 
    magmaDouble_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( double ); 
    magmaDouble_ptr aux1 = d1, aux2 = d2;
    int b = 1;        
    magma_dcgmerge_xrbeta_kernel<<< Gs, Bs, Ms, queue->hip_stream()>>>
                                    ( n, dx, dr, dd, dz, skp, d1);  



    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_dcgreduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_dcopyvector( 1, aux1, 1, skp+1, 1, queue );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_dcg_alphabetakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

    dim3 Bs3( local_block_size );
    dim3 Gs3( magma_ceildiv( n, local_block_size ) );
    magma_dcg_d_kernel<<< Gs3, Bs3, 0, queue->hip_stream()>>>( n, skp, dr, dd );  

    return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

// updates x and r
__global__ void
magma_dpcgmerge_xrbeta_kernel(  
    int n, 
    double * x, 
    double * r,
    double * d,
    double * z,
    double * skp )
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    double rho = skp[3];
    double mrho = MAGMA_D_MAKE( -1.0, 0.0)*rho;

    if( i<n ) {
        x[i] += rho * d[i];
        r[i] += mrho * z[i];
    }
}


// dot product for multiple vectors
__global__ void
magma_dmddot_one_kernel_1( 
    int n, 
    double * v0,
    double * w0,
    double * vtmp)
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    // 1 vectors v(i)/w(i)
    
    temp[ Idx ]                 = ( i < n ) ?
                v0[ i ] * w0[ i ] : MAGMA_D_ZERO;
    temp[ Idx + blockDim.x ]    = ( i < n ) ?
                v0[ i ] * v0[ i ] : MAGMA_D_ZERO;
    
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif  
    
    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
            vtmp[ blockIdx.x+n ] = temp[ blockDim.x ];
    }
}

/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in,out]
    dx          magmaDouble_ptr
                input vector x

    @param[in,out]
    dr          magmaDouble_ptr 
                input/output vector r

    @param[in]
    dd          magmaDouble_ptr 
                input vector d

    @param[in]
    dz          magmaDouble_ptr 
                input vector z
    @param[in]
    skp         magmaDouble_ptr 
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dsygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_dpcgmerge_xrbeta1(
    magma_int_t n,
    magmaDouble_ptr dx,
    magmaDouble_ptr dr,
    magmaDouble_ptr dd,
    magmaDouble_ptr dz, 
    magmaDouble_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    magma_dpcgmerge_xrbeta_kernel<<< Gs, Bs, 0, queue->hip_stream()>>>
                                    ( n, dx, dr, dd, dz, skp );  
                                    
    return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */


/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    d1          magmaDouble_ptr 
                temporary vector

    @param[in]
    d2          magmaDouble_ptr 
                temporary vector

    @param[in]
    dh          magmaDouble_ptr
                input vector x

    @param[in]
    dr          magmaDouble_ptr 
                input/output vector r
                
    @param[in]
    dd          magmaDouble_ptr 
                input/output vector d

    @param[in]
    skp         magmaDouble_ptr 
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dsygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_dpcgmerge_xrbeta2(
    magma_int_t n,
    magmaDouble_ptr d1,
    magmaDouble_ptr d2,
    magmaDouble_ptr dh,
    magmaDouble_ptr dr, 
    magmaDouble_ptr dd, 
    magmaDouble_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  4*local_block_size * sizeof( double ); 
    magmaDouble_ptr aux1 = d1, aux2 = d2;
    int b = 1;        
                                    
    magma_dmddot_one_kernel_1<<< Gs, Bs, Ms, queue->hip_stream()>>>
                                    ( n, dr, dh, d1);  

    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_dcgreduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_dcopyvector( 1, aux1, 1, skp+1, 1, queue );
    magma_dcopyvector( 1, aux1+n, 1, skp+6, 1, queue );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_dcg_alphabetakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

    dim3 Bs3( local_block_size );
    dim3 Gs3( magma_ceildiv( n, local_block_size ) );
    magma_dcg_d_kernel<<< Gs3, Bs3, 0, queue->hip_stream()>>>( n, skp, dh, dd );  

    return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */



// updates x and r
__global__ void
magma_djcgmerge_xrbeta_kernel(  
    int n, 
    double * diag, 
    double * x,     
    double * r,
    double * d,
    double * z,
    double * h,
    double * vtmp,
    double * skp )
{
    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    double rho = skp[3];
    double mrho = MAGMA_D_MAKE( -1.0, 0.0)*rho;

    if( i<n ) {
        x[i] += rho * d[i];
        r[i] += mrho * z[i];
        h[i] = r[i] * diag[i];
    }
    __syncthreads();
    temp[ Idx ]                 = ( i < n ) ?
                h[ i ] * r[ i ] : MAGMA_D_ZERO;
    temp[ Idx + blockDim.x ]    = ( i < n ) ?
                r[ i ] * r[ i ] : MAGMA_D_ZERO;
    
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif  
    
    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
            vtmp[ blockIdx.x+n ] = temp[ blockDim.x ];
    }
}


/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    d1          magmaDouble_ptr 
                temporary vector

    @param[in]
    d2          magmaDouble_ptr 
                temporary vector
                
    @param[in]
    diag        magmaDouble_ptr 
                inverse diagonal (Jacobi preconditioner)

    @param[in]
    dx          magmaDouble_ptr
                iteration vector x

    @param[in]
    dr          magmaDouble_ptr 
                input/output vector r
                
    @param[in]
    dd          magmaDouble_ptr
                input vector d

                
    @param[in]
    dz          magmaDouble_ptr
                input vector z
                
    @param[in]
    dh          magmaDouble_ptr
                input vector h

    @param[in]
    skp         magmaDouble_ptr 
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dsygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_djcgmerge_xrbeta(
    magma_int_t n,
    magmaDouble_ptr d1,
    magmaDouble_ptr d2,
    magmaDouble_ptr diag,
    magmaDouble_ptr dx,
    magmaDouble_ptr dr,
    magmaDouble_ptr dd,
    magmaDouble_ptr dz,
    magmaDouble_ptr dh, 
    magmaDouble_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  4*local_block_size * sizeof( double ); 
    magmaDouble_ptr aux1 = d1, aux2 = d2;
    int b = 1;    
                                    
    magma_djcgmerge_xrbeta_kernel<<< Gs, Bs, Ms, queue->hip_stream() >>>
                                    ( n, diag, dx, dr, dd, dz, dh, d1, skp );  
                                    
    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_dcgreduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream() >>> 
                                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_dcopyvector( 1, aux1, 1, skp+1, 1, queue );
    magma_dcopyvector( 1, aux1+n, 1, skp+6, 1, queue );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_dcg_alphabetakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

    dim3 Bs3( local_block_size );
    dim3 Gs3( magma_ceildiv( n, local_block_size ) );
    magma_dcg_d_kernel<<< Gs3, Bs3, 0, queue->hip_stream()>>>( n, skp, dh, dd );  

    return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */
