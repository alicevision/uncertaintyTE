#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zmergebicgstab.cu, normal z -> c, Sun Nov 20 20:20:40 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512

#define PRECISION_c


// These routines merge multiple kernels from bicgstab into one.

/* -------------------------------------------------------------------------- */

__global__ void
magma_cbicgstab_1_kernel(  
    int num_rows, 
    int num_cols, 
    magmaFloatComplex beta,
    magmaFloatComplex omega,
    magmaFloatComplex *r, 
    magmaFloatComplex *v,
    magmaFloatComplex *p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            p[ i+j*num_rows ] = r[ i+j*num_rows ] + 
                beta * ( p[ i+j*num_rows ] - omega * v[ i+j*num_rows ] );
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    p = r + beta * ( p - omega * v )
    
    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    beta        magmaFloatComplex
                scalar
                
    @param[in]
    omega       magmaFloatComplex
                scalar
                
    @param[in]
    r           magmaFloatComplex_ptr 
                vector
                
    @param[in]
    v           magmaFloatComplex_ptr 
                vector
                
    @param[in,out]
    p           magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_cbicgstab_1(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex beta,
    magmaFloatComplex omega,
    magmaFloatComplex_ptr r, 
    magmaFloatComplex_ptr v,
    magmaFloatComplex_ptr p,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_cbicgstab_1_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, beta, omega,
                     r, v, p );

    return MAGMA_SUCCESS;
}


__global__ void
magma_cbicgstab_2_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr r,
    magmaFloatComplex_ptr v,
    magmaFloatComplex_ptr s )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            s[ i+j*num_rows ] = r[ i+j*num_rows ] - alpha * v[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    s = r - alpha v

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaFloatComplex
                scalar
                
    @param[in]
    r           magmaFloatComplex_ptr 
                vector
                
    @param[in]
    v           magmaFloatComplex_ptr 
                vector

    @param[in,out]
    s           magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_cbicgstab_2(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr r,
    magmaFloatComplex_ptr v,
    magmaFloatComplex_ptr s, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_cbicgstab_2_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, r, v, s );

    return MAGMA_SUCCESS;
}


__global__ void
magma_cbicgstab_3_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex alpha,
    magmaFloatComplex omega,
    magmaFloatComplex *p,
    magmaFloatComplex *s,
    magmaFloatComplex *t,
    magmaFloatComplex *x,
    magmaFloatComplex *r )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            magmaFloatComplex tmp = s[ i+j*num_rows ];
            x[ i+j*num_rows ] = x[ i+j*num_rows ] 
                        + alpha * p[ i+j*num_rows ] + omega * tmp;
            r[ i+j*num_rows ] = tmp - omega * t[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x = x + alpha * p + omega * s
    r = s - omega * t

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaFloatComplex
                scalar
                
    @param[in]
    omega       magmaFloatComplex
                scalar
                
    @param[in]
    p           magmaFloatComplex_ptr 
                vector
                    
    @param[in]
    s           magmaFloatComplex_ptr 
                vector
                    
    @param[in]
    t           magmaFloatComplex_ptr 
                vector

    @param[in,out]
    x           magmaFloatComplex_ptr 
                vector
                
    @param[in,out]
    r           magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_cbicgstab_3(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex alpha,
    magmaFloatComplex omega,
    magmaFloatComplex_ptr p,
    magmaFloatComplex_ptr s,
    magmaFloatComplex_ptr t,
    magmaFloatComplex_ptr x,
    magmaFloatComplex_ptr r,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_cbicgstab_3_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, omega, p, s, t, x, r );

    return MAGMA_SUCCESS;
}


__global__ void
magma_cbicgstab_4_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex alpha,
    magmaFloatComplex omega,
    magmaFloatComplex *y,
    magmaFloatComplex *z,
    magmaFloatComplex *s,
    magmaFloatComplex *t,
    magmaFloatComplex *x,
    magmaFloatComplex *r )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            x[ i+j*num_rows ] = x[ i+j*num_rows ] 
                        + alpha * y[ i+j*num_rows ] + omega * z[ i+j*num_rows ];
            r[ i+j*num_rows ] = s[ i+j*num_rows ] - omega * t[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x = x + alpha * y + omega * z
    r = s - omega * t

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaFloatComplex
                scalar
                
    @param[in]
    omega       magmaFloatComplex
                scalar
                
    @param[in]
    y           magmaFloatComplex_ptr 
                vector
                
    @param[in]
    z           magmaFloatComplex_ptr 
                vector
                    
    @param[in]
    s           magmaFloatComplex_ptr 
                vector
                    
    @param[in]
    t           magmaFloatComplex_ptr 
                vector

    @param[in,out]
    x           magmaFloatComplex_ptr 
                vector
                
    @param[in,out]
    r           magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_cbicgstab_4(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex alpha,
    magmaFloatComplex omega,
    magmaFloatComplex_ptr y,
    magmaFloatComplex_ptr z,
    magmaFloatComplex_ptr s,
    magmaFloatComplex_ptr t,
    magmaFloatComplex_ptr x,
    magmaFloatComplex_ptr r,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_cbicgstab_4_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, omega, y, z, s, t, x, r );

    return MAGMA_SUCCESS;
}
