#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zjacobisetup.cu, normal z -> d, Sun Nov 20 20:20:40 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512


#define PRECISION_d

__global__ void 
dvjacobisetup_gpu(  int num_rows, 
                    int num_vecs,
                    double *b, 
                    double *d, 
                    double *c,
                    double *x)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ ){
            c[row+i*num_rows] = b[row+i*num_rows] / d[row];
            x[row+i*num_rows] = c[row+i*num_rows];
        }
    }
}


/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_d_matrix
                RHS b

    @param[in]
    d           magma_d_matrix
                vector with diagonal entries

    @param[out]
    c           magma_d_matrix*
                c = D^(-1) * b

    @param[out]
    x           magma_d_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_djacobisetup_vector_gpu(
    magma_int_t num_rows, 
    magma_d_matrix b, 
    magma_d_matrix d, 
    magma_d_matrix c,
    magma_d_matrix *x,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    int num_vecs = b.num_rows / num_rows;
    magma_int_t threads = BLOCK_SIZE;
    dvjacobisetup_gpu<<< grid, threads, 0, queue->hip_stream()>>>
                ( num_rows, num_vecs, b.dval, d.dval, c.dval, x->val );

    return MAGMA_SUCCESS;
}


__global__ void 
djacobidiagscal_kernel(  int num_rows,
                         int num_vecs, 
                    double *b, 
                    double *d, 
                    double *c)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
            c[row+i*num_rows] = b[row+i*num_rows] * d[row];
    }
}


/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_d_matrix
                RHS b

    @param[in]
    d           magma_d_matrix
                vector with diagonal entries

    @param[out]
    c           magma_d_matrix*
                c = D^(-1) * b
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_djacobi_diagscal(
    magma_int_t num_rows, 
    magma_d_matrix d, 
    magma_d_matrix b, 
    magma_d_matrix *c,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, 512 ));
    int num_vecs = b.num_rows*b.num_cols/num_rows;
    magma_int_t threads = 512;
    djacobidiagscal_kernel<<< grid, threads, 0, queue->hip_stream()>>>( num_rows, num_vecs, b.dval, d.dval, c->val );

    return MAGMA_SUCCESS;
}


__global__ void 
djacobiupdate_kernel(  int num_rows,
                       int num_cols, 
                    double *t, 
                    double *b, 
                    double *d, 
                    double *x)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < num_rows ){
        for( int i=0; i<num_cols; i++)
            x[row+i*num_rows] += (b[row+i*num_rows]-t[row+i*num_rows]) * d[row];
    }
}


/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-t)
    where d is the diagonal of the system matrix A and t=Ax.

    Arguments
    ---------
                
    @param[in]
    t           magma_d_matrix
                t = A*x
                
    @param[in]
    b           magma_d_matrix
                RHS b
                
    @param[in]
    d           magma_d_matrix
                vector with diagonal entries

    @param[out]
    x           magma_d_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_djacobiupdate(
    magma_d_matrix t, 
    magma_d_matrix b, 
    magma_d_matrix d, 
    magma_d_matrix *x,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;
    djacobiupdate_kernel<<< grid, threads, 0, queue->hip_stream()>>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );

    return MAGMA_SUCCESS;
}


__global__ void 
djacobispmvupdate_kernel(  
    int num_rows,
    int num_cols, 
    double * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    double *t, 
    double *b, 
    double *d, 
    double *x )
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int j;

    if(row<num_rows){
        double dot = MAGMA_D_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( int i=0; i<num_cols; i++){
            for( j=start; j<end; j++){
                dot += dval[ j ] * x[ dcolind[j]+i*num_rows ];
            }
            x[row+i*num_rows] += (b[row+i*num_rows]-dot) * d[row];
        }
    }
}


/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-Ax)


    Arguments
    ---------

    @param[in]
    maxiter     magma_int_t
                number of Jacobi iterations   
                
    @param[in]
    A           magma_d_matrix
                system matrix
                
    @param[in]
    t           magma_d_matrix
                workspace
                
    @param[in]
    b           magma_d_matrix
                RHS b
                
    @param[in]
    d           magma_d_matrix
                vector with diagonal entries

    @param[out]
    x           magma_d_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_djacobispmvupdate(
    magma_int_t maxiter,
    magma_d_matrix A,
    magma_d_matrix t, 
    magma_d_matrix b, 
    magma_d_matrix d, 
    magma_d_matrix *x,
    magma_queue_t queue )
{
    // local variables
    //double c_zero = MAGMA_D_ZERO;
    //double c_one = MAGMA_D_ONE;

    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;

    for( magma_int_t i=0; i<maxiter; i++ ) {
        // distinct routines imply synchronization
        // magma_d_spmv( c_one, A, *x, c_zero, t, queue );                // t =  A * x
        // djacobiupdate_kernel<<< grid, threads, 0, queue->hip_stream()>>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );
        // merged in one implies asynchronous update
        djacobispmvupdate_kernel<<< grid, threads, 0, queue->hip_stream()>>>
            ( t.num_rows, t.num_cols, A.dval, A.drow, A.dcol, t.dval, b.dval, d.dval, x->dval );
    }

    return MAGMA_SUCCESS;
}


__global__ void 
djacobispmvupdate_bw_kernel(  
    int num_rows,
    int num_cols, 
    double * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    double *t, 
    double *b, 
    double *d, 
    double *x )
{
    int row_tmp = blockDim.x * blockIdx.x + threadIdx.x;
    int row = num_rows-1 - row_tmp;
    int j;

    if( row>-1 ){
        double dot = MAGMA_D_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( int i=0; i<num_cols; i++){
            for( j=start; j<end; j++){
                dot += dval[ j ] * x[ dcolind[j]+i*num_rows ];
            }
            x[row+i*num_rows] += (b[row+i*num_rows]-dot) * d[row];
        }
    }
}


/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-Ax)
    This kernel processes the thread blocks in reversed order.

    Arguments
    ---------

    @param[in]
    maxiter     magma_int_t
                number of Jacobi iterations   
                
    @param[in]
    A           magma_d_matrix
                system matrix
                
    @param[in]
    t           magma_d_matrix
                workspace
                
    @param[in]
    b           magma_d_matrix
                RHS b
                
    @param[in]
    d           magma_d_matrix
                vector with diagonal entries

    @param[out]
    x           magma_d_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_djacobispmvupdate_bw(
    magma_int_t maxiter,
    magma_d_matrix A,
    magma_d_matrix t, 
    magma_d_matrix b, 
    magma_d_matrix d, 
    magma_d_matrix *x,
    magma_queue_t queue )
{
    // local variables
    //double c_zero = MAGMA_D_ZERO;
    //double c_one = MAGMA_D_ONE;

    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;

    for( magma_int_t i=0; i<maxiter; i++ ) {
        // distinct routines imply synchronization
        // magma_d_spmv( c_one, A, *x, c_zero, t, queue );                // t =  A * x
        // djacobiupdate_kernel<<< grid, threads, 0, queue->hip_stream()>>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );
        // merged in one implies asynchronous update
        djacobispmvupdate_bw_kernel<<< grid, threads, 0, queue->hip_stream()>>>
            ( t.num_rows, t.num_cols, A.dval, A.drow, A.dcol, t.dval, b.dval, d.dval, x->dval );
    }

    return MAGMA_SUCCESS;
}


__global__ void 
djacobispmvupdateselect_kernel(  
    int num_rows,
    int num_cols, 
    int num_updates, 
    magma_index_t * indices, 
    double * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    double *t, 
    double *b, 
    double *d, 
    double *x,
    double *y )
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int j;

    if(  idx<num_updates){
        int row = indices[ idx ];
        printf(" ");    
        //if( row < num_rows ){
        double dot = MAGMA_D_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( int i=0; i<num_cols; i++){
            for( j=start; j<end; j++){
                dot += dval[ j ] * x[ dcolind[j]+i*num_rows ];
            }
            x[row+i*num_rows] = x[row+i*num_rows] + (b[row+i*num_rows]-dot) * d[row];
            
            //double add = (b[row+i*num_rows]-dot) * d[row];
            //#if defined(PRECISION_s) //|| defined(PRECISION_d)
            //    atomicAdd( x + row + i*num_rows, add );  
            //#endif
            // ( unsigned int* address, unsigned int val);
        //}
        }
    }
}


/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-Ax)
        
    This kernel allows for overlapping domains: the indices-array contains
    the locations that are updated. Locations may be repeated to simulate
    overlapping domains.


    Arguments
    ---------

    @param[in]
    maxiter     magma_int_t
                number of Jacobi iterations
                
    @param[in]
    num_updates magma_int_t
                number of updates - length of the indices array
                    
    @param[in]
    indices     magma_index_t*
                indices, which entries of x to update
                
    @param[in]
    A           magma_d_matrix
                system matrix
                
    @param[in]
    t           magma_d_matrix
                workspace
                
    @param[in]
    b           magma_d_matrix
                RHS b
                
    @param[in]
    d           magma_d_matrix
                vector with diagonal entries
   
    @param[in]
    tmp         magma_d_matrix
                workspace

    @param[out]
    x           magma_d_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_djacobispmvupdateselect(
    magma_int_t maxiter,
    magma_int_t num_updates,
    magma_index_t *indices,
    magma_d_matrix A,
    magma_d_matrix t, 
    magma_d_matrix b, 
    magma_d_matrix d, 
    magma_d_matrix tmp, 
    magma_d_matrix *x,
    magma_queue_t queue )
{
    // local variables
    //double c_zero = MAGMA_D_ZERO
    //double c_one = MAGMA_D_ONE;
    
    //magma_d_matrix swp;

    dim3 grid( magma_ceildiv( num_updates, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;
    printf("num updates:%d %d %d\n", int(num_updates), int(threads), int(grid.x) );

    for( magma_int_t i=0; i<maxiter; i++ ) {
        djacobispmvupdateselect_kernel<<< grid, threads, 0, queue->hip_stream()>>>
            ( t.num_rows, t.num_cols, num_updates, indices, A.dval, A.drow, A.dcol, t.dval, b.dval, d.dval, x->dval, tmp.dval );
        //swp.dval = x->dval;
        //x->dval = tmp.dval;
        //tmp.dval = swp.dval;
    }
    
    return MAGMA_SUCCESS;
}


__global__ void 
dftjacobicontractions_kernel(
    int num_rows,
    double * xkm2val, 
    double * xkm1val, 
    double * xkval, 
    double * zval,
    double * cval )
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(  idx<num_rows ){
        zval[idx] = MAGMA_D_MAKE( MAGMA_D_ABS( xkm1val[idx] - xkval[idx] ), 0.0);
        cval[ idx ] = MAGMA_D_MAKE(
            MAGMA_D_ABS( xkm2val[idx] - xkm1val[idx] ) 
                / MAGMA_D_ABS( xkm1val[idx] - xkval[idx] )
                                        ,0.0 );
    }
}


/**
    Purpose
    -------

    Computes the contraction coefficients c_i:
    
    c_i = z_i^{k-1} / z_i^{k} 
        
        = | x_i^{k-1} - x_i^{k-2} | / |  x_i^{k} - x_i^{k-1} |

    Arguments
    ---------

    @param[in]
    xkm2        magma_d_matrix
                vector x^{k-2}
                
    @param[in]
    xkm1        magma_d_matrix
                vector x^{k-2}
                
    @param[in]
    xk          magma_d_matrix
                vector x^{k-2}
   
    @param[out]
    z           magma_d_matrix*
                ratio
                
    @param[out]
    c           magma_d_matrix*
                contraction coefficients
                
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_dftjacobicontractions(
    magma_d_matrix xkm2,
    magma_d_matrix xkm1, 
    magma_d_matrix xk, 
    magma_d_matrix *z,
    magma_d_matrix *c,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( xk.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;

    dftjacobicontractions_kernel<<< grid, threads, 0, queue->hip_stream()>>>
            ( xkm2.num_rows, xkm2.dval, xkm1.dval, xk.dval, z->dval, c->dval );
    
    return MAGMA_SUCCESS;
}


__global__ void 
dftjacobiupdatecheck_kernel(
    int num_rows,
    double delta,
    double * xold, 
    double * xnew, 
    double * zprev,
    double * cval, 
    magma_int_t *flag_t,
    magma_int_t *flag_fp )
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(  idx<num_rows ){
        double t1 = delta * MAGMA_D_ABS(cval[idx]);
        double  vkv = 1.0;
        for( magma_int_t i=0; i<min( flag_fp[idx], 100 ); i++){
            vkv = vkv*2;
        }
        double xold_l = xold[idx];
        double xnew_l = xnew[idx];
        double znew = MAGMA_D_MAKE(
                        max( MAGMA_D_ABS( xold_l - xnew_l), 1e-15), 0.0 );
                        
        double znr = zprev[idx] / znew; 
        double t2 = MAGMA_D_ABS( znr - cval[idx] );
        
        //% evaluate fp-cond
        magma_int_t fpcond = 0;
        if( MAGMA_D_ABS(znr)>vkv ){
            fpcond = 1;
        }
        
        // % combine t-cond and fp-cond + flag_t == 1
        magma_int_t cond = 0;
        if( t2<t1 || (flag_t[idx]>0 && fpcond > 0 ) ){
            cond = 1;
        }
        flag_fp[idx] = flag_fp[idx]+1;
        if( fpcond>0 ){
            flag_fp[idx] = 0;
        }
        if( cond > 0 ){
            flag_t[idx] = 0;
            zprev[idx] = znew;
            xold[idx] = xnew_l;
        } else {
            flag_t[idx] = 1;
            xnew[idx] = xold_l;
        }
    }
}


/**
    Purpose
    -------

    Checks the Jacobi updates accorting to the condition in the ScaLA'15 paper.

    Arguments
    ---------
    
    @param[in]
    delta       double
                threshold

    @param[in,out]
    xold        magma_d_matrix*
                vector xold
                
    @param[in,out]
    xnew        magma_d_matrix*
                vector xnew
                
    @param[in,out]
    zprev       magma_d_matrix*
                vector z = | x_k-1 - x_k |
   
    @param[in]
    c           magma_d_matrix
                contraction coefficients
                
    @param[in,out]
    flag_t      magma_int_t
                threshold condition
                
    @param[in,out]
    flag_fp     magma_int_t
                false positive condition
                
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_dftjacobiupdatecheck(
    double delta,
    magma_d_matrix *xold,
    magma_d_matrix *xnew, 
    magma_d_matrix *zprev, 
    magma_d_matrix c,
    magma_int_t *flag_t,
    magma_int_t *flag_fp,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( xnew->num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;

    dftjacobiupdatecheck_kernel<<< grid, threads, 0, queue->hip_stream()>>>
            ( xold->num_rows, delta, xold->dval, xnew->dval, zprev->dval, c.dval, 
                flag_t, flag_fp );
    
    return MAGMA_SUCCESS;
}
