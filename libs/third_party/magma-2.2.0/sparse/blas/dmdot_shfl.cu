#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zmdot_shfl.cu, normal z -> d, Sun Nov 20 20:20:40 2016
       @author Moritz Kreutzer

*/
#include "magmasparse_internal.h"

#include "magmasparse_d.h"
#define BLOCK_SIZE 512

#define PRECISION_d

#include <hip/hip_runtime.h>  // for CUDA_VERSION

#if (CUDA_VERSION <= 6000)
// CUDA 6.5 adds Double precision version; here's an implementation for CUDA 6.0 and earlier.
// from https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
__device__ inline
real_Double_t __shfl_down(real_Double_t var, unsigned int srcLane, int width=32) {
  int2 a = *reinterpret_cast<int2*>(&var);
  a.x = __shfl_down(a.x, srcLane, width);
  a.y = __shfl_down(a.y, srcLane, width);
  return *reinterpret_cast<double*>(&a);
}
#endif


template<typename T>
__inline__ __device__
T warpReduceSum(T val)
{
#if __CUDA_ARCH__ >= 300
    val += __shfl_down(val, 16);
    val += __shfl_down(val, 8);
    val += __shfl_down(val, 4);
    val += __shfl_down(val, 2);
    val += __shfl_down(val, 1);
#endif
    return val;
}


#ifdef PRECISION_z
template<>
__inline__ __device__
double warpReduceSum<double>(double val)
{
#if __CUDA_ARCH__ >= 300
    int4 a = *reinterpret_cast<int4*>(&val);
    a.x += __shfl_down(a.x, 16);
    a.y += __shfl_down(a.y, 16);
    a.z += __shfl_down(a.z, 16);
    a.w += __shfl_down(a.w, 16);
    a.x += __shfl_down(a.x, 8);
    a.y += __shfl_down(a.y, 8);
    a.z += __shfl_down(a.z, 8);
    a.w += __shfl_down(a.w, 8);
    a.x += __shfl_down(a.x, 4);
    a.y += __shfl_down(a.y, 4);
    a.z += __shfl_down(a.z, 4);
    a.w += __shfl_down(a.w, 4);
    a.x += __shfl_down(a.x, 2);
    a.y += __shfl_down(a.y, 2);
    a.z += __shfl_down(a.z, 2);
    a.w += __shfl_down(a.w, 2);
    a.x += __shfl_down(a.x, 1);
    a.y += __shfl_down(a.y, 1);
    a.z += __shfl_down(a.z, 1);
    a.w += __shfl_down(a.w, 1);
#endif
    return val;
}
#endif // PRECISION_z


#ifdef PRECISION_c
template<>
__inline__ __device__
magmaFloatComplex warpReduceSum<magmaFloatComplex>(magmaFloatComplex val)
{
#if __CUDA_ARCH__ >= 300
    float2 a = *reinterpret_cast<float2*>(&val);
    a.x += __shfl_down(a.x, 16);
    a.y += __shfl_down(a.y, 16);
    a.x += __shfl_down(a.x, 8);
    a.y += __shfl_down(a.y, 8);
    a.x += __shfl_down(a.x, 4);
    a.y += __shfl_down(a.y, 4);
    a.x += __shfl_down(a.x, 2);
    a.y += __shfl_down(a.y, 2);
    a.x += __shfl_down(a.x, 1);
    a.y += __shfl_down(a.y, 1);
#endif
    return val;
}
#endif // PRECISION_c


template<typename T>
__inline__ __device__
T blockReduceSum_1D(T val)
{
    extern __shared__ T shared[]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warpReduceSum<T>(val);     // Each warp performs partial reduction

    if (lane == 0) shared[wid]=val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : MAGMA_D_ZERO;
    
    if (wid == 0) val = warpReduceSum<T>(val); //Final reduce within first warp
    return val;
}


template<typename T>
__inline__ __device__
T blockReduceSum(T val)
{
    extern __shared__ T shared[]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warpReduceSum<T>(val);     // Each warp performs partial reduction

    if (lane == 0) shared[threadIdx.y*32+wid]=val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[threadIdx.y*32+lane] : MAGMA_D_ZERO;
    
    if (wid == 0) val = warpReduceSum<T>(val); //Final reduce within first warp
    return val;
}


template<typename T> 
__global__ void deviceReduceKernel(const T * __restrict__ in, T * __restrict__ out, int N)
{
    T sum = MAGMA_D_MAKE(0.0, 0.0);
    //reduce multiple elements per thread
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += in[i];
    }
    sum = blockReduceSum<T>(sum);
    if (threadIdx.x == 0)
        out[blockIdx.x]=sum;
}


// dot product for multiple vectors using shuffle intrinsics and less shared memory
__global__ void
magma_dblockdot_kernel_shuffle( 
    int n, 
    int k,
    const double * __restrict__ v,
    const double * __restrict__ r,
    double * __restrict__ vtmp)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = threadIdx.y;
    double tmp;
    if (i < n) {
        tmp = v[i+j*n] * r[i];
    } else {
        tmp = MAGMA_D_ZERO;
    }
    tmp = blockReduceSum(tmp);
    if (threadIdx.x == 0 ){
        vtmp[ blockIdx.x+j*gridDim.x ] = tmp;
    }
}


// dot product for multiple vectors using shuffle intrinsics and less shared memory
__global__ void
magma_dblockdot_kernel_shuffle_1dblock( 
    int n, 
    int k,
    const double * __restrict__ v,
    const double * __restrict__ r,
    double * __restrict__ vtmp)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;
    for (j=0; j < k; j++) {
        double tmp;
        if (i < n) {
            tmp = v[i+j*n] * r[i];
        } else {
            tmp = MAGMA_D_ZERO;
        }
        tmp = blockReduceSum_1D(tmp);
        if (threadIdx.x == 0 ){
            vtmp[ blockIdx.x+j*gridDim.x ] = tmp;
        }
    }
}


/**
    Purpose
    -------

    Computes the scalar product of a set of vectors v_i such that

    skp = ( <v_0,r>, <v_1,r>, .. )

    Returns the vector skp.

    Arguments
    ---------

    @param[in]
    n           int
                length of v_i and r

    @param[in]
    k           int
                # vectors v_i

    @param[in]
    v           magmaDouble_ptr 
                v = (v_0 .. v_i.. v_k)

    @param[in]
    r           magmaDouble_ptr 
                r

    @param[in]
    d1          magmaDouble_ptr 
                workspace

    @param[in]
    d2          magmaDouble_ptr 
                workspace

    @param[out]
    skp         magmaDouble_ptr 
                vector[k] of scalar products (<v_i,r>...)

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" magma_int_t
magma_dmdotc_shfl(
    magma_int_t n, 
    magma_int_t k, 
    magmaDouble_ptr v, 
    magmaDouble_ptr r,
    magmaDouble_ptr d1,
    magmaDouble_ptr d2,
    magmaDouble_ptr skp,
    magma_queue_t queue )
{
    if ( magma_getdevice_arch() < 300 ) {
        return magma_dmdotc( n, k, v, r, d1, d2, skp, queue );
    }
    else if (1) { // 1D block kernel seems to be always faster
        dim3 block( BLOCK_SIZE );
        dim3 grid( magma_ceildiv( n, block.x ) );
        magma_dblockdot_kernel_shuffle_1dblock<<< grid, block, 32*sizeof(double), queue->hip_stream() >>>( n, k, v, r, d1 );
        int j;
        for (j=0; j < k; j++) {
            deviceReduceKernel<double> <<<1, 1024, 32*sizeof(double), queue->hip_stream()>>>(d1+grid.x*j, skp+j, grid.x);
        }
    } else {
        dim3 block( magma_roundup( magma_ceildiv(BLOCK_SIZE, k), 32 ), k );
        while (block.x*block.y > 1024) {
            block.x -= 32;
        }
        dim3 grid( magma_ceildiv( n, block.x ) );
        magma_dblockdot_kernel_shuffle<<< grid, block, 32*k*sizeof(double), queue->hip_stream() >>>( n, k, v, r, d1 );
        int j;
        for (j=0; j < k; j++) {
            deviceReduceKernel<double> <<<1, 1024, 32*sizeof(double), queue->hip_stream()>>>(d1+grid.x*j, skp+j, grid.x);
        }
    }
   
    return MAGMA_SUCCESS;
}


/**
    Purpose
    -------

    This is an extension of the merged dot product above by chunking
    the set of vectors v_i such that the data always fits into cache.
    It is equivalent to a matrix vecor product Vr where V
    contains few rows and many columns. The computation is the same:

    skp = ( <v_0,r>, <v_1,r>, .. )

    Returns the vector skp.

    Arguments
    ---------

    @param[in]
    n           int
                length of v_i and r

    @param[in]
    k           int
                # vectors v_i

    @param[in]
    v           magmaDouble_ptr 
                v = (v_0 .. v_i.. v_k)

    @param[in]
    r           magmaDouble_ptr 
                r

    @param[in]
    d1          magmaDouble_ptr 
                workspace

    @param[in]
    d2          magmaDouble_ptr 
                workspace

    @param[out]
    skp         magmaDouble_ptr 
                vector[k] of scalar products (<v_i,r>...)

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_dgemvmdot_shfl(
    magma_int_t n, 
    magma_int_t k, 
    magmaDouble_ptr v, 
    magmaDouble_ptr r,
    magmaDouble_ptr d1,
    magmaDouble_ptr d2,
    magmaDouble_ptr skp,
    magma_queue_t queue )
{
    if (k == 1) { // call CUBLAS dotc, we will never be faster
        double res = magma_ddot( n, v, 1, r, 1, queue );
        magma_dsetvector( 1, &res, 1, skp, 1, queue );
    }
    else if ( magma_getdevice_arch() < 300 ) {
        return magma_dgemvmdot( n, k, v, r, d1, d2, skp, queue );
    }
    else {
        magma_dmdotc_shfl( n, k, v, r, d1, d2, skp, queue );
    }

    return MAGMA_SUCCESS;
}
