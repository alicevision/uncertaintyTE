#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zgeisai_trsv.cu, normal z -> s, Sun Nov 20 20:20:43 2016

*/
#include "magmasparse_internal.h"
//#include <hip/hip_runtime_api.h>

#define PRECISION_s
#define REAL
#define BLOCKSIZE 256
#define WARP_SIZE 32
#define WRP 32
#define WRQ 1



#include <hip/hip_runtime.h>  // for CUDA_VERSION

#if (CUDA_VERSION >= 7000)
#if (CUDA_ARCH >= 300)

__device__
void strsv_lower_kernel_general(float *dA, float *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;

    float rB[ 2 ];
    float rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;


    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];


    // Triangular solve in regs.
    #pragma unroll
    for (k = 0; k < N; k++)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];

        float top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
            if (n*WARP_SIZE+idn > k)
                rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];

#endif
}


__device__
void strsv_upper_kernel_general(float *dA, float *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;

    float rB[ 2 ];
    float rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;


    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];


    // Triangular solve in regs.
    #pragma unroll
    for (int k = N-1; k > -1; k--)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];

        float top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
            if (n*WARP_SIZE+idn < k)
                rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];

#endif
}



__device__
void strsv_lower_kernel_1(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 1; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_2(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 2; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_3(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 3; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_4(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 4; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_5(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 5; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_6(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 6; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_7(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 7; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_8(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 8; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_9(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 9; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_10(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 10; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_11(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 11; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_12(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 12; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_13(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 13; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_14(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 14; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_15(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 15; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_16(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 16; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_17(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 17; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_18(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 18; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_19(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 19; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_20(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 20; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_21(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 21; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_22(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 22; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_23(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 23; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_24(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 24; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_25(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 25; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_26(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 26; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_27(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 27; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_28(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 28; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_29(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 29; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_30(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 30; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_31(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 31; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_lower_kernel_32(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 32; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}


__global__
void strsv_lower_kernel_switch(float *dA, float *dB, int *sizes, int num_rows )
{
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    if (j < num_rows) {
        int N = sizes[j];
        switch( N ) {
            case  1:
                strsv_lower_kernel_1( dA, dB ); break;
            case  2:
                strsv_lower_kernel_2( dA, dB ); break;
            case  3:
                strsv_lower_kernel_3( dA, dB ); break;
            case  4:
                strsv_lower_kernel_4( dA, dB ); break;
            case  5:
                strsv_lower_kernel_5( dA, dB ); break;
            case  6:
                strsv_lower_kernel_6( dA, dB ); break;
            case  7:
                strsv_lower_kernel_7( dA, dB ); break;
            case  8:
                strsv_lower_kernel_8( dA, dB ); break;
            case  9:
                strsv_lower_kernel_9( dA, dB ); break;
            case  10:
                strsv_lower_kernel_10( dA, dB ); break;
            case  11:
                strsv_lower_kernel_11( dA, dB ); break;
            case  12:
                strsv_lower_kernel_12( dA, dB ); break;
            case  13:
                strsv_lower_kernel_13( dA, dB ); break;
            case  14:
                strsv_lower_kernel_14( dA, dB ); break;
            case  15:
                strsv_lower_kernel_15( dA, dB ); break;
            case  16:
                strsv_lower_kernel_16( dA, dB ); break;
            case  17:
                strsv_lower_kernel_17( dA, dB ); break;
            case  18:
                strsv_lower_kernel_18( dA, dB ); break;
            case  19:
                strsv_lower_kernel_19( dA, dB ); break;
            case  20:
                strsv_lower_kernel_20( dA, dB ); break;
            case  21:
                strsv_lower_kernel_21( dA, dB ); break;
            case  22:
                strsv_lower_kernel_22( dA, dB ); break;
            case  23:
                strsv_lower_kernel_23( dA, dB ); break;
            case  24:
                strsv_lower_kernel_24( dA, dB ); break;
            case  25:
                strsv_lower_kernel_25( dA, dB ); break;
            case  26:
                strsv_lower_kernel_26( dA, dB ); break;
            case  27:
                strsv_lower_kernel_27( dA, dB ); break;
            case  28:
                strsv_lower_kernel_28( dA, dB ); break;
            case  29:
                strsv_lower_kernel_29( dA, dB ); break;
            case  30:
                strsv_lower_kernel_30( dA, dB ); break;
            case  31:
                strsv_lower_kernel_31( dA, dB ); break;
            case  32:
                strsv_lower_kernel_32( dA, dB ); break;
            default:
                strsv_lower_kernel_general( dA, dB, sizes ); break;
        }
    }
}
__device__
void strsv_upper_kernel_1(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 1-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_2(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 2-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_3(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 3-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_4(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 4-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_5(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 5-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_6(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 6-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_7(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 7-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_8(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 8-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_9(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 9-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_10(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 10-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_11(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 11-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_12(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 12-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_13(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 13-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_14(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 14-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_15(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 15-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_16(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 16-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_17(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 17-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_18(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 18-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_19(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 19-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_20(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 20-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_21(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 21-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_22(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 22-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_23(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 23-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_24(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 24-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_25(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 25-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_26(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 26-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_27(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 27-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_28(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 28-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_29(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 29-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_30(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 30-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_31(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 31-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void strsv_upper_kernel_32(float *dA, float *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    float rB;
    float rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 32-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        float bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}


__global__
void strsv_upper_kernel_switch(float *dA, float *dB, int *sizes, int num_rows )
{
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    if (j < num_rows) {
        int N = sizes[j];
        switch( N ) {
            case  1:
                strsv_upper_kernel_1( dA, dB ); break;
            case  2:
                strsv_upper_kernel_2( dA, dB ); break;
            case  3:
                strsv_upper_kernel_3( dA, dB ); break;
            case  4:
                strsv_upper_kernel_4( dA, dB ); break;
            case  5:
                strsv_upper_kernel_5( dA, dB ); break;
            case  6:
                strsv_upper_kernel_6( dA, dB ); break;
            case  7:
                strsv_upper_kernel_7( dA, dB ); break;
            case  8:
                strsv_upper_kernel_8( dA, dB ); break;
            case  9:
                strsv_upper_kernel_9( dA, dB ); break;
            case  10:
                strsv_upper_kernel_10( dA, dB ); break;
            case  11:
                strsv_upper_kernel_11( dA, dB ); break;
            case  12:
                strsv_upper_kernel_12( dA, dB ); break;
            case  13:
                strsv_upper_kernel_13( dA, dB ); break;
            case  14:
                strsv_upper_kernel_14( dA, dB ); break;
            case  15:
                strsv_upper_kernel_15( dA, dB ); break;
            case  16:
                strsv_upper_kernel_16( dA, dB ); break;
            case  17:
                strsv_upper_kernel_17( dA, dB ); break;
            case  18:
                strsv_upper_kernel_18( dA, dB ); break;
            case  19:
                strsv_upper_kernel_19( dA, dB ); break;
            case  20:
                strsv_upper_kernel_20( dA, dB ); break;
            case  21:
                strsv_upper_kernel_21( dA, dB ); break;
            case  22:
                strsv_upper_kernel_22( dA, dB ); break;
            case  23:
                strsv_upper_kernel_23( dA, dB ); break;
            case  24:
                strsv_upper_kernel_24( dA, dB ); break;
            case  25:
                strsv_upper_kernel_25( dA, dB ); break;
            case  26:
                strsv_upper_kernel_26( dA, dB ); break;
            case  27:
                strsv_upper_kernel_27( dA, dB ); break;
            case  28:
                strsv_upper_kernel_28( dA, dB ); break;
            case  29:
                strsv_upper_kernel_29( dA, dB ); break;
            case  30:
                strsv_upper_kernel_30( dA, dB ); break;
            case  31:
                strsv_upper_kernel_31( dA, dB ); break;
            case  32:
                strsv_upper_kernel_32( dA, dB ); break;
            default:
                strsv_upper_kernel_general( dA, dB, sizes ); break;
        }
    }
}
#endif
#endif
/**
    Purpose
    -------
    Does all triangular solves

    Arguments
    ---------


    @param[in]
    uplotype    magma_uplo_t
                lower or upper triangular

    @param[in]
    transtype   magma_trans_t
                possibility for transposed matrix

    @param[in]
    diagtype    magma_diag_t
                unit diagonal or not

    @param[in]
    L           magma_s_matrix
                Matrix in CSR format

    @param[in]
    LC          magma_s_matrix
                same matrix, also CSR, but col-major

    @param[out]
    sizes       magma_int_t*
                Number of Elements that are replaced.

    @param[out]
    locations   magma_int_t*
                Array indicating the locations.

    @param[out]
    trisystems  float*
                trisystems

    @param[out]
    rhs         float*
                right-hand sides

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_saux
    ********************************************************************/

extern "C" magma_int_t
magma_smtrisolve_batched_gpu(
    magma_uplo_t uplotype,
    magma_trans_t transtype,
    magma_diag_t diagtype,
    magma_s_matrix L,
    magma_s_matrix LC,
    magma_index_t *sizes,
    magma_index_t *locations,
    float *trisystems,
    float *rhs,
    magma_queue_t queue )
{
    magma_int_t info = 0;

    int blocksize1 = WARP_SIZE;
    int blocksize2 = 1;
    int dimgrid1 = min( int( sqrt( float( LC.num_rows ))), 65535 );
    int dimgrid2 = min(magma_ceildiv( LC.num_rows, dimgrid1 ), 65535);
    int dimgrid3 = magma_ceildiv( LC.num_rows, dimgrid1*dimgrid2 );

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );

#if (CUDA_VERSION >= 7000)
#if (CUDA_ARCH >= 300)
    if( uplotype == MagmaLower ){
        //hipProfilerStart();
        strsv_lower_kernel_switch<<< grid, block, 0, queue->hip_stream() >>>(
                trisystems,
                rhs,
                sizes,
                LC.num_rows );
        //hipProfilerStop();
    } else {
        strsv_upper_kernel_switch<<< grid, block, 0, queue->hip_stream() >>>(
                trisystems,
                rhs,
                sizes,
                LC.num_rows );
    }
#endif
#endif

    return info;
}
