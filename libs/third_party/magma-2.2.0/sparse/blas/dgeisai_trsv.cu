#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zgeisai_trsv.cu, normal z -> d, Sun Nov 20 20:20:43 2016

*/
#include "magmasparse_internal.h"
//#include <hip/hip_runtime_api.h>

#define PRECISION_d
#define REAL
#define BLOCKSIZE 256
#define WARP_SIZE 32
#define WRP 32
#define WRQ 1



#include <hip/hip_runtime.h>  // for CUDA_VERSION

#if (CUDA_VERSION >= 7000)
#if (CUDA_ARCH >= 300)

__device__
void dtrsv_lower_kernel_general(double *dA, double *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;

    double rB[ 2 ];
    double rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;


    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];


    // Triangular solve in regs.
    #pragma unroll
    for (k = 0; k < N; k++)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];

        double top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
            if (n*WARP_SIZE+idn > k)
                rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];

#endif
}


__device__
void dtrsv_upper_kernel_general(double *dA, double *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;

    double rB[ 2 ];
    double rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;


    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];


    // Triangular solve in regs.
    #pragma unroll
    for (int k = N-1; k > -1; k--)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];

        double top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
            if (n*WARP_SIZE+idn < k)
                rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];

#endif
}



__device__
void dtrsv_lower_kernel_1(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 1; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_2(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 2; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_3(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 3; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_4(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 4; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_5(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 5; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_6(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 6; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_7(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 7; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_8(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 8; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_9(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 9; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_10(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 10; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_11(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 11; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_12(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 12; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_13(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 13; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_14(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 14; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_15(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 15; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_16(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 16; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_17(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 17; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_18(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 18; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_19(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 19; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_20(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 20; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_21(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 21; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_22(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 22; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_23(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 23; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_24(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 24; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_25(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 25; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_26(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 26; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_27(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 27; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_28(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 28; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_29(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 29; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_30(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 30; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_31(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 31; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_lower_kernel_32(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 32; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}


__global__
void dtrsv_lower_kernel_switch(double *dA, double *dB, int *sizes, int num_rows )
{
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    if (j < num_rows) {
        int N = sizes[j];
        switch( N ) {
            case  1:
                dtrsv_lower_kernel_1( dA, dB ); break;
            case  2:
                dtrsv_lower_kernel_2( dA, dB ); break;
            case  3:
                dtrsv_lower_kernel_3( dA, dB ); break;
            case  4:
                dtrsv_lower_kernel_4( dA, dB ); break;
            case  5:
                dtrsv_lower_kernel_5( dA, dB ); break;
            case  6:
                dtrsv_lower_kernel_6( dA, dB ); break;
            case  7:
                dtrsv_lower_kernel_7( dA, dB ); break;
            case  8:
                dtrsv_lower_kernel_8( dA, dB ); break;
            case  9:
                dtrsv_lower_kernel_9( dA, dB ); break;
            case  10:
                dtrsv_lower_kernel_10( dA, dB ); break;
            case  11:
                dtrsv_lower_kernel_11( dA, dB ); break;
            case  12:
                dtrsv_lower_kernel_12( dA, dB ); break;
            case  13:
                dtrsv_lower_kernel_13( dA, dB ); break;
            case  14:
                dtrsv_lower_kernel_14( dA, dB ); break;
            case  15:
                dtrsv_lower_kernel_15( dA, dB ); break;
            case  16:
                dtrsv_lower_kernel_16( dA, dB ); break;
            case  17:
                dtrsv_lower_kernel_17( dA, dB ); break;
            case  18:
                dtrsv_lower_kernel_18( dA, dB ); break;
            case  19:
                dtrsv_lower_kernel_19( dA, dB ); break;
            case  20:
                dtrsv_lower_kernel_20( dA, dB ); break;
            case  21:
                dtrsv_lower_kernel_21( dA, dB ); break;
            case  22:
                dtrsv_lower_kernel_22( dA, dB ); break;
            case  23:
                dtrsv_lower_kernel_23( dA, dB ); break;
            case  24:
                dtrsv_lower_kernel_24( dA, dB ); break;
            case  25:
                dtrsv_lower_kernel_25( dA, dB ); break;
            case  26:
                dtrsv_lower_kernel_26( dA, dB ); break;
            case  27:
                dtrsv_lower_kernel_27( dA, dB ); break;
            case  28:
                dtrsv_lower_kernel_28( dA, dB ); break;
            case  29:
                dtrsv_lower_kernel_29( dA, dB ); break;
            case  30:
                dtrsv_lower_kernel_30( dA, dB ); break;
            case  31:
                dtrsv_lower_kernel_31( dA, dB ); break;
            case  32:
                dtrsv_lower_kernel_32( dA, dB ); break;
            default:
                dtrsv_lower_kernel_general( dA, dB, sizes ); break;
        }
    }
}
__device__
void dtrsv_upper_kernel_1(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 1-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_2(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 2-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_3(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 3-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_4(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 4-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_5(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 5-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_6(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 6-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_7(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 7-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_8(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 8-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_9(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 9-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_10(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 10-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_11(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 11-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_12(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 12-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_13(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 13-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_14(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 14-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_15(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 15-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_16(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 16-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_17(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 17-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_18(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 18-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_19(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 19-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_20(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 20-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_21(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 21-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_22(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 22-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_23(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 23-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_24(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 24-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_25(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 25-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_26(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 26-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_27(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 27-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_28(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 28-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_29(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 29-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_30(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 30-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_31(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 31-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void dtrsv_upper_kernel_32(double *dA, double *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    double rB;
    double rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 32-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        double bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}


__global__
void dtrsv_upper_kernel_switch(double *dA, double *dB, int *sizes, int num_rows )
{
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    if (j < num_rows) {
        int N = sizes[j];
        switch( N ) {
            case  1:
                dtrsv_upper_kernel_1( dA, dB ); break;
            case  2:
                dtrsv_upper_kernel_2( dA, dB ); break;
            case  3:
                dtrsv_upper_kernel_3( dA, dB ); break;
            case  4:
                dtrsv_upper_kernel_4( dA, dB ); break;
            case  5:
                dtrsv_upper_kernel_5( dA, dB ); break;
            case  6:
                dtrsv_upper_kernel_6( dA, dB ); break;
            case  7:
                dtrsv_upper_kernel_7( dA, dB ); break;
            case  8:
                dtrsv_upper_kernel_8( dA, dB ); break;
            case  9:
                dtrsv_upper_kernel_9( dA, dB ); break;
            case  10:
                dtrsv_upper_kernel_10( dA, dB ); break;
            case  11:
                dtrsv_upper_kernel_11( dA, dB ); break;
            case  12:
                dtrsv_upper_kernel_12( dA, dB ); break;
            case  13:
                dtrsv_upper_kernel_13( dA, dB ); break;
            case  14:
                dtrsv_upper_kernel_14( dA, dB ); break;
            case  15:
                dtrsv_upper_kernel_15( dA, dB ); break;
            case  16:
                dtrsv_upper_kernel_16( dA, dB ); break;
            case  17:
                dtrsv_upper_kernel_17( dA, dB ); break;
            case  18:
                dtrsv_upper_kernel_18( dA, dB ); break;
            case  19:
                dtrsv_upper_kernel_19( dA, dB ); break;
            case  20:
                dtrsv_upper_kernel_20( dA, dB ); break;
            case  21:
                dtrsv_upper_kernel_21( dA, dB ); break;
            case  22:
                dtrsv_upper_kernel_22( dA, dB ); break;
            case  23:
                dtrsv_upper_kernel_23( dA, dB ); break;
            case  24:
                dtrsv_upper_kernel_24( dA, dB ); break;
            case  25:
                dtrsv_upper_kernel_25( dA, dB ); break;
            case  26:
                dtrsv_upper_kernel_26( dA, dB ); break;
            case  27:
                dtrsv_upper_kernel_27( dA, dB ); break;
            case  28:
                dtrsv_upper_kernel_28( dA, dB ); break;
            case  29:
                dtrsv_upper_kernel_29( dA, dB ); break;
            case  30:
                dtrsv_upper_kernel_30( dA, dB ); break;
            case  31:
                dtrsv_upper_kernel_31( dA, dB ); break;
            case  32:
                dtrsv_upper_kernel_32( dA, dB ); break;
            default:
                dtrsv_upper_kernel_general( dA, dB, sizes ); break;
        }
    }
}
#endif
#endif
/**
    Purpose
    -------
    Does all triangular solves

    Arguments
    ---------


    @param[in]
    uplotype    magma_uplo_t
                lower or upper triangular

    @param[in]
    transtype   magma_trans_t
                possibility for transposed matrix

    @param[in]
    diagtype    magma_diag_t
                unit diagonal or not

    @param[in]
    L           magma_d_matrix
                Matrix in CSR format

    @param[in]
    LC          magma_d_matrix
                same matrix, also CSR, but col-major

    @param[out]
    sizes       magma_int_t*
                Number of Elements that are replaced.

    @param[out]
    locations   magma_int_t*
                Array indicating the locations.

    @param[out]
    trisystems  double*
                trisystems

    @param[out]
    rhs         double*
                right-hand sides

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_daux
    ********************************************************************/

extern "C" magma_int_t
magma_dmtrisolve_batched_gpu(
    magma_uplo_t uplotype,
    magma_trans_t transtype,
    magma_diag_t diagtype,
    magma_d_matrix L,
    magma_d_matrix LC,
    magma_index_t *sizes,
    magma_index_t *locations,
    double *trisystems,
    double *rhs,
    magma_queue_t queue )
{
    magma_int_t info = 0;

    int blocksize1 = WARP_SIZE;
    int blocksize2 = 1;
    int dimgrid1 = min( int( sqrt( double( LC.num_rows ))), 65535 );
    int dimgrid2 = min(magma_ceildiv( LC.num_rows, dimgrid1 ), 65535);
    int dimgrid3 = magma_ceildiv( LC.num_rows, dimgrid1*dimgrid2 );

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );

#if (CUDA_VERSION >= 7000)
#if (CUDA_ARCH >= 300)
    if( uplotype == MagmaLower ){
        //hipProfilerStart();
        dtrsv_lower_kernel_switch<<< grid, block, 0, queue->hip_stream() >>>(
                trisystems,
                rhs,
                sizes,
                LC.num_rows );
        //hipProfilerStop();
    } else {
        dtrsv_upper_kernel_switch<<< grid, block, 0, queue->hip_stream() >>>(
                trisystems,
                rhs,
                sizes,
                LC.num_rows );
    }
#endif
#endif

    return info;
}
