#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zmergecgs.cu, normal z -> s, Sun Nov 20 20:20:40 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512

#define PRECISION_s


// These routines merge multiple kernels from scgs into one.

/* -------------------------------------------------------------------------- */

__global__ void
magma_scgs_1_kernel(  
    int num_rows,
    int num_cols,
    float beta,
    float *r,
    float *q,
    float *u,
    float *p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            float tmp;
            tmp =  r[ i+j*num_rows ] + beta * q[ i+j*num_rows ];
            p[ i+j*num_rows ] = tmp + beta * q[ i+j*num_rows ] 
                                + beta * beta * p[ i+j*num_rows ];
            u[ i+j*num_rows ] = tmp;
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    u = r + beta q
    p = u + beta*(q + beta*p)

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    beta        float
                scalar

    @param[in]
    r           magmaFloat_ptr 
                vector

    @param[in]
    q           magmaFloat_ptr 
                vector

    @param[in,out]
    u           magmaFloat_ptr 
                vector
                
    @param[in,out]
    p           magmaFloat_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_scgs_1(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    float beta,
    magmaFloat_ptr r,
    magmaFloat_ptr q, 
    magmaFloat_ptr u,
    magmaFloat_ptr p,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_scgs_1_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, beta, r, q, u, p );

    return MAGMA_SUCCESS;
}


__global__ void
magma_scgs_2_kernel(  
    int num_rows,
    int num_cols,
    float *r,
    float *u,
    float *p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            float tmp;
            tmp = r[ i+j*num_rows ];
            u[ i+j*num_rows ] = tmp;
            p[ i+j*num_rows ] = tmp;
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    u = r
    p = r

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    r           magmaFloat_ptr 
                vector

    @param[in,out]
    u           magmaFloat_ptr 
                vector
                
    @param[in,out]
    p           magmaFloat_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_scgs_2(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloat_ptr r,
    magmaFloat_ptr u,
    magmaFloat_ptr p, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_scgs_2_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, r, u, p);

    return MAGMA_SUCCESS;
}


__global__ void
magma_scgs_3_kernel(  
    int num_rows,
    int num_cols,
    float alpha,
    float *v_hat,
    float *u,
    float *q,
    float *t )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            float uloc,  tmp;
            uloc = u[ i+j*num_rows ];
            tmp = uloc - alpha * v_hat[ i+j*num_rows ];
            t[ i+j*num_rows ] = tmp + uloc;
            q[ i+j*num_rows ] = tmp;
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    q = u - alpha v_hat
    t = u + q

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       float
                scalar
                
    @param[in]
    v_hat       magmaFloat_ptr 
                vector
    
    @param[in]
    u           magmaFloat_ptr 
                vector

    @param[in,out]
    q           magmaFloat_ptr 
                vector
                
    @param[in,out]
    t           magmaFloat_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_scgs_3(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    float alpha,
    magmaFloat_ptr v_hat,
    magmaFloat_ptr u, 
    magmaFloat_ptr q,
    magmaFloat_ptr t, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_scgs_3_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, v_hat, u, q, t );

    return MAGMA_SUCCESS;
}


__global__ void
magma_scgs_4_kernel(  
    int num_rows,
    int num_cols,
    float alpha,
    float *u_hat,
    float *t,
    float *x,
    float *r )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            x[ i+j*num_rows ] = x[ i+j*num_rows ] 
                                + alpha * u_hat[ i+j*num_rows ];
            r[ i+j*num_rows ] = r[ i+j*num_rows ] 
                                - alpha * t[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x = x + alpha u_hat
    r = r -alpha*A u_hat = r -alpha*t

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       float
                scalar
                
    @param[in]
    u_hat       magmaFloat_ptr 
                vector
                
    @param[in]
    t           magmaFloat_ptr 
                vector

    @param[in,out]
    x           magmaFloat_ptr 
                vector
                
    @param[in,out]
    r           magmaFloat_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_scgs_4(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    float alpha,
    magmaFloat_ptr u_hat,
    magmaFloat_ptr t,
    magmaFloat_ptr x, 
    magmaFloat_ptr r,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_scgs_4_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, u_hat, t, x, r );

    return MAGMA_SUCCESS;
}
