#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zgeaxpy.cu, normal z -> c, Sun Nov 20 20:20:39 2016

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 256


// axpy kernel for matrices stored in the MAGMA format
__global__ void 
cgeaxpy_kernel( 
    int num_rows, 
    int num_cols, 
    magmaFloatComplex alpha, 
    magmaFloatComplex * dx, 
    magmaFloatComplex beta, 
    magmaFloatComplex * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if( row<num_rows ){
        for( j=0; j<num_cols; j++ ){
            int idx = row + j*num_rows;
            dy[ idx ] = alpha * dx[ idx ] + beta * dy[ idx ];
        }
    }
}

/**
    Purpose
    -------
    
    This routine computes Y = alpha *  X + beta * Y on the GPU.
    The input format is a dense matrix (vector block) stored in 
    magma_c_matrix format.
    
    Arguments
    ---------

    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier.
                
    @param[in]
    X           magma_c_matrix
                input/output matrix Y.
                
    @param[in]
    beta        magmaFloatComplex
                scalar multiplier.
                
    @param[in,out]
    Y           magma_c_matrix*
                input matrix X.
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" 
magma_int_t
magma_cgeaxpy(
    magmaFloatComplex alpha,
    magma_c_matrix X,
    magmaFloatComplex beta,
    magma_c_matrix *Y,
    magma_queue_t queue )
{
    int m = X.num_rows;
    int n = X.num_cols;
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    cgeaxpy_kernel<<< grid, threads, 0, queue->hip_stream() >>>
                    ( m, n, alpha, X.dval, beta, Y->dval );
                    
    return MAGMA_SUCCESS;
}
