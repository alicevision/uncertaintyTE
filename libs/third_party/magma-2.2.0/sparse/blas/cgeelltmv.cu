#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zgeelltmv.cu, normal z -> c, Sun Nov 20 20:20:39 2016

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512


// ELL SpMV kernel
//Michael Garland
template<bool betazero>
__global__ void 
cgeelltmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magmaFloatComplex alpha, 
    magmaFloatComplex * dval, 
    magma_index_t * dcolind,
    magmaFloatComplex * dx,
    magmaFloatComplex beta, 
    magmaFloatComplex * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows ) {
        magmaFloatComplex dot = MAGMA_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row; n++ ) {
            int col = dcolind [ num_rows * n + row ];
            magmaFloatComplex val = dval [ num_rows * n + row ];
            //if ( val != MAGMA_C_ZERO )
                dot += val * dx[col ];
        }
        if (betazero) {
            dy[ row ] = dot * alpha;
        } else {
            dy[ row ] = dot * alpha + beta * dy [ row ];
        }
    }
}

// shifted ELL SpMV kernel
//Michael Garland
__global__ void 
cgeelltmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magmaFloatComplex alpha, 
    magmaFloatComplex lambda, 
    magmaFloatComplex * dval, 
    magma_index_t * dcolind,
    magmaFloatComplex * dx,
    magmaFloatComplex beta, 
    int offset,
    int blocksize,
    magma_index_t * addrows,
    magmaFloatComplex * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows ) {
        magmaFloatComplex dot = MAGMA_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row; n++ ) {
            int col = dcolind [ num_rows * n + row ];
            magmaFloatComplex val = dval [ num_rows * n + row ];
            if ( val != 0)
                dot += val * dx[col ];
        }
        if ( row < blocksize )
            dy[ row ] = dot * alpha - lambda 
                    * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                    * dx[ addrows[row-blocksize] ] + beta * dy [ row ];            
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier

    @param[in]
    dval        magmaFloatComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magmaFloatComplex_ptr
                input vector x

    @param[in]
    beta        magmaFloatComplex
                scalar multiplier

    @param[out]
    dy          magmaFloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cgeelltmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t nnz_per_row,
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr dval,
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta,
    magmaFloatComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    if (beta == MAGMA_C_ZERO) {
        cgeelltmv_kernel<true><<< grid, threads, 0, queue->hip_stream() >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );
    } else {
        cgeelltmv_kernel<false><<< grid, threads, 0, queue->hip_stream() >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );
    }


    return MAGMA_SUCCESS;
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------

    @param[in]
    transA      magma_trans_t
                transposition parameter for A    

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier

    @param[in]
    lambda      magmaFloatComplex
                scalar multiplier

    @param[in]
    dval        magmaFloatComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magmaFloatComplex_ptr
                input vector x

    @param[in]
    beta        magmaFloatComplex
                scalar multiplier
                
    @param[in]
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magmaIndex_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magmaFloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" magma_int_t
magma_cgeelltmv_shift(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t nnz_per_row,
    magmaFloatComplex alpha,
    magmaFloatComplex lambda,
    magmaFloatComplex_ptr dval,
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta,
    magma_int_t offset,
    magma_int_t blocksize,
    magmaIndex_ptr addrows,
    magmaFloatComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    magmaFloatComplex tmp_shift;
    //magma_csetvector(1,&lambda,1,&tmp_shift,1); 
    tmp_shift = lambda;
    cgeelltmv_kernel_shift<<< grid, threads, 0, queue->hip_stream() >>>
                  ( m, n, nnz_per_row, alpha, tmp_shift, dval, dcolind, dx, 
                            beta, offset, blocksize, addrows, dy );


    return MAGMA_SUCCESS;
}
