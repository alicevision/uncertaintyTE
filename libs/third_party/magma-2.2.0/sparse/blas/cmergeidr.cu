#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zmergeidr.cu, normal z -> c, Sun Nov 20 20:20:40 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512

#define PRECISION_c


// These routines merge multiple kernels from cidr into one.

/* -------------------------------------------------------------------------- */

__global__ void
magma_cidr_smoothing_1_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex *drs,
    magmaFloatComplex *dr,
    magmaFloatComplex *dt )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            dt[ i+j*num_rows ] =  drs[ i+j*num_rows ] - dr[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    dt = drs - dr

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n

    @param[in]
    drs         magmaFloatComplex_ptr 
                vector

    @param[in]
    dr          magmaFloatComplex_ptr 
                vector

    @param[in,out]
    dt          magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_cidr_smoothing_1(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex_ptr drs,
    magmaFloatComplex_ptr dr, 
    magmaFloatComplex_ptr dt, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_cidr_smoothing_1_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, drs, dr, dt );

    return MAGMA_SUCCESS;
}



__global__ void
magma_cidr_smoothing_2_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex omega,
    magmaFloatComplex *dx,
    magmaFloatComplex *dxs )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            dxs[ i+j*num_rows ] = dxs[ i+j*num_rows ] + omega * dxs[ i+j*num_rows ]
                    - omega * dx[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    dxs = dxs - gamma*(dxs-dx)

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    omega       magmaFloatComplex
                scalar
                
    @param[in]
    dx          magmaFloatComplex_ptr 
                vector

    @param[in,out]
    dxs         magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_cidr_smoothing_2(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex omega,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex_ptr dxs, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_cidr_smoothing_2_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, omega, dx, dxs);

    return MAGMA_SUCCESS;
}
