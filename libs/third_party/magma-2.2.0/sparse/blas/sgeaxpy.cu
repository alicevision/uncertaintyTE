#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zgeaxpy.cu, normal z -> s, Sun Nov 20 20:20:39 2016

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 256


// axpy kernel for matrices stored in the MAGMA format
__global__ void 
sgeaxpy_kernel( 
    int num_rows, 
    int num_cols, 
    float alpha, 
    float * dx, 
    float beta, 
    float * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if( row<num_rows ){
        for( j=0; j<num_cols; j++ ){
            int idx = row + j*num_rows;
            dy[ idx ] = alpha * dx[ idx ] + beta * dy[ idx ];
        }
    }
}

/**
    Purpose
    -------
    
    This routine computes Y = alpha *  X + beta * Y on the GPU.
    The input format is a dense matrix (vector block) stored in 
    magma_s_matrix format.
    
    Arguments
    ---------

    @param[in]
    alpha       float
                scalar multiplier.
                
    @param[in]
    X           magma_s_matrix
                input/output matrix Y.
                
    @param[in]
    beta        float
                scalar multiplier.
                
    @param[in,out]
    Y           magma_s_matrix*
                input matrix X.
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sblas
    ********************************************************************/

extern "C" 
magma_int_t
magma_sgeaxpy(
    float alpha,
    magma_s_matrix X,
    float beta,
    magma_s_matrix *Y,
    magma_queue_t queue )
{
    int m = X.num_rows;
    int n = X.num_cols;
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    sgeaxpy_kernel<<< grid, threads, 0, queue->hip_stream() >>>
                    ( m, n, alpha, X.dval, beta, Y->dval );
                    
    return MAGMA_SUCCESS;
}
