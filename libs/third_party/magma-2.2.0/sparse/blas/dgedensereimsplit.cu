#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zgedensereimsplit.cu, normal z -> d, Sun Nov 20 20:20:41 2016

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 256


// axpy kernel for matrices stored in the MAGMA format
__global__ void 
dgedensereimsplit_kernel( 
    int num_rows, 
    int num_cols,  
    magma_index_t* rowidx,
    double * A, 
    double * ReA, 
    double * ImA )
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if( row<num_rows ){
        for( j=0; j<num_cols; j++ ){
            ReA[ j ] = MAGMA_D_MAKE( MAGMA_D_REAL( A[ j ] ), 0.0 );
            ImA[ j ] = MAGMA_D_MAKE( MAGMA_D_IMAG( A[ j ] ), 0.0 );
        }
    }
}

/**
    Purpose
    -------
    
    This routine takes an input matrix A in DENSE format and located on the GPU
    and splits it into two matrixes ReA and ImA containing the real and the 
    imaginary contributions of A.
    The output matrices are allocated within the routine.
    
    Arguments
    ---------

    @param[in]
    A           magma_d_matrix
                input matrix A.
                
    @param[out]
    ReA         magma_d_matrix*
                output matrix contaning real contributions.
                
    @param[out]
    ImA         magma_d_matrix*
                output matrix contaning real contributions.
                
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" 
magma_int_t
magma_dgedensereimsplit(
    magma_d_matrix A,
    magma_d_matrix *ReA,
    magma_d_matrix *ImA,
    magma_queue_t queue )
{
    magma_dmtransfer( A, ReA, Magma_DEV, Magma_DEV, queue );
    magma_dmtransfer( A, ImA, Magma_DEV, Magma_DEV, queue );
        
    int m = A.num_rows;
    int n = A.num_cols;
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    dgedensereimsplit_kernel<<< grid, threads, 0, queue->hip_stream() >>>
                    ( m, n, A.row, A.dval, ReA->dval, ImA->dval );
                    
    return MAGMA_SUCCESS;
}
