#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zmergecgs.cu, normal z -> c, Sun Nov 20 20:20:41 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512

#define PRECISION_c


// These routines merge multiple kernels from ccgs into one.

/* -------------------------------------------------------------------------- */

__global__ void
magma_ccgs_1_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex beta,
    magmaFloatComplex *r,
    magmaFloatComplex *q,
    magmaFloatComplex *u,
    magmaFloatComplex *p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            magmaFloatComplex tmp;
            tmp =  r[ i+j*num_rows ] + beta * q[ i+j*num_rows ];
            p[ i+j*num_rows ] = tmp + beta * q[ i+j*num_rows ] 
                                + beta * beta * p[ i+j*num_rows ];
            u[ i+j*num_rows ] = tmp;
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    u = r + beta q
    p = u + beta*(q + beta*p)

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    beta        magmaFloatComplex
                scalar

    @param[in]
    r           magmaFloatComplex_ptr 
                vector

    @param[in]
    q           magmaFloatComplex_ptr 
                vector

    @param[in,out]
    u           magmaFloatComplex_ptr 
                vector
                
    @param[in,out]
    p           magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ccgs_1(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex beta,
    magmaFloatComplex_ptr r,
    magmaFloatComplex_ptr q, 
    magmaFloatComplex_ptr u,
    magmaFloatComplex_ptr p,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ccgs_1_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, beta, r, q, u, p );

    return MAGMA_SUCCESS;
}


__global__ void
magma_ccgs_2_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex *r,
    magmaFloatComplex *u,
    magmaFloatComplex *p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            magmaFloatComplex tmp;
            tmp = r[ i+j*num_rows ];
            u[ i+j*num_rows ] = tmp;
            p[ i+j*num_rows ] = tmp;
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    u = r
    p = r

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    r           magmaFloatComplex_ptr 
                vector

    @param[in,out]
    u           magmaFloatComplex_ptr 
                vector
                
    @param[in,out]
    p           magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ccgs_2(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex_ptr r,
    magmaFloatComplex_ptr u,
    magmaFloatComplex_ptr p, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ccgs_2_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, r, u, p);

    return MAGMA_SUCCESS;
}


__global__ void
magma_ccgs_3_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex alpha,
    magmaFloatComplex *v_hat,
    magmaFloatComplex *u,
    magmaFloatComplex *q,
    magmaFloatComplex *t )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            magmaFloatComplex uloc,  tmp;
            uloc = u[ i+j*num_rows ];
            tmp = uloc - alpha * v_hat[ i+j*num_rows ];
            t[ i+j*num_rows ] = tmp + uloc;
            q[ i+j*num_rows ] = tmp;
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    q = u - alpha v_hat
    t = u + q

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaFloatComplex
                scalar
                
    @param[in]
    v_hat       magmaFloatComplex_ptr 
                vector
    
    @param[in]
    u           magmaFloatComplex_ptr 
                vector

    @param[in,out]
    q           magmaFloatComplex_ptr 
                vector
                
    @param[in,out]
    t           magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ccgs_3(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr v_hat,
    magmaFloatComplex_ptr u, 
    magmaFloatComplex_ptr q,
    magmaFloatComplex_ptr t, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ccgs_3_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, v_hat, u, q, t );

    return MAGMA_SUCCESS;
}


__global__ void
magma_ccgs_4_kernel(  
    int num_rows,
    int num_cols,
    magmaFloatComplex alpha,
    magmaFloatComplex *u_hat,
    magmaFloatComplex *t,
    magmaFloatComplex *x,
    magmaFloatComplex *r )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            x[ i+j*num_rows ] = x[ i+j*num_rows ] 
                                + alpha * u_hat[ i+j*num_rows ];
            r[ i+j*num_rows ] = r[ i+j*num_rows ] 
                                - alpha * t[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x = x + alpha u_hat
    r = r -alpha*A u_hat = r -alpha*t

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaFloatComplex
                scalar
                
    @param[in]
    u_hat       magmaFloatComplex_ptr 
                vector
                
    @param[in]
    t           magmaFloatComplex_ptr 
                vector

    @param[in,out]
    x           magmaFloatComplex_ptr 
                vector
                
    @param[in,out]
    r           magmaFloatComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ccgs_4(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr u_hat,
    magmaFloatComplex_ptr t,
    magmaFloatComplex_ptr x, 
    magmaFloatComplex_ptr r,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ccgs_4_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, u_hat, t, x, r );

    return MAGMA_SUCCESS;
}
