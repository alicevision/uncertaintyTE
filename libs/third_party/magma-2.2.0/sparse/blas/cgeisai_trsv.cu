#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zgeisai_trsv.cu, normal z -> c, Sun Nov 20 20:20:43 2016

*/
#include "magmasparse_internal.h"
//#include <hip/hip_runtime_api.h>

#define PRECISION_c
#define COMPLEX
#define BLOCKSIZE 256
#define WARP_SIZE 32
#define WRP 32
#define WRQ 1



#include <hip/hip_runtime.h>  // for CUDA_VERSION

#if (CUDA_VERSION >= 7000)
#if (CUDA_ARCH >= 300)

__device__
void ctrsv_lower_kernel_general(magmaFloatComplex *dA, magmaFloatComplex *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;

    magmaFloatComplex rB[ 2 ];
    magmaFloatComplex rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;


    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];


    // Triangular solve in regs.
    #pragma unroll
    for (k = 0; k < N; k++)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];

        magmaFloatComplex top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
            if (n*WARP_SIZE+idn > k)
                rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];

#endif
}


__device__
void ctrsv_upper_kernel_general(magmaFloatComplex *dA, magmaFloatComplex *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;

    magmaFloatComplex rB[ 2 ];
    magmaFloatComplex rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;


    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];


    // Triangular solve in regs.
    #pragma unroll
    for (int k = N-1; k > -1; k--)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];

        magmaFloatComplex top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
            if (n*WARP_SIZE+idn < k)
                rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];

#endif
}



__device__
void ctrsv_lower_kernel_1(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 1; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_2(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 2; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_3(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 3; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_4(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 4; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_5(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 5; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_6(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 6; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_7(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 7; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_8(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 8; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_9(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 9; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_10(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 10; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_11(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 11; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_12(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 12; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_13(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 13; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_14(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 14; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_15(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 15; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_16(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 16; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_17(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 17; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_18(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 18; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_19(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 19; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_20(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 20; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_21(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 21; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_22(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 22; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_23(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 23; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_24(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 24; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_25(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 25; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_26(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 26; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_27(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 27; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_28(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 28; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_29(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 29; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_30(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 30; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_31(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 31; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_lower_kernel_32(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 32; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}


__global__
void ctrsv_lower_kernel_switch(magmaFloatComplex *dA, magmaFloatComplex *dB, int *sizes, int num_rows )
{
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    if (j < num_rows) {
        int N = sizes[j];
        switch( N ) {
            case  1:
                ctrsv_lower_kernel_1( dA, dB ); break;
            case  2:
                ctrsv_lower_kernel_2( dA, dB ); break;
            case  3:
                ctrsv_lower_kernel_3( dA, dB ); break;
            case  4:
                ctrsv_lower_kernel_4( dA, dB ); break;
            case  5:
                ctrsv_lower_kernel_5( dA, dB ); break;
            case  6:
                ctrsv_lower_kernel_6( dA, dB ); break;
            case  7:
                ctrsv_lower_kernel_7( dA, dB ); break;
            case  8:
                ctrsv_lower_kernel_8( dA, dB ); break;
            case  9:
                ctrsv_lower_kernel_9( dA, dB ); break;
            case  10:
                ctrsv_lower_kernel_10( dA, dB ); break;
            case  11:
                ctrsv_lower_kernel_11( dA, dB ); break;
            case  12:
                ctrsv_lower_kernel_12( dA, dB ); break;
            case  13:
                ctrsv_lower_kernel_13( dA, dB ); break;
            case  14:
                ctrsv_lower_kernel_14( dA, dB ); break;
            case  15:
                ctrsv_lower_kernel_15( dA, dB ); break;
            case  16:
                ctrsv_lower_kernel_16( dA, dB ); break;
            case  17:
                ctrsv_lower_kernel_17( dA, dB ); break;
            case  18:
                ctrsv_lower_kernel_18( dA, dB ); break;
            case  19:
                ctrsv_lower_kernel_19( dA, dB ); break;
            case  20:
                ctrsv_lower_kernel_20( dA, dB ); break;
            case  21:
                ctrsv_lower_kernel_21( dA, dB ); break;
            case  22:
                ctrsv_lower_kernel_22( dA, dB ); break;
            case  23:
                ctrsv_lower_kernel_23( dA, dB ); break;
            case  24:
                ctrsv_lower_kernel_24( dA, dB ); break;
            case  25:
                ctrsv_lower_kernel_25( dA, dB ); break;
            case  26:
                ctrsv_lower_kernel_26( dA, dB ); break;
            case  27:
                ctrsv_lower_kernel_27( dA, dB ); break;
            case  28:
                ctrsv_lower_kernel_28( dA, dB ); break;
            case  29:
                ctrsv_lower_kernel_29( dA, dB ); break;
            case  30:
                ctrsv_lower_kernel_30( dA, dB ); break;
            case  31:
                ctrsv_lower_kernel_31( dA, dB ); break;
            case  32:
                ctrsv_lower_kernel_32( dA, dB ); break;
            default:
                ctrsv_lower_kernel_general( dA, dB, sizes ); break;
        }
    }
}
__device__
void ctrsv_upper_kernel_1(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 1-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_2(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 2-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_3(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 3-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_4(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 4-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_5(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 5-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_6(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 6-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_7(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 7-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_8(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 8-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_9(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 9-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_10(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 10-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_11(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 11-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_12(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 12-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_13(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 13-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_14(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 14-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_15(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 15-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_16(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 16-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_17(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 17-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_18(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 18-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_19(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 19-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_20(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 20-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_21(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 21-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_22(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 22-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_23(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 23-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_24(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 24-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_25(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 25-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_26(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 26-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_27(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 27-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_28(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 28-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_29(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 29-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_30(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 30-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_31(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 31-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ctrsv_upper_kernel_32(magmaFloatComplex *dA, magmaFloatComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaFloatComplex rB;
    magmaFloatComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 32-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaFloatComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}


__global__
void ctrsv_upper_kernel_switch(magmaFloatComplex *dA, magmaFloatComplex *dB, int *sizes, int num_rows )
{
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    if (j < num_rows) {
        int N = sizes[j];
        switch( N ) {
            case  1:
                ctrsv_upper_kernel_1( dA, dB ); break;
            case  2:
                ctrsv_upper_kernel_2( dA, dB ); break;
            case  3:
                ctrsv_upper_kernel_3( dA, dB ); break;
            case  4:
                ctrsv_upper_kernel_4( dA, dB ); break;
            case  5:
                ctrsv_upper_kernel_5( dA, dB ); break;
            case  6:
                ctrsv_upper_kernel_6( dA, dB ); break;
            case  7:
                ctrsv_upper_kernel_7( dA, dB ); break;
            case  8:
                ctrsv_upper_kernel_8( dA, dB ); break;
            case  9:
                ctrsv_upper_kernel_9( dA, dB ); break;
            case  10:
                ctrsv_upper_kernel_10( dA, dB ); break;
            case  11:
                ctrsv_upper_kernel_11( dA, dB ); break;
            case  12:
                ctrsv_upper_kernel_12( dA, dB ); break;
            case  13:
                ctrsv_upper_kernel_13( dA, dB ); break;
            case  14:
                ctrsv_upper_kernel_14( dA, dB ); break;
            case  15:
                ctrsv_upper_kernel_15( dA, dB ); break;
            case  16:
                ctrsv_upper_kernel_16( dA, dB ); break;
            case  17:
                ctrsv_upper_kernel_17( dA, dB ); break;
            case  18:
                ctrsv_upper_kernel_18( dA, dB ); break;
            case  19:
                ctrsv_upper_kernel_19( dA, dB ); break;
            case  20:
                ctrsv_upper_kernel_20( dA, dB ); break;
            case  21:
                ctrsv_upper_kernel_21( dA, dB ); break;
            case  22:
                ctrsv_upper_kernel_22( dA, dB ); break;
            case  23:
                ctrsv_upper_kernel_23( dA, dB ); break;
            case  24:
                ctrsv_upper_kernel_24( dA, dB ); break;
            case  25:
                ctrsv_upper_kernel_25( dA, dB ); break;
            case  26:
                ctrsv_upper_kernel_26( dA, dB ); break;
            case  27:
                ctrsv_upper_kernel_27( dA, dB ); break;
            case  28:
                ctrsv_upper_kernel_28( dA, dB ); break;
            case  29:
                ctrsv_upper_kernel_29( dA, dB ); break;
            case  30:
                ctrsv_upper_kernel_30( dA, dB ); break;
            case  31:
                ctrsv_upper_kernel_31( dA, dB ); break;
            case  32:
                ctrsv_upper_kernel_32( dA, dB ); break;
            default:
                ctrsv_upper_kernel_general( dA, dB, sizes ); break;
        }
    }
}
#endif
#endif
/**
    Purpose
    -------
    Does all triangular solves

    Arguments
    ---------


    @param[in]
    uplotype    magma_uplo_t
                lower or upper triangular

    @param[in]
    transtype   magma_trans_t
                possibility for transposed matrix

    @param[in]
    diagtype    magma_diag_t
                unit diagonal or not

    @param[in]
    L           magma_c_matrix
                Matrix in CSR format

    @param[in]
    LC          magma_c_matrix
                same matrix, also CSR, but col-major

    @param[out]
    sizes       magma_int_t*
                Number of Elements that are replaced.

    @param[out]
    locations   magma_int_t*
                Array indicating the locations.

    @param[out]
    trisystems  magmaFloatComplex*
                trisystems

    @param[out]
    rhs         magmaFloatComplex*
                right-hand sides

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_caux
    ********************************************************************/

extern "C" magma_int_t
magma_cmtrisolve_batched_gpu(
    magma_uplo_t uplotype,
    magma_trans_t transtype,
    magma_diag_t diagtype,
    magma_c_matrix L,
    magma_c_matrix LC,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaFloatComplex *trisystems,
    magmaFloatComplex *rhs,
    magma_queue_t queue )
{
    magma_int_t info = 0;

    int blocksize1 = WARP_SIZE;
    int blocksize2 = 1;
    int dimgrid1 = min( int( sqrt( float( LC.num_rows ))), 65535 );
    int dimgrid2 = min(magma_ceildiv( LC.num_rows, dimgrid1 ), 65535);
    int dimgrid3 = magma_ceildiv( LC.num_rows, dimgrid1*dimgrid2 );

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );

#if (CUDA_VERSION >= 7000)
#if (CUDA_ARCH >= 300)
    if( uplotype == MagmaLower ){
        //hipProfilerStart();
        ctrsv_lower_kernel_switch<<< grid, block, 0, queue->hip_stream() >>>(
                trisystems,
                rhs,
                sizes,
                LC.num_rows );
        //hipProfilerStop();
    } else {
        ctrsv_upper_kernel_switch<<< grid, block, 0, queue->hip_stream() >>>(
                trisystems,
                rhs,
                sizes,
                LC.num_rows );
    }
#endif
#endif

    return info;
}
