#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/magma_zmconjugate.cu, normal z -> s, Sun Nov 20 20:20:41 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 256


__global__ void 
magma_smconjugate_kernel(  
    int num_rows,
    magma_index_t *rowptr, 
    float *values )
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < num_rows ){
        for( int i = rowptr[row]; i < rowptr[row+1]; i++){
            values[i] = MAGMA_S_CONJ( values[i] );
        }
    }
}



/**
    Purpose
    -------

    This function conjugates a matrix. For a real matrix, no value is changed.

    Arguments
    ---------

    @param[in,out]
    A           magma_s_matrix*
                input/output matrix
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_saux
    ********************************************************************/

extern "C" magma_int_t
magma_smconjugate(
    magma_s_matrix *A,
    magma_queue_t queue )
{
    magma_int_t info = 0;

    dim3 grid( magma_ceildiv( A->num_rows, BLOCK_SIZE ));
    magma_smconjugate_kernel<<< grid, BLOCK_SIZE, 0, queue->hip_stream() >>> 
                                    ( A->num_rows, A->drow, A->dval );
        
    return info;
}
