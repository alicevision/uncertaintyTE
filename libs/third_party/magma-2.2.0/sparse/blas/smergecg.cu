#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zmergecg.cu, normal z -> s, Sun Nov 20 20:20:42 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512

#define PRECISION_s


// These routines merge multiple kernels from smergecg into one
// for a description see 
// "Reformulated Conjugate Gradient for the Energy-Aware 
// Solution of Linear Systems on GPUs (ICPP '13)

// accelerated reduction for one vector
__global__ void
magma_scgreduce_kernel_spmv1( 
    int Gs,
    int n, 
    float * vtmp,
    float * vtmp2 )
{
    extern __shared__ float temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    temp[Idx] = MAGMA_S_MAKE( 0.0, 0.0);
    int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
    while (i < Gs ) {
        temp[ Idx  ] += vtmp[ i ]; 
        temp[ Idx  ] += ( i + blockSize < Gs ) ? vtmp[ i + blockSize ] 
                                                : MAGMA_S_MAKE( 0.0, 0.0); 
        i += gridSize;
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ) {
        vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}


// accelerated reduction for two vectors
__global__ void
magma_scgreduce_kernel_spmv2( 
    int Gs,
    int n, 
    float * vtmp,
    float * vtmp2 )
{
    extern __shared__ float temp[];     
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    int j;

    for( j=0; j<2; j++){
        int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
        temp[Idx+j*(blockSize)] = MAGMA_S_ZERO;
        while (i < Gs ) {
            temp[ Idx+j*(blockSize)  ] += vtmp[ i+j*n ]; 
            temp[ Idx+j*(blockSize)  ] += 
                ( i + (blockSize) < Gs ) ? vtmp[ i+j*n + (blockSize) ] 
                                                : MAGMA_S_ZERO;
            i += gridSize;
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp2[ blockIdx.x+j*n ] = temp[ j*(blockSize) ];
        }
    }
}



// computes the SpMV using CSR and the first step of the reduction
__global__ void
magma_scgmerge_spmvcsr_kernel(  
    int n,
    float * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    float * d,
    float * z,
    float * vtmp )
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    temp[ Idx ] = MAGMA_S_MAKE( 0.0, 0.0);

    if( i<n ) {
        float dot = MAGMA_S_ZERO;
        int start = drowptr[ i ];
        int end = drowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * d[ dcolind[j] ];
        z[ i ] =  dot;
        temp[ Idx ] =  d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

// computes the SpMV using ELL and the first step of the reduction
__global__ void
magma_scgmerge_spmvell_kernel(  
    int n,
    int num_cols_per_row,
    float * dval, 
    magma_index_t * dcolind,
    float * d,
    float * z,
    float * vtmp )
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = MAGMA_S_MAKE( 0.0, 0.0);

    if(i < n ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        for ( int k = 0; k < num_cols_per_row; k++ ) {
            int col = dcolind [ n * k + i ];
            float val = dval [ n * k + i ];
            if( val != 0)
                dot += val * d[ col ];
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// computes the SpMV using ELLPACK and the first step of the reduction
__global__ void
magma_scgmerge_spmvellpack_kernel(  
    int n,
    int num_cols_per_row,
    float * dval, 
    magma_index_t * dcolind,
    float * d,
    float * z,
    float * vtmp )
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = MAGMA_S_MAKE( 0.0, 0.0);

    if(i < n ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        for ( int k = 0; k < num_cols_per_row; k++ ) {
            int col = dcolind [ num_cols_per_row * i + k ];
            float val = dval [ num_cols_per_row * i + k ];
            if( val != 0)
                dot += val * d[ col ];
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// computes the SpMV using SELL alignment 1 and the first step of the reduction
__global__ void
magma_scgmerge_spmvell_kernelb1(  
    int n,
    int blocksize,
    float * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    float * d,
    float * z,
    float * vtmp )
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = MAGMA_S_MAKE( 0.0, 0.0);
    
    int idx = threadIdx.x;      // local row
    int bdx = blockIdx.x; // global block index
    int row = bdx * 256 + idx;  // global row index
    // int lblocksize = ( row + blocksize < num_rows) ? blocksize : ( num_rows - blocksize * (row/blocksize) );
    int lrow = threadIdx.x%blocksize; // local row;
    
    if( row < n ) {
        int offset = drowptr[ row/blocksize ];
        int border = (drowptr[ row/blocksize+1 ]-offset)/blocksize;
    
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        for ( int n = 0; n < border; n++) { 
            int col = dcolind [ offset+ blocksize * n + lrow ];
            float val = dval[ offset+ blocksize * n + lrow ];
            dot = dot + val * d [ col ];
        }
        z[ i ] = dot;
        temp[ Idx ] = d[ i ] * dot;
    }
    
/*
    if(i < n ) {
        int offset = drowptr[ blockIdx.x ];
        int border = (drowptr[ blockIdx.x+1 ]-offset)/blocksize;
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        for ( int k = 0; k < border; k++){ 
            int col = dcolind [ offset+ blocksize * k + threadIdx.x ];
            float val = dval[offset+ blocksize * k + threadIdx.x];
            if( val != 0){
                  dot += val*d[col];
            }
        }
        
        
        //float dot = MAGMA_S_MAKE(0.0, 0.0);
        //for ( int k = 0; k < num_cols_per_row; k++ ) {
        //    int col = dcolind [ n * k + i ];
        //    float val = dval [ n * k + i ];
        //    if( val != 0)
        //        dot += val * d[ col ];
        //}
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }*/

    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_scgmerge_spmvellpackrt_kernel_8(  
    int n,
    float * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    float * d,
    float * z,
    float * vtmp,
    magma_int_t T, 
    magma_int_t alignment  )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ float shared[];

    if(i < n ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //float val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            float val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ) {
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }
        }
    }
}

// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_scgmerge_spmvellpackrt_kernel_16(  
    int n,
    float * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    float * d,
    float * z,
    float * vtmp,
    magma_int_t T, 
    magma_int_t alignment  )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ float shared[];

    if(i < n ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //float val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            float val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ) {
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }
        }
    }
}

// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_scgmerge_spmvellpackrt_kernel_32(  
    int n,
    float * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    float * d,
    float * z,
    float * vtmp,
    magma_int_t T, 
    magma_int_t alignment  )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ float shared[];

    if(i < n ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //float val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            float val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ) {
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }
        }
    }
}


// additional kernel necessary to compute first reduction step
__global__ void
magma_scgmerge_spmvellpackrt_kernel2(  
    int n,
    float * z,
    float * d,
    float * vtmp2 )
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    

    temp[ Idx ] = ( i < n ) ? z[i]*d[i] : MAGMA_S_MAKE(0.0, 0.0);
    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}



// computes the SpMV using SELLC
__global__ void
magma_scgmerge_spmvsellc_kernel(   
    int num_rows, 
    int blocksize,
    float * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    float * d,
    float * z,
    float * vtmp)
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int offset = drowptr[ blockIdx.x ];
    int border = (drowptr[ blockIdx.x+1 ]-offset)/blocksize;

    temp[ Idx ] = MAGMA_S_MAKE( 0.0, 0.0);


    if(i < num_rows ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        for ( int n = 0; n < border; n ++) {
            int col = dcolind [offset+ blocksize * n + Idx ];
            float val = dval[offset+ blocksize * n + Idx];
            if( val != 0) {
                  dot=dot+val*d[col];
            }
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }
    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_scgmerge_spmvsellpt_kernel_8( 
    int num_rows, 
    int blocksize,
    int T,
    float * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    float * d,
    float * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y;      // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ float shared[];

    if(row < num_rows ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_; k++ ) {
            float val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 4 ) {
            shared[ldx]+=shared[ldx+blocksize*4];              
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }
        }
    }
}
// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_scgmerge_spmvsellpt_kernel_16( 
    int num_rows, 
    int blocksize,
    int T,
    float * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    float * d,
    float * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y;      // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ float shared[];

    if(row < num_rows ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_; k++ ) {
            float val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 8 ) {
            shared[ldx]+=shared[ldx+blocksize*8];              
            __syncthreads();
            if( idx < 4 ) shared[ldx]+=shared[ldx+blocksize*4];   
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }
        }
    }
}


// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_scgmerge_spmvsellpt_kernel_32( 
    int num_rows, 
    int blocksize,
    int T,
    float * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowptr,
    float * d,
    float * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y;      // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ float shared[];

    if(row < num_rows ) {
        float dot = MAGMA_S_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_; k++ ) {
            float val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 16 ) {
            shared[ldx]+=shared[ldx+blocksize*16];              
            __syncthreads();
            if( idx < 8 ) shared[ldx]+=shared[ldx+blocksize*8];  
            __syncthreads();
            if( idx < 4 ) shared[ldx]+=shared[ldx+blocksize*4];   
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }
        }
    }
}


// kernel to handle scalars
__global__ void // rho = beta/tmp; gamma = beta;
magma_scg_rhokernel(  
    float * skp ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ) {
        float tmp = skp[1];
        skp[3] = tmp/skp[4];
        skp[2] = tmp;
    }
}

/**
    Purpose
    -------

    Merges the first SpmV using different formats with the dot product 
    and the computation of rho

    Arguments
    ---------

    @param[in]
    A           magma_s_matrix
                input matrix 

    @param[in]
    d1          magmaFloat_ptr 
                temporary vector

    @param[in]
    d2          magmaFloat_ptr 
                temporary vector

    @param[in]
    dd          magmaFloat_ptr 
                input vector d

    @param[out]
    dz          magmaFloat_ptr 
                input vector z

    @param[out]
    skp         magmaFloat_ptr 
                array for parameters ( skp[3]=rho )

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_scgmerge_spmv1(
    magma_s_matrix A,
    magmaFloat_ptr d1,
    magmaFloat_ptr d2,
    magmaFloat_ptr dd,
    magmaFloat_ptr dz,
    magmaFloat_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( A.num_rows, local_block_size ) );
    dim3 Gs_next;
    int Ms =  local_block_size * sizeof( float ); 
    magmaFloat_ptr aux1 = d1, aux2 = d2;
    int b = 1;        

    if ( A.storage_type == Magma_CSR )
        magma_scgmerge_spmvcsr_kernel<<< Gs, Bs, Ms, queue->hip_stream() >>>
        ( A.num_rows, A.dval, A.drow, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_ELLPACKT )
        magma_scgmerge_spmvellpack_kernel<<< Gs, Bs, Ms, queue->hip_stream() >>>
        ( A.num_rows, A.max_nnz_row, A.dval, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_ELL )
        magma_scgmerge_spmvell_kernel<<< Gs, Bs, Ms, queue->hip_stream() >>>
        ( A.num_rows, A.max_nnz_row, A.dval, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_CUCSR ) {
        hipsparseHandle_t cusparseHandle = 0;
        hipsparseMatDescr_t descr = 0;
        float c_one = MAGMA_S_ONE;
        float c_zero = MAGMA_S_ZERO;
        hipsparseCreate( &cusparseHandle );
        hipsparseSetStream( cusparseHandle, queue->hip_stream() );
        hipsparseCreateMatDescr( &descr );
        hipsparseSetMatType( descr, HIPSPARSE_MATRIX_TYPE_GENERAL );
        hipsparseSetMatIndexBase( descr, HIPSPARSE_INDEX_BASE_ZERO );
        hipsparseScsrmv( cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
        A.num_rows, A.num_cols, A.nnz, &c_one, descr,
        A.dval, A.drow, A.dcol, dd, &c_zero, dz );
        hipsparseDestroyMatDescr( descr );
        hipsparseDestroy( cusparseHandle );
        cusparseHandle = 0;
        descr = 0;
        magma_scgmerge_spmvellpackrt_kernel2<<< Gs, Bs, Ms, queue->hip_stream() >>>
                      ( A.num_rows, dz, dd, d1 );
    }
    else if ( A.storage_type == Magma_SELLP && A.alignment == 1 ) {
            magma_scgmerge_spmvell_kernelb1<<< Gs, Bs, Ms, queue->hip_stream() >>>
            ( A.num_rows, A.blocksize, 
                A.dval, A.dcol, A.drow, dd, dz, d1 );
    }
    else if ( A.storage_type == Magma_SELLP && A.alignment > 1) {
            int num_threadssellp = A.blocksize*A.alignment;
            magma_int_t arch = magma_getdevice_arch();
            if ( arch < 200 && num_threadssellp > 256 )
                printf("error: too much shared memory requested.\n");

            dim3 block( A.blocksize, A.alignment, 1);
            int dimgrid1 = int( sqrt( float( A.numblocks )));
            int dimgrid2 = magma_ceildiv( A.numblocks, dimgrid1 );

            dim3 gridsellp( dimgrid1, dimgrid2, 1);
            int Mssellp = num_threadssellp * sizeof( float );

            if ( A.alignment == 8)
                magma_scgmerge_spmvsellpt_kernel_8
                <<< gridsellp, block, Mssellp, queue->hip_stream() >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else if ( A.alignment == 16)
                magma_scgmerge_spmvsellpt_kernel_16
                <<< gridsellp, block, Mssellp, queue->hip_stream() >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else if ( A.alignment == 32)
                magma_scgmerge_spmvsellpt_kernel_32
                <<< gridsellp, block, Mssellp, queue->hip_stream() >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else
                printf("error: alignment not supported.\n");

        // in case of using SELLP, we can't efficiently merge the 
        // dot product and the first reduction loop into the SpMV kernel
        // as the SpMV grid would result in low occupancy.
        magma_scgmerge_spmvellpackrt_kernel2<<< Gs, Bs, Ms, queue->hip_stream() >>>
                              ( A.num_rows, dz, dd, d1 );
    }
    else if ( A.storage_type == Magma_ELLRT ) {
        // in case of using ELLRT, we need a different grid, assigning
        // threads_per_row processors to each row
        // the block size is num_threads
        // fixed values


    int num_blocks = magma_ceildiv( A.num_rows, A.blocksize );

    int num_threads = A.alignment*A.blocksize;

    int real_row_length = magma_roundup( A.max_nnz_row, A.alignment );

    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = int( sqrt( float( num_blocks )));
    int dimgrid2 = magma_ceildiv( num_blocks, dimgrid1 );
    dim3 gridellrt( dimgrid1, dimgrid2, 1);

    int Mellrt = A.alignment * A.blocksize * sizeof( float );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if ( A.alignment == 32 ) {
        magma_scgmerge_spmvellpackrt_kernel_32
                <<< gridellrt, num_threads , Mellrt, queue->hip_stream() >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else if ( A.alignment == 16 ) {
        magma_scgmerge_spmvellpackrt_kernel_16
                <<< gridellrt, num_threads , Mellrt, queue->hip_stream() >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else if ( A.alignment == 8 ) {
        magma_scgmerge_spmvellpackrt_kernel_8
                <<< gridellrt, num_threads , Mellrt, queue->hip_stream() >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", int(A.alignment) );
        return MAGMA_ERR_NOT_SUPPORTED;
    }
        // in case of using ELLRT, we can't efficiently merge the 
        // dot product and the first reduction loop into the SpMV kernel
        // as the SpMV grid would result in low occupancy.

        magma_scgmerge_spmvellpackrt_kernel2<<< Gs, Bs, Ms, queue->hip_stream() >>>
                              ( A.num_rows, dz, dd, d1 );
    }

    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_scgreduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                                        ( Gs.x,  A.num_rows, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_scopyvector( 1, aux1, 1, skp+4, 1, queue );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_scg_rhokernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

    return MAGMA_SUCCESS;
}


/* -------------------------------------------------------------------------- */

// updates x and r and computes the first part of the dot product r*r
__global__ void
magma_scgmerge_xrbeta_kernel(  
    int n, 
    float * x, 
    float * r,
    float * d,
    float * z,
    float * skp,
    float * vtmp )
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    float rho = skp[3];
    float mrho = MAGMA_S_MAKE( -1.0, 0.0)*rho;

    temp[ Idx ] = MAGMA_S_MAKE( 0.0, 0.0);

    if( i<n ) {
        x[i] += rho * d[i];
        r[i] += mrho * z[i];
        temp[ Idx ] = r[i] * r[i];
    }
    __syncthreads();
    if ( Idx < 128 ) {
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ) {
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ) {
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ) {
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ) {
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

// kernel to handle scalars
__global__ void //alpha = beta / gamma
magma_scg_alphabetakernel(  
    float * skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ) {
        float tmp1 = skp[1];
        skp[0] =  tmp1/skp[2];
        //printf("beta=%e\n", MAGMA_S_REAL(tmp1));
    }
}

// update search Krylov vector d
__global__ void
magma_scg_d_kernel(  
    int n, 
    float * skp,
    float * r,
    float * d )
{
    int i   = blockIdx.x * blockDim.x + threadIdx.x;

    float alpha = skp[0];

    if( i<n ) {
        d[i] = r[i] + alpha * d[i];
    }
}



/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    d1          magmaFloat_ptr 
                temporary vector

    @param[in]
    d2          magmaFloat_ptr 
                temporary vector

    @param[in,out]
    dx          magmaFloat_ptr
                input vector x

    @param[in,out]
    dr          magmaFloat_ptr 
                input/output vector r

    @param[in]
    dd          magmaFloat_ptr 
                input vector d

    @param[in]
    dz          magmaFloat_ptr 
                input vector z
    @param[in]
    skp         magmaFloat_ptr 
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_ssygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_scgmerge_xrbeta(
    magma_int_t n,
    magmaFloat_ptr d1,
    magmaFloat_ptr d2,
    magmaFloat_ptr dx,
    magmaFloat_ptr dr,
    magmaFloat_ptr dd,
    magmaFloat_ptr dz, 
    magmaFloat_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( float ); 
    magmaFloat_ptr aux1 = d1, aux2 = d2;
    int b = 1;        
    magma_scgmerge_xrbeta_kernel<<< Gs, Bs, Ms, queue->hip_stream()>>>
                                    ( n, dx, dr, dd, dz, skp, d1);  



    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_scgreduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_scopyvector( 1, aux1, 1, skp+1, 1, queue );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_scg_alphabetakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

    dim3 Bs3( local_block_size );
    dim3 Gs3( magma_ceildiv( n, local_block_size ) );
    magma_scg_d_kernel<<< Gs3, Bs3, 0, queue->hip_stream()>>>( n, skp, dr, dd );  

    return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

// updates x and r
__global__ void
magma_spcgmerge_xrbeta_kernel(  
    int n, 
    float * x, 
    float * r,
    float * d,
    float * z,
    float * skp )
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    float rho = skp[3];
    float mrho = MAGMA_S_MAKE( -1.0, 0.0)*rho;

    if( i<n ) {
        x[i] += rho * d[i];
        r[i] += mrho * z[i];
    }
}


// dot product for multiple vectors
__global__ void
magma_smsdot_one_kernel_1( 
    int n, 
    float * v0,
    float * w0,
    float * vtmp)
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    // 1 vectors v(i)/w(i)
    
    temp[ Idx ]                 = ( i < n ) ?
                v0[ i ] * w0[ i ] : MAGMA_S_ZERO;
    temp[ Idx + blockDim.x ]    = ( i < n ) ?
                v0[ i ] * v0[ i ] : MAGMA_S_ZERO;
    
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif  
    
    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
            vtmp[ blockIdx.x+n ] = temp[ blockDim.x ];
    }
}

/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in,out]
    dx          magmaFloat_ptr
                input vector x

    @param[in,out]
    dr          magmaFloat_ptr 
                input/output vector r

    @param[in]
    dd          magmaFloat_ptr 
                input vector d

    @param[in]
    dz          magmaFloat_ptr 
                input vector z
    @param[in]
    skp         magmaFloat_ptr 
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_ssygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_spcgmerge_xrbeta1(
    magma_int_t n,
    magmaFloat_ptr dx,
    magmaFloat_ptr dr,
    magmaFloat_ptr dd,
    magmaFloat_ptr dz, 
    magmaFloat_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    magma_spcgmerge_xrbeta_kernel<<< Gs, Bs, 0, queue->hip_stream()>>>
                                    ( n, dx, dr, dd, dz, skp );  
                                    
    return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */


/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    d1          magmaFloat_ptr 
                temporary vector

    @param[in]
    d2          magmaFloat_ptr 
                temporary vector

    @param[in]
    dh          magmaFloat_ptr
                input vector x

    @param[in]
    dr          magmaFloat_ptr 
                input/output vector r
                
    @param[in]
    dd          magmaFloat_ptr 
                input/output vector d

    @param[in]
    skp         magmaFloat_ptr 
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_ssygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_spcgmerge_xrbeta2(
    magma_int_t n,
    magmaFloat_ptr d1,
    magmaFloat_ptr d2,
    magmaFloat_ptr dh,
    magmaFloat_ptr dr, 
    magmaFloat_ptr dd, 
    magmaFloat_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  4*local_block_size * sizeof( float ); 
    magmaFloat_ptr aux1 = d1, aux2 = d2;
    int b = 1;        
                                    
    magma_smsdot_one_kernel_1<<< Gs, Bs, Ms, queue->hip_stream()>>>
                                    ( n, dr, dh, d1);  

    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_scgreduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_scopyvector( 1, aux1, 1, skp+1, 1, queue );
    magma_scopyvector( 1, aux1+n, 1, skp+6, 1, queue );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_scg_alphabetakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

    dim3 Bs3( local_block_size );
    dim3 Gs3( magma_ceildiv( n, local_block_size ) );
    magma_scg_d_kernel<<< Gs3, Bs3, 0, queue->hip_stream()>>>( n, skp, dh, dd );  

    return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */



// updates x and r
__global__ void
magma_sjcgmerge_xrbeta_kernel(  
    int n, 
    float * diag, 
    float * x,     
    float * r,
    float * d,
    float * z,
    float * h,
    float * vtmp,
    float * skp )
{
    extern __shared__ float temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    float rho = skp[3];
    float mrho = MAGMA_S_MAKE( -1.0, 0.0)*rho;

    if( i<n ) {
        x[i] += rho * d[i];
        r[i] += mrho * z[i];
        h[i] = r[i] * diag[i];
    }
    __syncthreads();
    temp[ Idx ]                 = ( i < n ) ?
                h[ i ] * r[ i ] : MAGMA_S_ZERO;
    temp[ Idx + blockDim.x ]    = ( i < n ) ?
                r[ i ] * r[ i ] : MAGMA_S_ZERO;
    
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif  
    
    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
            vtmp[ blockIdx.x+n ] = temp[ blockDim.x ];
    }
}


/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    d1          magmaFloat_ptr 
                temporary vector

    @param[in]
    d2          magmaFloat_ptr 
                temporary vector
                
    @param[in]
    diag        magmaFloat_ptr 
                inverse diagonal (Jacobi preconditioner)

    @param[in]
    dx          magmaFloat_ptr
                iteration vector x

    @param[in]
    dr          magmaFloat_ptr 
                input/output vector r
                
    @param[in]
    dd          magmaFloat_ptr
                input vector d

                
    @param[in]
    dz          magmaFloat_ptr
                input vector z
                
    @param[in]
    dh          magmaFloat_ptr
                input vector h

    @param[in]
    skp         magmaFloat_ptr 
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_ssygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_sjcgmerge_xrbeta(
    magma_int_t n,
    magmaFloat_ptr d1,
    magmaFloat_ptr d2,
    magmaFloat_ptr diag,
    magmaFloat_ptr dx,
    magmaFloat_ptr dr,
    magmaFloat_ptr dd,
    magmaFloat_ptr dz,
    magmaFloat_ptr dh, 
    magmaFloat_ptr skp,
    magma_queue_t queue )
{
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  4*local_block_size * sizeof( float ); 
    magmaFloat_ptr aux1 = d1, aux2 = d2;
    int b = 1;    
                                    
    magma_sjcgmerge_xrbeta_kernel<<< Gs, Bs, Ms, queue->hip_stream() >>>
                                    ( n, diag, dx, dr, dd, dz, dh, d1, skp );  
                                    
    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_scgreduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream() >>> 
                                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_scopyvector( 1, aux1, 1, skp+1, 1, queue );
    magma_scopyvector( 1, aux1+n, 1, skp+6, 1, queue );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_scg_alphabetakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

    dim3 Bs3( local_block_size );
    dim3 Gs3( magma_ceildiv( n, local_block_size ) );
    magma_scg_d_kernel<<< Gs3, Bs3, 0, queue->hip_stream()>>>( n, skp, dh, dd );  

    return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */
