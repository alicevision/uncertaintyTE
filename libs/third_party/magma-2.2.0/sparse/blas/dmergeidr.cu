#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from sparse/blas/zmergeidr.cu, normal z -> d, Sun Nov 20 20:20:40 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512

#define PRECISION_d


// These routines merge multiple kernels from didr into one.

/* -------------------------------------------------------------------------- */

__global__ void
magma_didr_smoothing_1_kernel(  
    int num_rows,
    int num_cols,
    double *drs,
    double *dr,
    double *dt )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            dt[ i+j*num_rows ] =  drs[ i+j*num_rows ] - dr[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    dt = drs - dr

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n

    @param[in]
    drs         magmaDouble_ptr 
                vector

    @param[in]
    dr          magmaDouble_ptr 
                vector

    @param[in,out]
    dt          magmaDouble_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_didr_smoothing_1(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDouble_ptr drs,
    magmaDouble_ptr dr, 
    magmaDouble_ptr dt, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_didr_smoothing_1_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, drs, dr, dt );

    return MAGMA_SUCCESS;
}



__global__ void
magma_didr_smoothing_2_kernel(  
    int num_rows,
    int num_cols,
    double omega,
    double *dx,
    double *dxs )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            dxs[ i+j*num_rows ] = dxs[ i+j*num_rows ] + omega * dxs[ i+j*num_rows ]
                    - omega * dx[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    dxs = dxs - gamma*(dxs-dx)

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    omega       double
                scalar
                
    @param[in]
    dx          magmaDouble_ptr 
                vector

    @param[in,out]
    dxs         magmaDouble_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_didr_smoothing_2(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    double omega,
    magmaDouble_ptr dx,
    magmaDouble_ptr dxs, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_didr_smoothing_2_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, omega, dx, dxs);

    return MAGMA_SUCCESS;
}
