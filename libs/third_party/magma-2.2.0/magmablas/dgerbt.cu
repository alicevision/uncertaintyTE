#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zgerbt.cu, normal z -> d, Sun Nov 20 20:20:28 2016


       @author Adrien REMY
*/
#include "magma_internal.h"
#include "dgerbt.h"

#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64

/***************************************************************************//**
    Purpose
    -------
    DPRBT_MVT compute B = UTB to randomize B
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in]
    du     DOUBLE PRECISION array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in,out]
    db     DOUBLE PRECISION array, dimension (n)
            The n vector db computed by DGESV_NOPIV_GPU
            On exit db = du*db
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void
magmablas_dprbt_mtv(
    magma_int_t n, 
    double *du, double *db,
    magma_queue_t queue)
{
    /*

     */
    magma_int_t threads = block_length;
    magma_int_t grid = magma_ceildiv( n, 4*block_length );

    magmablas_dapply_transpose_vector_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n/2, du, n, db, 0);
    magmablas_dapply_transpose_vector_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n/2, du, n+n/2, db, n/2);

    threads = block_length;
    grid = magma_ceildiv( n, 2*block_length );
    magmablas_dapply_transpose_vector_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n, du, 0, db, 0);
}


/***************************************************************************//**
    Purpose
    -------
    DPRBT_MV compute B = VB to obtain the non randomized solution
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.
    
    @param[in,out]
    db      DOUBLE PRECISION array, dimension (n)
            The n vector db computed by DGESV_NOPIV_GPU
            On exit db = dv*db
    
    @param[in]
    dv      DOUBLE PRECISION array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void
magmablas_dprbt_mv(
    magma_int_t n, 
    double *dv, double *db,
    magma_queue_t queue)
{
    magma_int_t threads = block_length;
    magma_int_t grid = magma_ceildiv( n, 2*block_length );

    magmablas_dapply_vector_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n, dv, 0, db, 0);

    threads = block_length;
    grid = magma_ceildiv( n, 4*block_length );

    magmablas_dapply_vector_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dv, n, db, 0);
    magmablas_dapply_vector_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dv, n+n/2, db, n/2);
}


/***************************************************************************//**
    Purpose
    -------
    DPRBT randomize a square general matrix using partial randomized transformation
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns and rows of the matrix dA.  n >= 0.
    
    @param[in,out]
    dA      DOUBLE PRECISION array, dimension (n,ldda)
            The n-by-n matrix dA
            On exit dA = duT*dA*d_V
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDA >= max(1,n).
    
    @param[in]
    du      DOUBLE PRECISION array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix U
    
    @param[in]
    dv      DOUBLE PRECISION array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void 
magmablas_dprbt(
    magma_int_t n, 
    double *dA, magma_int_t ldda, 
    double *du, double *dv,
    magma_queue_t queue)
{
    du += ldda;
    dv += ldda;

    dim3 threads(block_height, block_width);
    dim3 grid( magma_ceildiv( n, 4*block_height ), 
               magma_ceildiv( n, 4*block_width  ));

    magmablas_delementary_multiplication_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA,            0, ldda, du,   0, dv,   0);
    magmablas_delementary_multiplication_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA,     ldda*n/2, ldda, du,   0, dv, n/2);
    magmablas_delementary_multiplication_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA,          n/2, ldda, du, n/2, dv,   0);
    magmablas_delementary_multiplication_kernel<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA, ldda*n/2+n/2, ldda, du, n/2, dv, n/2);

    dim3 threads2(block_height, block_width);
    dim3 grid2( magma_ceildiv( n, 2*block_height ), 
                magma_ceildiv( n, 2*block_width  ));
    magmablas_delementary_multiplication_kernel<<< grid2, threads2, 0, queue->hip_stream() >>>(n, dA, 0, ldda, du, -ldda, dv, -ldda);
}
