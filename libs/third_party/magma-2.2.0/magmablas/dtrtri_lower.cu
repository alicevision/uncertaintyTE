#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/ztrtri_lower.cu, normal z -> d, Sun Nov 20 20:20:29 2016

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       This file implements lower case, and is called by dtrtri_kernel.cu.
       It's convenient to have separate files for lower & upper, to diff the sources.
*/

#include "magma_internal.h"

#define TRTRI_NONBATCHED
#include "dtrtri.cuh"
#include "dtrtri_lower_device.cuh"


/******************************************************************************/
__global__ void
dtrtri_diag_lower_kernel(
    magma_diag_t diag, int n, const double *A, int lda, double *d_dinvA)
{
    dtrtri_diag_lower_device(diag, n, A, lda, d_dinvA);
}


/******************************************************************************/
__global__ void
triple_dgemm16_part1_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm16_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm16_part2_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm16_part2_lower_device( n,  Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm32_part1_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm32_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm32_part2_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm32_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm64_part1_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm64_part2_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm_above64_part1_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm_above64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm_above64_part2_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm_above64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm_above64_part3_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm_above64_part3_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}
