#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/ztrtri_lower.cu, normal z -> c, Sun Nov 20 20:20:30 2016

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       This file implements lower case, and is called by ctrtri_kernel.cu.
       It's convenient to have separate files for lower & upper, to diff the sources.
*/

#include "magma_internal.h"

#define TRTRI_NONBATCHED
#include "ctrtri.cuh"
#include "ctrtri_lower_device.cuh"


/******************************************************************************/
__global__ void
ctrtri_diag_lower_kernel(
    magma_diag_t diag, int n, const magmaFloatComplex *A, int lda, magmaFloatComplex *d_dinvA)
{
    ctrtri_diag_lower_device(diag, n, A, lda, d_dinvA);
}


/******************************************************************************/
__global__ void
triple_cgemm16_part1_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm16_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm16_part2_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm16_part2_lower_device( n,  Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm32_part1_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm32_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm32_part2_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm32_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm64_part1_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm64_part2_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part1_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part2_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part3_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part3_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}
