#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zlascl_2x2.cu, normal z -> d, Sun Nov 20 20:20:29 2016

       @author Ichitaro Yamazaki
*/
#include "magma_internal.h"

#define NB 64
#define A(i,j) (A[(i) + (j)*lda])
#define W(i,j) (W[(i) + (j)*ldw])


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
dlascl_2x2_lower(
    int m,
    const double* W, int ldw,
    double* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    double D21 = W( 1, 0 );
    double D11 = MAGMA_D_DIV( W( 1, 1 ), D21 );
    double D22 = MAGMA_D_DIV( W( 0, 0 ), MAGMA_D_CONJ( D21 ) );
    double T = 1.0 / ( MAGMA_D_REAL( D11*D22 ) - 1.0 );
    D21 = MAGMA_D_DIV( MAGMA_D_MAKE(T,0.0), D21 );

    if (ind < m) {
        A( ind, 0 ) = MAGMA_D_CONJ( D21 )*( D11*W( 2+ind, 0 )-W( 2+ind, 1 ) );
        A( ind, 1 ) = D21*( D22*W( 2+ind, 1 )-W( 2+ind, 0 ) );
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
dlascl_2x2_upper(
    int m,
    const double *W, int ldw,
    double* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    double D21 = W( m, 1 );
    double D11 = MAGMA_D_DIV( W( m+1, 1 ), MAGMA_D_CONJ( D21 ) );
    double D22 = MAGMA_D_DIV( W( m, 0 ), D21 );
    double T = 1.0 / ( MAGMA_D_REAL( D11*D22 ) - 1.0 );
    D21 = MAGMA_D_DIV( MAGMA_D_MAKE(T,0.0), D21 );

    if (ind < m) {
        A( ind, 0 ) = D21*( D11*W( ind, 0 )-W( ind, 1 ) );
        A( ind, 1 ) = MAGMA_D_CONJ( D21 )*( D22*W( ind, 1 )-W( ind, 0 ) );
    }
}


/***************************************************************************//**
    Purpose
    -------
    DLASCL_2x2 scales the M by M real matrix A by the 2-by-2 pivot.
    TYPE specifies that A may be upper or lower triangular.

    Arguments
    ---------
    @param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    dW      DOUBLE PRECISION vector, dimension (2*lddw)
            The matrix containing the 2-by-2 pivot.

    @param[in]
    lddw    INTEGER
            The leading dimension of the array W.  LDDA >= max(1,M).

    @param[in,out]
    dA      DOUBLE PRECISION array, dimension (LDDA,N)
            The matrix to be scaled by dW.  See TYPE for the
            storage type.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_lascl_2x2
*******************************************************************************/
extern "C" void
magmablas_dlascl_2x2(
    magma_type_t type, magma_int_t m,
    magmaDouble_const_ptr dW, magma_int_t lddw,
    magmaDouble_ptr       dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( ldda < max(1,m) )
        *info = -4;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    
    if (type == MagmaLower) {
        dlascl_2x2_lower <<< grid, threads, 0, queue->hip_stream() >>> (m, dW, lddw, dA, ldda);
    }
    else {
        dlascl_2x2_upper <<< grid, threads, 0, queue->hip_stream() >>> (m, dW, lddw, dA, ldda);
    }
}
