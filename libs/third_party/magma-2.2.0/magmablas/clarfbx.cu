#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zlarfbx.cu, normal z -> c, Sun Nov 20 20:20:29 2016

*/
#include "magma_internal.h"
#include "commonblas_c.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512


/******************************************************************************/
extern "C"
__global__ void 
magma_cgemv_kernel1(int m, const magmaFloatComplex * __restrict__ V, int ldv, 
                    const magmaFloatComplex * __restrict__ c, 
                    magmaFloatComplex *dwork)
{
    const int i = threadIdx.x;
    const magmaFloatComplex *dV = V + (blockIdx.x) * ldv;

    __shared__ magmaFloatComplex sum[ BLOCK_SIZE ];
    magmaFloatComplex lsum;

    /*  lsum := v**H * C  */
    lsum = MAGMA_C_ZERO;
    for (int j = i; j < m; j += BLOCK_SIZE)
       lsum += MAGMA_C_MUL( MAGMA_C_CONJ( dV[j] ), c[j] );
    
    sum[i] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i == 0)
       dwork [blockIdx.x] = sum[0];
}

/******************************************************************************/
/*
    Call 
        magma_cgemv_kernel3<<< n, BLOCK_SIZE, 0, queue->hip_stream() >>>(m, V, ldv, c, dwork, tau)
    to compute
        CGEMV( "Conjugate transpose", m, n, -tau[0], V, ldv, c, 1, zero, dwork, 1)
        and to set c[0] to 1.
    i.e., 
        work = -tau[0] V**H c
*/
extern "C"
__global__ void
magma_cgemv_kernel3(int m, const magmaFloatComplex * __restrict__ V, int ldv, magmaFloatComplex *c,
                    magmaFloatComplex *dwork, magmaFloatComplex *tau)
{
    const int i = threadIdx.x;
    const magmaFloatComplex *dV = V + (blockIdx.x) * ldv;

    __shared__ magmaFloatComplex sum[ BLOCK_SIZE ];
    magmaFloatComplex lsum;

    if (i == 0)
       c[0] = MAGMA_C_ONE;           

    /*  lsum := v**H * C  */
    lsum = MAGMA_C_ZERO;
    for (int j = i; j < m; j += BLOCK_SIZE)
       lsum += MAGMA_C_MUL( MAGMA_C_CONJ( dV[j] ), c[j] );

    sum[i] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i == 0)
       dwork [blockIdx.x] = -tau[0]*sum[0];
}


/******************************************************************************/
extern "C"
__global__ void
magma_cgemv_kernel2(int m, int n, const magmaFloatComplex * __restrict__ V, int ldv, 
                    const magmaFloatComplex * __restrict__ x, magmaFloatComplex *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    magmaFloatComplex lsum;

    V += j;

    lsum = MAGMA_C_ZERO;
    if (j < m) {
        for (int k=0; k < n; k++)
            lsum += MAGMA_C_MUL( V[k*ldv], x[k]);
        
        c[j] -= lsum;
    }
}


/******************************************************************************/
/*
    Apply a complex block reflector H to a complex vector C from the left
    (i.e., C = H C). H is represented in the form
          H = I - V T V**H
    where T is the complex k-by-k upper triangular matrix in the 
    representation of the block reflector, and V is a complex block of
    k elementary reflectors. 
*/
extern "C" void
magma_clarfbx_gpu(
    magma_int_t m, magma_int_t k,
    magmaFloatComplex_ptr V,  magma_int_t ldv,
    magmaFloatComplex_ptr dT, magma_int_t ldt,
    magmaFloatComplex_ptr c,
    magmaFloatComplex_ptr dwork,
    magma_queue_t queue )
{
    /* dwork = V**H c     */
    magma_cgemv_kernel1
        <<< k, BLOCK_SIZE, 0, queue->hip_stream() >>>
        (m, V, ldv, c, dwork); 

    /* dwork = T**H dwork */
    magma_ctrmv_tkernel
        <<< k, k, 0, queue->hip_stream() >>>
        ( dT, ldt, dwork, dwork+k);
 
    /* c = c - V dwork    */
    dim3  blocks3( magma_ceildiv( m, BLOCK_SIZE ) );
    dim3 threads3( BLOCK_SIZE );     
    magma_cgemv_kernel2
        <<< blocks3, threads3, 0, queue->hip_stream() >>>
        ( m, k, V, ldv, dwork+k, c);
}
