#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zsymmetrize.cu, normal z -> d, Sun Nov 20 20:20:29 2016
       @author Mark Gates
*/
#include "magma_internal.h"

#define NB 64

/*
    Matrix is m x m, and is divided into block rows, each NB x m.
    Each block has NB threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
dsymmetrize_lower( int m, double *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    double *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        double *dAend = dA + i*ldda;  // end at diagonal dA(i,i)
        while( dA < dAend ) {
            *dAT = MAGMA_D_CONJ(*dA);  // upper := lower
            dA  += ldda;
            dAT += 1;
        }
        *dA = MAGMA_D_MAKE( MAGMA_D_REAL(*dA), 0 );  // make diagonal real
    }
}


// only difference with _lower version is direction dA=dAT instead of dAT=dA.
__global__ void
dsymmetrize_upper( int m, double *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    double *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        double *dAend = dA + i*ldda;  // end at diagonal dA(i,i)
        while( dA < dAend ) {
            *dA = MAGMA_D_CONJ(*dAT);  // lower := upper
            dA  += ldda;
            dAT += 1;
        }
        *dA = MAGMA_D_MAKE( MAGMA_D_REAL(*dA), 0 );  // make diagonal real
    }
}


/***************************************************************************//**
    Purpose
    -------
    
    DSYMMETRIZE copies lower triangle to upper triangle, or vice-versa,
    to make dA a general representation of a symmetric matrix.
    In Complex, it sets the diagonal to be Real.
    
    Arguments
    ---------
    
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of the matrix dA that is valid on input.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in,out]
    dA      DOUBLE PRECISION array, dimension (LDDA,N)
            The m by m matrix dA.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_symmetrize
*******************************************************************************/
extern "C" void
magmablas_dsymmetrize(
    magma_uplo_t uplo, magma_int_t m,
    magmaDouble_ptr dA, magma_int_t ldda,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( uplo != MagmaLower && uplo != MagmaUpper )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( ldda < max(1,m) )
        info = -4;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 )
        return;
    
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    
    if ( uplo == MagmaUpper ) {
        dsymmetrize_upper<<< grid, threads, 0, queue->hip_stream() >>>( m, dA, ldda );
    }
    else {
        dsymmetrize_lower<<< grid, threads, 0, queue->hip_stream() >>>( m, dA, ldda );
    }
}
