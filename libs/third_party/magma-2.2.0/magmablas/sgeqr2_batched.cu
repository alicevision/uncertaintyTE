#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @author Azzam Haidar
       @author Tingxing Dong

       @generated from magmablas/zgeqr2_batched.cu, normal z -> s, Sun Nov 20 20:20:31 2016
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "batched_kernel_param.h"

#define BLOCK_SIZE 256


#define dA(a_1,a_2) (dA  + (a_1) + (a_2)*(local_lda))


#include "slarfg_devicesfunc.cuh"

/******************************************************************************/
static __device__
void slarfx_device(
    int m, int n,  float *v, float *tau,
    float *dc, magma_int_t ldc, float* sum)
{
    if (n <= 0) return;
    if (MAGMA_S_EQUAL(*tau, MAGMA_S_ZERO) )  return; // check singularity

    const int tx = threadIdx.x;

    float lsum;
    
    for (int k=0; k < n; k++)
    {
        /* perform  w := v' * C  */
        if (tx < BLOCK_SIZE)
        {
            if (tx == 0)
                lsum = dc[0+ldc*k]; //since V[0] should be one
            else
                lsum = MAGMA_S_ZERO;
            for (int j = tx+1; j < m; j += BLOCK_SIZE) {
                lsum += MAGMA_S_MUL( MAGMA_S_CONJ( v[j] ), dc[j+ldc*k] );
            }

            sum[tx] = lsum;
        }

        magma_sum_reduce< BLOCK_SIZE >( tx, sum );
        __syncthreads();

        float z__1 = - MAGMA_S_CONJ(*tau) * sum[0];
        /*  C := C - v * w  */
        if (tx < BLOCK_SIZE)
        {
            for (int j = tx+1; j < m; j += BLOCK_SIZE)
                dc[j+ldc*k] += z__1 * v[j];
        }
        if (tx == 0) dc[0+ldc*k] += z__1;

        __syncthreads();
    }
}


/******************************************************************************/
static __device__
void sgeqr2_device( magma_int_t m, magma_int_t n,
                               float* dA, magma_int_t lda,
                               float *dtau,
                               float *dv,
                               float *sum,
                               float *swork,
                               float *scale,
                               float *sscale)
{
    //lapack slarfg, compute the norm, scale and generate the householder vector
    slarfg_device(m, dv, &(dv[1]), 1, dtau, swork, sscale, scale);
    
    __syncthreads();
    
    //update the trailing matix with the householder
    slarfx_device(m, n, dv, dtau, dA, lda, sum);
    
    __syncthreads();
}


/******************************************************************************/
extern __shared__ float shared_data[];


/******************************************************************************/
__global__
void sgeqr2_sm_kernel_batched( int m, int n, float** dA_array, magma_int_t lda,
                               float **dtau_array)
{
    float* dA = dA_array[blockIdx.z];
    float* dtau = dtau_array[blockIdx.z];

    float *sdata = (float*)shared_data;

    const int tx = threadIdx.x;

    __shared__ float scale;
    __shared__ float sum[ BLOCK_SIZE ];

    __shared__ float swork[ BLOCK_SIZE ];
    __shared__ float sscale;
    
    //load data from global to shared memory
    for (int s=0; s < n; s++)
    {
        for (int j = tx; j < m; j += BLOCK_SIZE)
        {
            sdata[j + s * m] = dA[j + s * lda];
        }
    }


    __syncthreads();
 
    for (int s=0; s < min(m,n); s++)
    {
        sgeqr2_device( m-s, n-(s+1),
                       &(sdata[s+(s+1)*m]), m,
                       dtau+s,
                       &(sdata[s+s*m]),
                       sum,
                       swork,
                       &scale,
                       &sscale);
    } // end of s

    //copy back to global memory
    for (int s=0; s < n; s++)
    {
        for (int j = tx; j < m; j += BLOCK_SIZE)
        {
            dA[j + s * lda] = sdata[j + s * m];
        }
    }
}


/******************************************************************************/
__global__
void sgeqr2_column_sm_kernel_batched( int m, int n, float** dA_array, magma_int_t lda,
                               float **dtau_array)
{
    float* dA = dA_array[blockIdx.z];
    float* dtau = dtau_array[blockIdx.z];

    float *sdata = (float*)shared_data;


    __shared__ float scale;
    __shared__ float sum[ BLOCK_SIZE ];

    __shared__ float swork[ BLOCK_SIZE ];
    __shared__ float sscale;

    const int tx = threadIdx.x;

    for (int s=0; s < min(m,n); s++)
    {
        //load one vector in shared memory: sdata
        for (int j = tx; j < m-s; j += BLOCK_SIZE)
        {
            sdata[j] = dA[s + j + s * lda];
        }
        
        __syncthreads();
        
        //sdata is written
        sgeqr2_device(m-s, n-(s+1),
                                &(dA[s+(s+1)*lda]), lda,
                                dtau+s,
                                sdata,
                                sum,
                                swork,
                                &scale,
                                &sscale);
        
        for (int j = tx; j < m-s; j += BLOCK_SIZE)
        {
            dA[s + j + s * lda] = sdata[j];
        }
        
        __syncthreads();
    }  
}


/******************************************************************************/
__global__
void sgeqr2_kernel_batched( int m, int n, float** dA_array, magma_int_t lda,
                               float **dtau_array)
{
    float* dA = dA_array[blockIdx.z];
    float* dtau = dtau_array[blockIdx.z];

    __shared__ float scale;
    __shared__ float sum[ BLOCK_SIZE ];

    __shared__ float swork[ BLOCK_SIZE ];
    __shared__ float sscale;



    for (int s=0; s < min(m,n); s++)
    {
        sgeqr2_device( m-s, n-(s+1),
                       &(dA[s+(s+1)*lda]), lda,
                       dtau+s,
                       &(dA[s+s*lda]),
                       sum,
                       swork,
                       &scale,
                       &sscale );
    }
}


/***************************************************************************//**
    Purpose
    -------
    SGEQR2 computes a QR factorization of a real m by n matrix A:
    A = Q * R.

    This version implements the right-looking QR with non-blocking.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in,out]
    dA_array Array of pointers, dimension (batchCount).
             Each is a REAL array on the GPU, dimension (LDDA,N)
             On entry, the M-by-N matrix A.
             On exit, the elements on and above the diagonal of the array
             contain the min(M,N)-by-N upper trapezoidal matrix R (R is
             upper triangular if m >= n); the elements below the diagonal,
             with the array TAU, represent the orthogonal matrix Q as a
             product of min(m,n) elementary reflectors (see Further
             Details).

    @param[in]
    ldda     INTEGER
             The leading dimension of the array dA.  LDDA >= max(1,M).
             To benefit from coalescent memory accesses LDDA must be
             divisible by 16.

    @param[out]
    dtau_array Array of pointers, dimension (batchCount).
             Each is a REAL array, dimension (min(M,N))
             The scalar factors of the elementary reflectors (see Further
             Details).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
                  or another error occured, such as memory allocation failed.

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    Further Details
    ---------------
    The matrix Q is represented as a product of elementary reflectors

       Q = H(1) H(2) . . . H(k), where k = min(m,n).

    Each H(i) has the form

       H(i) = I - tau * v * v'

    where tau is a real scalar, and v is a real vector with
    v(1:i-1) = 0 and v(i) = 1; v(i+1:m) is stored on exit in A(i+1:m,i),
    and tau in TAU(i).

    @ingroup magma_geqr2_batched
*******************************************************************************/
extern "C" magma_int_t
magma_sgeqr2_batched(magma_int_t m, magma_int_t n, 
                     float **dA_array, magma_int_t ldda, 
                     float **dtau_array,
                     magma_int_t *info_array, magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t k;

    /* Check arguments */
    magma_int_t arginfo = 0;
    if (m < 0)
        arginfo = -1;
    else if (n < 0)
        arginfo = -2;
    else if (ldda < max(1,m))
        arginfo = -4;

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }

    k = min(m,n);

    dim3 blocks(1, 1, batchCount);
    dim3 threads(BLOCK_SIZE);

    if (sizeof(float)*(m*k) <= 42000 /*sizeof(float) * 128 * k*/) // there are some static shared memory besides of dynamic ones
    {
        //load panel in shared memory and factorize it and copy back to gloabl memory
        //intend for small panel to avoid overfill of shared memory.
        //this kernel is composed of device routine and thus clean
        sgeqr2_sm_kernel_batched<<< blocks, threads, sizeof(float)*(m*k), queue->hip_stream() >>>
                                      (m, k, dA_array, ldda, dtau_array);
    }
    else
    {
        //load one column vector in shared memory and householder it and used it to update trailing matrix which is global memory
        // one vector is normally smaller than  48K shared memory
        if (sizeof(float)*(m) < 42000)
            sgeqr2_column_sm_kernel_batched<<< blocks, threads, sizeof(float)*(m), queue->hip_stream() >>>
                                      (m, k, dA_array, ldda, dtau_array);
        else
            //not use dynamic shared memory at all
            sgeqr2_kernel_batched<<< blocks, threads, 0, queue->hip_stream() >>>
                                      (m, k, dA_array, ldda, dtau_array);
    }

    return arginfo;
}
