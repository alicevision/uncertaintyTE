#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zlarfbx.cu, normal z -> s, Sun Nov 20 20:20:29 2016

*/
#include "magma_internal.h"
#include "commonblas_s.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512


/******************************************************************************/
extern "C"
__global__ void 
magma_sgemv_kernel1(int m, const float * __restrict__ V, int ldv, 
                    const float * __restrict__ c, 
                    float *dwork)
{
    const int i = threadIdx.x;
    const float *dV = V + (blockIdx.x) * ldv;

    __shared__ float sum[ BLOCK_SIZE ];
    float lsum;

    /*  lsum := v**H * C  */
    lsum = MAGMA_S_ZERO;
    for (int j = i; j < m; j += BLOCK_SIZE)
       lsum += MAGMA_S_MUL( MAGMA_S_CONJ( dV[j] ), c[j] );
    
    sum[i] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i == 0)
       dwork [blockIdx.x] = sum[0];
}

/******************************************************************************/
/*
    Call 
        magma_sgemv_kernel3<<< n, BLOCK_SIZE, 0, queue->hip_stream() >>>(m, V, ldv, c, dwork, tau)
    to compute
        SGEMV( "Conjugate transpose", m, n, -tau[0], V, ldv, c, 1, zero, dwork, 1)
        and to set c[0] to 1.
    i.e., 
        work = -tau[0] V**H c
*/
extern "C"
__global__ void
magma_sgemv_kernel3(int m, const float * __restrict__ V, int ldv, float *c,
                    float *dwork, float *tau)
{
    const int i = threadIdx.x;
    const float *dV = V + (blockIdx.x) * ldv;

    __shared__ float sum[ BLOCK_SIZE ];
    float lsum;

    if (i == 0)
       c[0] = MAGMA_S_ONE;           

    /*  lsum := v**H * C  */
    lsum = MAGMA_S_ZERO;
    for (int j = i; j < m; j += BLOCK_SIZE)
       lsum += MAGMA_S_MUL( MAGMA_S_CONJ( dV[j] ), c[j] );

    sum[i] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i == 0)
       dwork [blockIdx.x] = -tau[0]*sum[0];
}


/******************************************************************************/
extern "C"
__global__ void
magma_sgemv_kernel2(int m, int n, const float * __restrict__ V, int ldv, 
                    const float * __restrict__ x, float *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    float lsum;

    V += j;

    lsum = MAGMA_S_ZERO;
    if (j < m) {
        for (int k=0; k < n; k++)
            lsum += MAGMA_S_MUL( V[k*ldv], x[k]);
        
        c[j] -= lsum;
    }
}


/******************************************************************************/
/*
    Apply a real block reflector H to a real vector C from the left
    (i.e., C = H C). H is represented in the form
          H = I - V T V**H
    where T is the real k-by-k upper triangular matrix in the 
    representation of the block reflector, and V is a real block of
    k elementary reflectors. 
*/
extern "C" void
magma_slarfbx_gpu(
    magma_int_t m, magma_int_t k,
    magmaFloat_ptr V,  magma_int_t ldv,
    magmaFloat_ptr dT, magma_int_t ldt,
    magmaFloat_ptr c,
    magmaFloat_ptr dwork,
    magma_queue_t queue )
{
    /* dwork = V**H c     */
    magma_sgemv_kernel1
        <<< k, BLOCK_SIZE, 0, queue->hip_stream() >>>
        (m, V, ldv, c, dwork); 

    /* dwork = T**H dwork */
    magma_strmv_tkernel
        <<< k, k, 0, queue->hip_stream() >>>
        ( dT, ldt, dwork, dwork+k);
 
    /* c = c - V dwork    */
    dim3  blocks3( magma_ceildiv( m, BLOCK_SIZE ) );
    dim3 threads3( BLOCK_SIZE );     
    magma_sgemv_kernel2
        <<< blocks3, threads3, 0, queue->hip_stream() >>>
        ( m, k, V, ldv, dwork+k, c);
}
