#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zlascl_diag.cu, normal z -> s, Sun Nov 20 20:20:29 2016
*/
#include "magma_internal.h"

#define MB 64
#define NB 160


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
slascl_diag_lower(
    int m, int n,
    const float* D, int ldd,
    float*       A, int lda)
{
    int ind_x = blockIdx.x * MB + threadIdx.x;
    int ind_y = blockIdx.y * NB;

    A += ind_x;
    if (ind_x < m) {
        for (int j=ind_y; j < min(ind_y+NB, n); j++ ) {
            A[j*lda] = MAGMA_S_DIV( A[j*lda], D[j + j*ldd] );
        }
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
slascl_diag_upper(
    int m, int n,
    const float* D, int ldd,
    float*       A, int lda)
{
    int ind_x = blockIdx.x * MB + threadIdx.x;
    int ind_y = blockIdx.y * NB;

    A += ind_x;
    if (ind_x < m) {
        for (int j=ind_y; j < min(ind_y+NB, n); j++ ) {
            A[j*lda] = MAGMA_S_DIV( A[j*lda], D[ind_x + ind_x*ldd] );
        }
    }
}


/***************************************************************************//**
    Purpose
    -------
    SLASCL_DIAG scales the M by N real matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be upper triangular or lower triangular.

    Arguments
    ---------
    @param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in]
    dD      REAL vector, dimension (LDDD,M)
            The matrix storing the scaling factor on its diagonal.

    @param[in]
    lddd    INTEGER
            The leading dimension of the array D.

    @param[in,out]
    dA      REAL array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_lascl_diag
*******************************************************************************/
extern "C" void
magmablas_slascl_diag(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaFloat_const_ptr dD, magma_int_t lddd,
    magmaFloat_ptr       dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    else if ( lddd < max(1,m) )
        *info = -5;
    else if ( ldda < max(1,m) )
        *info = -7;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 threads( MB );
    dim3 grid( magma_ceildiv( m, MB ), magma_ceildiv( n, NB ) );
    
    if (type == MagmaLower) {
        slascl_diag_lower
            <<< grid, threads, 0, queue->hip_stream() >>>
            (m, n, dD, lddd, dA, ldda);
    }
    else if (type == MagmaUpper) {
        slascl_diag_upper
            <<< grid, threads, 0, queue->hip_stream() >>>
            (m, n, dD, lddd, dA, ldda);
    }
}
