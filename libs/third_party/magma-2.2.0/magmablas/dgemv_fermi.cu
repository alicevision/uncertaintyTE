#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016
       
       @author Mark Gates
       @author Tingxing Dong
       @author Azzam Haidar

       @generated from magmablas/zgemv_fermi.cu, normal z -> d, Sun Nov 20 20:20:28 2016
*/
#include "magma_internal.h"
#include "commonblas_d.h"
#include "magma_templates.h"

#define PRECISION_d

#include "gemv_template_device.cuh"

#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"

#define version(s,v) s ## _V_ ## v


/******************************************************************************/
// NoTrans kernel
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
__global__ void
dgemvn_template_kernel_fermi(
    int m, int n, double alpha,
    const double * __restrict__ A, int lda,
    const double * __restrict__ x, int incx, double beta,
    double       * __restrict__ y, int incy)
{
#if (__CUDA_ARCH__ >= 200)
    gemvn_template_device<double, DIM_X, DIM_Y, TILE_SIZE>
        (m, n, alpha, A, lda, x, incx, beta, y, incy);
#endif /* (__CUDA_ARCH__ >= 200) */
}


/******************************************************************************/
// Trans/ConjTans kernel
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE, magma_trans_t trans>
__global__ void
dgemvc_template_kernel_fermi(
    int m, int n, double alpha,
    const double * __restrict__ A, int lda,
    const double * __restrict__ x, int incx, double beta,
    double       * __restrict__ y, int incy)
{
#if (__CUDA_ARCH__ >= 200)
    gemvc_template_device< double, DIM_X, DIM_Y, TILE_SIZE, trans >
        (m, n, alpha, A, lda, x, incx, beta, y, incy);
#endif /* (__CUDA_ARCH__ >= 200) */
}


/******************************************************************************/
// NoTrans CPU driver
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
void
dgemvn_template_fermi(
    magma_int_t m, magma_int_t n, double alpha,
    const double * __restrict__ A, magma_int_t lda,
    const double * __restrict__ x, magma_int_t incx, double beta,
    double       * __restrict__ y, magma_int_t incy,
    magma_queue_t queue)
{
    dim3 grid( magma_ceildiv(m, TILE_SIZE), 1 );
    dim3 threads( DIM_X, DIM_Y );

    dgemvn_template_kernel_fermi<DIM_X, DIM_Y, TILE_SIZE>
        <<< grid, threads, 0, queue->hip_stream() >>>
        (m, n, alpha, A, lda, x, incx, beta, y, incy);
}


/******************************************************************************/
// Trans/ConjTans CPU driver
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
void
dgemvc_template_fermi(
    magma_trans_t trans, magma_int_t m, magma_int_t n, double alpha,
    const double * __restrict__ A, magma_int_t lda,
    const double * __restrict__ x, magma_int_t incx, double beta,
    double       * __restrict__ y, magma_int_t incy,
    magma_queue_t queue)
{
    dim3 grid    ( magma_ceildiv(n, TILE_SIZE), 1 );
    dim3 threads ( DIM_X, DIM_Y );
    
    if (trans == MagmaConjTrans) {
        dgemvc_template_kernel_fermi< DIM_X, DIM_Y, TILE_SIZE, MagmaConjTrans >
            <<< grid, threads, 0, queue->hip_stream() >>>
            (m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
    else {
        dgemvc_template_kernel_fermi< DIM_X, DIM_Y, TILE_SIZE, MagmaTrans >
            <<< grid, threads, 0, queue->hip_stream() >>>
            (m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
}


/***************************************************************************//**
    Purpose
    -------
    DGEMV performs one of the matrix-vector operations
    
        y := alpha*A*x    + beta*y,   or
        y := alpha*A**T*x + beta*y,   or
        y := alpha*A**H*x + beta*y,
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    trans   magma_trans_t
            On entry, TRANS specifies the operation to be performed as
            follows:
      -     = MagmaNoTrans:    y := alpha*A  *x + beta*y
      -     = MagmaTrans:      y := alpha*A^T*x + beta*y
      -     = MagmaConjTrans:  y := alpha*A^H*x + beta*y

    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A
 
    @param[in]
    alpha   DOUBLE PRECISION
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      DOUBLE PRECISION array of dimension ( LDDA, n ) on the GPU.
   
    @param[in]
    ldda    INTEGER
            LDDA specifies the leading dimension of A.

    @param[in]
    dx      DOUBLE PRECISION array of dimension
            n if trans == MagmaNoTrans
            m if trans == MagmaTrans or MagmaConjTrans
     
    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.
  
    @param[in]
    beta    DOUBLE PRECISION
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      DOUBLE PRECISION array of dimension
            m if trans == MagmaNoTrans
            n if trans == MagmaTrans or MagmaConjTrans

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_gemv
*******************************************************************************/
extern "C" void
magmablas_dgemv(
    magma_trans_t trans, magma_int_t m, magma_int_t n, 
    double alpha,
    magmaDouble_const_ptr dA, magma_int_t ldda,
    magmaDouble_const_ptr dx, magma_int_t incx,
    double beta,
    magmaDouble_ptr dy, magma_int_t incy, 
    magma_queue_t queue)
{
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }

    // --------------------
    // CUDA ARCH 2.x (Fermi) version
    if ( trans == MagmaNoTrans ) {
        if (m <= 256) {
            dgemvn_template_fermi<version(N, 137)>
                ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
        }
        else {
            dgemvn_template_fermi<version(N, 140)>
                ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
        }
    }
    else {
        dgemvc_template_fermi<version(T, 189)>
            ( trans, m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
    }
}
