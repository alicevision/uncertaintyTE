#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016
       
       @generated from magmablas/zpotf2.cu, normal z -> d, Sun Nov 20 20:20:31 2016
*/
#include "magma_internal.h"

#define REAL

#define ddot_max_bs 512  // 512 is max threads for 1.x cards

void dpotf2_dscal( magma_int_t n, double *x, magma_int_t incx, magma_queue_t queue );
void dpotf2_ddot(  magma_int_t n, double *x, magma_int_t incx, magma_queue_t queue );

#ifdef COMPLEX
void magmablas_dlacgv( magma_int_t n, double *x, magma_int_t incx, magma_queue_t queue );
#endif


// TODO: this function could be in .cpp file -- it has no CUDA code in it.
/***************************************************************************//**
    Purpose
    -------

    dpotf2 computes the Cholesky factorization of a real symmetric
    positive definite matrix A.

    The factorization has the form
        A = U**H * U,  if UPLO = MagmaUpper, or
        A = L  * L**H, if UPLO = MagmaLower,
    where U is an upper triangular matrix and L is lower triangular.

    This is the unblocked version of the algorithm, calling Level 2 BLAS.

    Arguments
    ---------

    @param[in]
    uplo    magma_uplo_t
            Specifies whether the upper or lower triangular part of the
            symmetric matrix A is stored.
      -     = MagmaUpper:  Upper triangular
      -     = MagmaLower:  Lower triangular

    @param[in]
    n       INTEGER
            The order of the matrix A.  N >= 0 and N <= 512.

    @param[in,out]
    dA      DOUBLE PRECISION array, dimension (LDDA,N)
            On entry, the symmetric matrix A.  If UPLO = MagmaUpper, the leading
            n by n upper triangular part of A contains the upper
            triangular part of the matrix A, and the strictly lower
            triangular part of A is not referenced.  If UPLO = MagmaLower, the
            leading n by n lower triangular part of A contains the lower
            triangular part of the matrix A, and the strictly upper
            triangular part of A is not referenced.
    \n
            On exit, if INFO = 0, the factor U or L from the Cholesky
            factorization A = U**H * U  or A = L * L**H.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,N).

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @param[out]
    info    INTEGER
      -     = 0: successful exit
      -     < 0: if INFO = -k, the k-th argument had an illegal value
      -     > 0: if INFO = k, the leading minor of order k is not
                 positive definite, and the factorization could not be
                 completed.

    @ingroup magma_potf2
*******************************************************************************/
extern "C" magma_int_t
magma_dpotf2_gpu(
    magma_uplo_t uplo, magma_int_t n,
    magmaDouble_ptr dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
#define dA(i_, j_)  (dA + (i_) + (j_)*ldda)

    magma_int_t j;

    *info = 0;
    if ( uplo != MagmaUpper && uplo != MagmaLower) {
        *info = -1;
    } else if (n < 0 || n > ddot_max_bs) {
        *info = -2;
    } else if (ldda < max(1,n)) {
        *info = -4;
    }

    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    // Quick return if possible
    if (n == 0) {
        return *info;
    }

    double alpha = MAGMA_D_NEG_ONE;
    double beta  = MAGMA_D_ONE;

    if (uplo == MagmaUpper) {
        for (j = 0; j < n; j++) {
            dpotf2_ddot( j, dA(0,j), 1, queue ); // including ddot product and update a(j,j)
            if (j < n) {
                #ifdef COMPLEX
                magmablas_dlacgv( j, dA(0, j), 1, queue );
                #endif
                magma_dgemv( MagmaTrans, j, n-j-1,
                             alpha, dA(0, j+1), ldda,
                                    dA(0, j),   1,
                             beta,  dA(j, j+1), ldda, queue );

                #ifdef COMPLEX
                magmablas_dlacgv( j, dA(0, j), 1, queue );
                #endif
                dpotf2_dscal( n-j, dA(j,j), ldda, queue );
            }
        }
    }
    else {
        for (j = 0; j < n; j++) {
            dpotf2_ddot( j, dA(j,0), ldda, queue ); // including ddot product and update a(j,j)
            if (j < n) {
                #ifdef COMPLEX
                magmablas_dlacgv( j, dA(j, 0), ldda, queue );
                #endif
                magma_dgemv( MagmaNoTrans, n-j-1, j,
                             alpha, dA(j+1, 0), ldda,
                                    dA(j,0),    ldda,
                             beta,  dA(j+1, j), 1, queue );

                #ifdef COMPLEX
                magmablas_dlacgv( j, dA(j, 0), ldda, queue );
                #endif
                dpotf2_dscal( n-j, dA(j,j), 1, queue );
            }
        }
    }

    return *info;
}

#define dscal_bs  32
#define ddot_bs  512
#define dlacgv_bs 512

// dynamically allocated shared memory, set to size number of threads when the kernel is launched.
// See CUDA Guide B.2.3
extern __shared__ double shared_data[];

__global__ void kernel_ddot(int n, double *x, int incx, int threadSize)
{
    int tx = threadIdx.x;

    double *sdata = shared_data;

    double res = MAGMA_D_ZERO;

    if (tx < n) {
        res = x[tx*incx];
    }

    sdata[tx] = MAGMA_D_REAL(res * MAGMA_D_CONJ(res));

    __syncthreads();

    for (int s = blockDim.x/2; s > 32; s >>= 1 ) {
        if (tx < s) {
            sdata[tx] += sdata[tx+s];
        }
        __syncthreads();
    }

    if (tx < 32) {
        volatile double* smem = sdata;
        smem[tx] += smem[tx+32];
        smem[tx] += smem[tx+16];
        smem[tx] += smem[tx+8];
        smem[tx] += smem[tx+4];
        smem[tx] += smem[tx+2];
        smem[tx] += smem[tx+1];
    }

    if (tx == 0) {
        double xreal = MAGMA_D_REAL(x[n*incx]);
        x[n*incx] = MAGMA_D_MAKE( sqrt(xreal - sdata[0]), 0 );
    }
}

void dpotf2_ddot(
    magma_int_t n, double *x, magma_int_t incx,
    magma_queue_t queue )
{
    /*
    Specialized Ddot
    1) performs ddot sum = x[0:n-1]*conj(x[0:n-1])
    2) updates x[n] = sqrt(x[n]-sum);

    */
    if (n > ddot_max_bs) {
        fprintf( stderr, "n = %lld > %lld is not supported in dpotf2_ddot\n",
                 (long long) n, (long long) ddot_max_bs );
        return;
    }
    int threadSize;

    if (n <= 1024 && n > 512) {
        threadSize = 1024;
    }
    else if (n <= 512 && n > 256 ) {
        threadSize = 512;
    }
    else if (n <= 256 && n > 128) {
        threadSize = 256;
    }
    else if (n <= 128 && n > 64) {
        threadSize = 128;
    }
    else {
        threadSize = 64;
    }

    size_t shmem = threadSize * sizeof(double);
    kernel_ddot
        <<< 1, threadSize, shmem, queue->hip_stream() >>>
        (n, x, incx, threadSize);
}

__global__ void kernel_dscal(int n, double *x, int incx)
{
    int id = blockIdx.x * dscal_bs + threadIdx.x;

    __shared__ double factor;

    if (threadIdx.x == 0) {
        factor = MAGMA_D_MAKE(1.0/MAGMA_D_REAL(x[0]), 0.0);
    }

    __syncthreads();

    if ( id < n && id > 0) {
        x[id*incx] = x[id*incx] * factor;
    }
}


void dpotf2_dscal(
    magma_int_t n, double *x, magma_int_t incx,
    magma_queue_t queue )
{
    /* Specialized dscal perform x[1:n-1] / x[0] */
    dim3 threads(dscal_bs, 1, 1);
    int num_blocks = magma_ceildiv( n, dscal_bs );
    dim3 grid(num_blocks,1);
    kernel_dscal
        <<< grid, threads, 0, queue->hip_stream() >>>
        (n, x, incx);
}


#ifdef COMPLEX

__global__ void kernel_dlacgv(int n, double *x, int incx)
{
    int id = blockIdx.x * dlacgv_bs + threadIdx.x;

    if ( id < n ) {
        x[id*incx] = MAGMA_D_CONJ(x[id*incx]);
    }
}


/***************************************************************************//**
    Purpose
    -------

    DLACGV conjugates a real vector of length N.

    Arguments
    ---------

    @param[in]
    n       INTEGER
            The length of the vector X.  N >= 0.

    @param[in,out]
    x       DOUBLE PRECISION array, dimension (1+(N-1)*abs(INCX))
            On entry, the vector of length N to be conjugated.
            On exit, X is overwritten with conjg(X).

    @param[in]
    incx    INTEGER
            The spacing between successive elements of X.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_lacgv
*******************************************************************************/
void magmablas_dlacgv(
    magma_int_t n, double *x, magma_int_t incx,
    magma_queue_t queue )
{
    dim3 threads(dlacgv_bs, 1, 1);
    int num_blocks = magma_ceildiv( n, dlacgv_bs );
    dim3 grid(num_blocks,1);
    kernel_dlacgv
        <<< grid, threads, 0, queue->hip_stream() >>>
        (n, x, incx);
}

#endif // COMPLEX
