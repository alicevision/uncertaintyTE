#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zgetf2.cu, normal z -> s, Sun Nov 20 20:20:30 2016
*/
#include "magma_internal.h"

#define sger_bs 512  // 512 is max threads for 1.x cards

void magma_sgetf2_swap(
    magma_int_t n, float *x, magma_int_t i, magma_int_t j, magma_int_t incx,
    magma_queue_t queue );

void magma_sscal_sger(
    magma_int_t m, magma_int_t n, float *dA, magma_int_t ldda,
    magma_queue_t );


// TODO: this function could be in .cpp file -- it has no CUDA code in it.
/***************************************************************************//**
    SGETF2 computes an LU factorization of a general m-by-n matrix A
    using partial pivoting with row interchanges.

    The factorization has the form
        A = P * L * U
    where P is a permutation matrix, L is lower triangular with unit
    diagonal elements (lower trapezoidal if m > n), and U is upper
    triangular (upper trapezoidal if m < n).

    This is the right-looking Level 2 BLAS version of the algorithm.

    Arguments
    ---------

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0 and N <= 1024.
            On CUDA architecture 1.x cards, N <= 512.

    @param[in,out]
    dA      REAL array, dimension (LDDA,N)
            On entry, the m by n matrix to be factored.
            On exit, the factors L and U from the factorization
            A = P*L*U; the unit diagonal elements of L are not stored.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    ipiv    INTEGER array, dimension (min(M,N))
            The pivot indices; for 1 <= i <= min(M,N), row i of the
            matrix was interchanged with row IPIV(i).

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @param[out]
    info    INTEGER
      -     = 0: successful exit
      -     < 0: if INFO = -k, the k-th argument had an illegal value
      -     > 0: if INFO = k, U(k,k) is exactly zero. The factorization
                 has been completed, but the factor U is exactly
                 singular, and division by zero will occur if it is used
                 to solve a system of equations.

    @ingroup magma_getf2
*******************************************************************************/
extern "C" magma_int_t
magma_sgetf2_gpu(
    magma_int_t m, magma_int_t n,
    magmaFloat_ptr dA, magma_int_t ldda,
    magma_int_t *ipiv,
    magma_queue_t queue,
    magma_int_t *info )
{
    #define dA(i, j)  (dA + (i) + (j)*ldda)

    *info = 0;
    if (m < 0) {
        *info = -1;
    } else if (n < 0 || n > sger_bs) {
        *info = -2;
    } else if (ldda < max(1,m)) {
        *info = -4;
    }

    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    // Quick return if possible
    if (m == 0 || n == 0) {
        return *info;
    }

    magma_int_t min_mn = min(m, n);
    magma_int_t j, jp;
    
    for (j=0; j < min_mn; j++) {
        hipDeviceSetCacheConfig( hipFuncCachePreferShared );

        // Find pivot and test for singularity.
        jp = j - 1 + magma_isamax( m-j, dA(j,j), 1, queue );
        ipiv[j] = jp + 1;  // ipiv uses Fortran one-based index
        // Can't check value of dA since it is on GPU
        //if ( dA(jp, j) != 0.0) {
            hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
            
            // Apply the interchange to columns 1:N.
            if (jp != j) {
                magma_sgetf2_swap( n, dA, j, jp, ldda, queue );
            }
            
            // Compute elements J+1:M of J-th column.
            if (j < m) {
                magma_sscal_sger( m-j, n-j, dA(j, j), ldda, queue );
            }
        //}
        //else if (*info == 0) {
        //    *info = j;
        //}
    }

    return *info;
}


// ===========================================================================
// TODO: use standard BLAS magma_sswap?
#define sswap_bs 64

/******************************************************************************/
__global__
void kernel_sswap(int n, float *x, int i, int j, int incx)
{
    int id = blockIdx.x * sswap_bs + threadIdx.x;

    if (id < n) {
        float tmp = x[i + incx*id];
        x[i + incx*id] = x[j + incx*id];
        x[j + incx*id] = tmp;
    }
}


/******************************************************************************/
void magma_sgetf2_swap(
    magma_int_t n, float *x, magma_int_t i, magma_int_t j, magma_int_t incx,
    magma_queue_t queue )
{
    /* sswap two row vectors: ith and jth */
    dim3 threads( sswap_bs );
    dim3 grid( magma_ceildiv( n, sswap_bs ) );
    kernel_sswap
        <<< grid, threads, 0, queue->hip_stream() >>>
        (n, x, i, j, incx);
}


/******************************************************************************/
// dynamically allocated shared memory, set to size n when the kernel is launched.
// See CUDA Guide B.2.3
extern __shared__ float shared_data[];


/******************************************************************************/
__global__
void kernel_sscal_sger(int m, int n, float *A, int lda)
{
    float *shared_y = shared_data;

    int tid = blockIdx.x * sger_bs + threadIdx.x;

    float reg = MAGMA_S_ZERO;

    if (threadIdx.x < n) {
        shared_y[threadIdx.x] = A[lda * threadIdx.x];
    }

    __syncthreads();

    if (tid < m && tid > 0) {
        reg = A[tid];

        reg *= MAGMA_S_DIV(MAGMA_S_ONE, shared_y[0]);

        A[tid] = reg;

        #pragma unroll
        for (int i=1; i < n; i++) {
            A[tid + i*lda] += (MAGMA_S_NEG_ONE) * shared_y[i] * reg;
        }
    }
}


/******************************************************************************/
void magma_sscal_sger(
    magma_int_t m, magma_int_t n,
    magmaFloat_ptr dA, magma_int_t ldda,
    magma_queue_t queue )
{
    /*
    Specialized kernel that merges sscal and sger
    1) sscale the first column vector A(1:M-1,0) with 1/A(0,0);
    2) Performe a sger Operation for trailing matrix of A(1:M-1,1:N-1) += alpha*x*y**T, where 
       alpha := -1.0; x := A(1:M-1,0) and y:= A(0,1:N-1);
    */
    dim3 threads( sger_bs );
    dim3 grid( magma_ceildiv( m, sger_bs ) );
    size_t shared_size = sizeof(float)*(n);
    kernel_sscal_sger
        <<< grid, threads, shared_size, queue->hip_stream() >>>
        (m, n, dA, ldda);
}
