#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zgeadd2.cu, normal z -> s, Sun Nov 20 20:20:27 2016
       @author Mark Gates
*/
#include "magma_internal.h"

#define BLK_X 64
#define BLK_Y 32

/*
    Divides matrix into ceil( m/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.

    Code similar to slaset.
*/
__global__
void sgeadd2_full(
    int m, int n,
    float alpha,
    const float *dA, int ldda,
    float beta,
    float       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column */
    bool full = (iby + BLK_Y <= n);
    /* do only rows inside matrix */
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            // full block-column
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = alpha*dA[j*ldda] + beta*dB[j*lddb];
            }
        }
        else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = alpha*dA[j*ldda] + beta*dB[j*lddb];
            }
        }
    }
}


/***************************************************************************//**
    Purpose
    -------
    ZGEADD2 adds two matrices, dB = alpha*dA + beta*dB.
    
    @see ZGEADD for dB = alpha*dA + dB, lacking beta.
    
    Arguments
    ---------
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix dA.  N >= 0.
    
    @param[in]
    alpha   REAL
            The scalar alpha.
            
    @param[in]
    dA      REAL array, dimension (LDDA,N)
            The m by n matrix dA.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[in]
    beta    REAL
            The scalar beta.
            
    @param[in,out]
    dB      REAL array, dimension (LDDB,N)
            The m by n matrix dB.
    
    @param[in]
    lddb    INTEGER
            The leading dimension of the array dB.  LDDB >= max(1,M).
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_geadd
*******************************************************************************/
extern "C" void
magmablas_sgeadd2(
    magma_int_t m, magma_int_t n,
    float alpha,
    magmaFloat_const_ptr dA, magma_int_t ldda,
    float beta,
    magmaFloat_ptr       dB, magma_int_t lddb,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < max(1,m))
        info = -5;
    else if ( lddb < max(1,m))
        info = -7;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 || n == 0 )
        return;
    
    dim3 threads( BLK_X, 1 );
    dim3 grid( magma_ceildiv( m, BLK_X ), magma_ceildiv( n, BLK_Y ) );
    
    sgeadd2_full<<< grid, threads, 0, queue->hip_stream() >>>
        ( m, n, alpha, dA, ldda, beta, dB, lddb );
}
