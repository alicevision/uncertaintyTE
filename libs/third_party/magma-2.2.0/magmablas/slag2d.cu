#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/clag2z.cu, mixed zc -> ds, Sun Nov 20 20:20:27 2016
       @author Mark Gates
*/
#include "magma_internal.h"

#define BLK_X 64
#define BLK_Y 32


/*
    Divides matrix into ceil( m/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.
    
    Code similar to slat2d and zlaset.
*/
__global__
void slag2d_kernel(
    int m, int n,
    const float *SA, int ldsa,
    double       *A, int lda )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column */
    bool full = (iby + BLK_Y <= n);
    /* do only rows inside matrix */
    if ( ind < m ) {
        A  += ind + iby*lda;
        SA += ind + iby*ldsa;
        if ( full ) {
            // full block-column
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                A[j*lda] = MAGMA_D_MAKE( MAGMA_S_REAL( SA[j*ldsa] ), MAGMA_S_IMAG( SA[j*ldsa] ));
            }
        }
        else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                A[j*lda] = MAGMA_D_MAKE( MAGMA_S_REAL( SA[j*ldsa] ), MAGMA_S_IMAG( SA[j*ldsa] ));
            }
        }
    }
}


/***************************************************************************//**
    Purpose
    -------
    SLAG2D converts a single-real matrix, SA,
                 to a double-real matrix, A.

    Note that while it is possible to overflow while converting
    from double to single, it is not possible to overflow when
    converting from single to double.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of lines of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in]
    SA      SINGLE PRECISION array, dimension (LDSA,N)
            On entry, the M-by-N coefficient matrix SA.

    @param[in]
    ldsa    INTEGER
            The leading dimension of the array SA.  LDSA >= max(1,M).

    @param[out]
    A       DOUBLE PRECISION array, dimension (LDA,N)
            On exit, the M-by-N coefficient matrix A.

    @param[in]
    lda     INTEGER
            The leading dimension of the array A.  LDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
    
    @ingroup magma_lag2
*******************************************************************************/
extern "C" void
magmablas_slag2d(
    magma_int_t m, magma_int_t n,
    magmaFloat_const_ptr SA, magma_int_t ldsa,
    magmaDouble_ptr       A, magma_int_t lda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( m < 0 )
        *info = -1;
    else if ( n < 0 )
        *info = -2;
    else if ( ldsa < max(1,m) )
        *info = -4;
    else if ( lda < max(1,m) )
        *info = -6;

    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return; //*info;
    }

    /* quick return */
    if ( m == 0 || n == 0 ) {
        return;
    }

    dim3 threads( BLK_X, 1 );
    dim3 grid( magma_ceildiv( m, BLK_X ), magma_ceildiv( n, BLK_Y ) );
    slag2d_kernel<<< grid, threads, 0, queue->hip_stream() >>> ( m, n, SA, ldsa, A, lda );
}
