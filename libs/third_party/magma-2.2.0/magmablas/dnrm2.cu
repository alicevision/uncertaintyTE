#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/dznrm2.cu, normal z -> d, Sun Nov 20 20:20:29 2016

*/
#include "magma_internal.h"
#include "commonblas_d.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  512
#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16

#define REAL


/******************************************************************************/
__global__ void
magmablas_dnrm2_kernel(
    int m,
    double *dA, int ldda,
    double *dxnorm )
{
    const int tx = threadIdx.x;
    double *dx = dA + blockIdx.x * ldda;

    __shared__ double sum[ BLOCK_SIZE ];

    // get norm of dx
    double lsum = 0;
    for( int j = tx; j < m; j += BLOCK_SIZE ) {
        #ifdef REAL
            double re = dx[j];
            lsum += re*re;
        #else
            double re = MAGMA_D_REAL( dx[j] );
            double im = MAGMA_D_IMAG( dx[j] );
            lsum += re*re + im*im;
        #endif
    }
    sum[tx] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( tx, sum );
    
    if (tx == 0)
        dxnorm[blockIdx.x] = sqrt(sum[0]);
}


/******************************************************************************/
__global__ void
magmablas_dnrm2_check_kernel(
    int m,
    double *dA, int ldda,
    double *dxnorm, 
    double *lsticc )
{
    const int tx = threadIdx.x;
    double *dx = dA + blockIdx.x * ldda;

    __shared__ double sum[ BLOCK_SIZE ];

    // get norm of dx only if lsticc[blockIdx+1] != 0
    if ( lsticc[blockIdx.x + 1] == 0 )
        return;

    double lsum = 0;
    for( int j = tx; j < m; j += BLOCK_SIZE ) {
        #ifdef REAL
            double re = dx[j];
            lsum += re*re;
        #else
            double re = MAGMA_D_REAL( dx[j] );
            double im = MAGMA_D_IMAG( dx[j] );
            lsum += re*re + im*im;
        #endif
    }
    sum[tx] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( tx, sum );
    
    if (tx == 0)
        dxnorm[blockIdx.x] = sqrt(sum[0]);
}


/******************************************************************************/
extern "C" void
magmablas_dnrm2_check(
    magma_int_t m, magma_int_t n,
    magmaDouble_ptr dA, magma_int_t ldda, 
    magmaDouble_ptr dxnorm,
    magmaDouble_ptr dlsticc,
    magma_queue_t queue ) 
{
    dim3 threads( BLOCK_SIZE );
    dim3 blocks( n );    
    magmablas_dnrm2_check_kernel
        <<< blocks, threads, 0, queue->hip_stream() >>>
        ( m, dA, ldda, dxnorm, dlsticc );
}


/******************************************************************************/
__global__ void
magmablas_dnrm2_smkernel(
    int m, int n,
    double *dA, int ldda,
    double *dxnorm )
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    __shared__ double sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];

    for( int k = ty; k < n; k += BLOCK_SIZEy ) {
        double *dx = dA + k * ldda;

        // get norm of dx
        double lsum = 0;
        for( int j = tx; j < m; j += BLOCK_SIZEx ) {
            #ifdef REAL
                double re = dx[j];
                lsum += re*re;
            #else
                double re = MAGMA_D_REAL( dx[j] );
                double im = MAGMA_D_IMAG( dx[j] );
                lsum += re*re + im*im;
            #endif
        }
        sum[tx][ty] = lsum;
        magma_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( tx, ty, sum );

        if (tx == 0)
            dxnorm[k] = sqrt(sum[0][ty]);
        __syncthreads();
    }
}


/******************************************************************************/
/*
    Compute the dnrm2 of each column of m-by-n matrix dA.
    The resulting norms are written in the dxnorm array.
    This routine uses only one SM (block).
*/
extern "C" void
magmablas_dnrm2_sm(
    magma_int_t m, magma_int_t n,
    magmaDouble_ptr dA, magma_int_t ldda,
    magmaDouble_ptr dxnorm,
    magma_queue_t queue )
{
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );
    dim3 blocks( 1, 1 );
    magmablas_dnrm2_smkernel
        <<< blocks, threads, 0, queue->hip_stream() >>>
        ( m, n, dA, ldda, dxnorm );
}


/******************************************************************************/
__global__ void
magma_dnrm2_adjust_kernel(double *xnorm, double *c)
{
    const int tx = threadIdx.x;

    __shared__ double sum[ BLOCK_SIZE ];
    double temp;

    temp = MAGMA_D_ABS( c[tx] ) / xnorm[0];
    sum[tx] = -temp * temp;
    magma_sum_reduce_n( blockDim.x, tx, sum );

    __syncthreads();
    if (tx == 0)
        xnorm[0] = xnorm[0] * sqrt(1+sum[0]);
}


/******************************************************************************/
/*
    Adjust the norm of c to give the norm of c[k+1:], assuming that
    c was changed with orthogonal transformations.
*/
extern "C" void
magmablas_dnrm2_adjust(
    magma_int_t k,
    magmaDouble_ptr dxnorm,
    magmaDouble_ptr dc,
    magma_queue_t queue )
{
    dim3 threads( k );
    dim3 blocks( 1 );
    magma_dnrm2_adjust_kernel
        <<< blocks, threads, 0, queue->hip_stream() >>>
        (dxnorm, dc);
}


/******************************************************************************/

#define BS 256

__global__ void
magma_dnrm2_row_check_adjust_kernel(
    int n, double tol, double *xnorm, double *xnorm2, 
    double *C, int ldc, double *lsticc)
{
    const int tx = threadIdx.x + blockIdx.x*BS;
    lsticc[tx+1] = 0;

    if (tx < n) {
        double temp = MAGMA_D_ABS( C[tx*ldc] ) / xnorm[tx];
        temp = max( 0.0, ((1.0 + temp) * (1.0 - temp)) );
        
        double temp2 = xnorm[tx] / xnorm2[tx];
        temp2 = temp * (temp2 * temp2);
        
        if (temp2 <= tol) {
            lsticc[tx+1] = 1;
        } else {
            xnorm[tx] *= sqrt(temp);
        }
    }
    if (tx == 0)
        lsticc[0] = 0;
    magma_sum_reduce_n( blockDim.x, tx, lsticc );
}


/******************************************************************************/
/*
    Adjust the norm of C[,1:k] to give the norm of C[k+1:,1:k], assuming that
    C was changed with orthogonal transformations.
    It also do checks for QP3
*/
extern "C" void
magmablas_dnrm2_row_check_adjust(
    magma_int_t k, double tol,
    magmaDouble_ptr dxnorm,
    magmaDouble_ptr dxnorm2, 
    magmaDouble_ptr dC, magma_int_t lddc,
    magmaDouble_ptr dlsticc,
    magma_queue_t queue )
{
    dim3 threads( BS );
    dim3 blocks( magma_ceildiv( k, BS ) );
    magma_dnrm2_row_check_adjust_kernel
        <<< blocks, threads, 0, queue->hip_stream() >>>
        (k, tol, dxnorm, dxnorm2, dC, lddc, dlsticc);
}


/******************************************************************************/
/*
    Compute the dnrm2 of each column of m-by-n matrix dA.
    The resulting norms are written in the dxnorm array. 
    The computation can be done using n blocks (default) or on one SM (commented).
*/
extern "C" void
magmablas_dnrm2_cols(
    magma_int_t m, magma_int_t n,
    magmaDouble_ptr dA, magma_int_t ldda, 
    magmaDouble_ptr dxnorm,
    magma_queue_t queue ) 
{
    dim3 threads( BLOCK_SIZE );
    dim3 blocks( n );    
    magmablas_dnrm2_kernel
        <<< blocks, threads, 0, queue->hip_stream() >>>
        ( m, dA, ldda, dxnorm );

    // The following would do the computation on one SM
    // magmablas_dnrm2_sm( m, n, dA, ldda, dxnorm, queue );
}
