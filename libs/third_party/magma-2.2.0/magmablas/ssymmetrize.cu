#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zsymmetrize.cu, normal z -> s, Sun Nov 20 20:20:30 2016
       @author Mark Gates
*/
#include "magma_internal.h"

#define NB 64

/*
    Matrix is m x m, and is divided into block rows, each NB x m.
    Each block has NB threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
ssymmetrize_lower( int m, float *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    float *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        float *dAend = dA + i*ldda;  // end at diagonal dA(i,i)
        while( dA < dAend ) {
            *dAT = MAGMA_S_CONJ(*dA);  // upper := lower
            dA  += ldda;
            dAT += 1;
        }
        *dA = MAGMA_S_MAKE( MAGMA_S_REAL(*dA), 0 );  // make diagonal real
    }
}


// only difference with _lower version is direction dA=dAT instead of dAT=dA.
__global__ void
ssymmetrize_upper( int m, float *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    float *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        float *dAend = dA + i*ldda;  // end at diagonal dA(i,i)
        while( dA < dAend ) {
            *dA = MAGMA_S_CONJ(*dAT);  // lower := upper
            dA  += ldda;
            dAT += 1;
        }
        *dA = MAGMA_S_MAKE( MAGMA_S_REAL(*dA), 0 );  // make diagonal real
    }
}


/***************************************************************************//**
    Purpose
    -------
    
    SSYMMETRIZE copies lower triangle to upper triangle, or vice-versa,
    to make dA a general representation of a symmetric matrix.
    In Complex, it sets the diagonal to be Real.
    
    Arguments
    ---------
    
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of the matrix dA that is valid on input.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in,out]
    dA      REAL array, dimension (LDDA,N)
            The m by m matrix dA.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_symmetrize
*******************************************************************************/
extern "C" void
magmablas_ssymmetrize(
    magma_uplo_t uplo, magma_int_t m,
    magmaFloat_ptr dA, magma_int_t ldda,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( uplo != MagmaLower && uplo != MagmaUpper )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( ldda < max(1,m) )
        info = -4;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 )
        return;
    
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    
    if ( uplo == MagmaUpper ) {
        ssymmetrize_upper<<< grid, threads, 0, queue->hip_stream() >>>( m, dA, ldda );
    }
    else {
        ssymmetrize_lower<<< grid, threads, 0, queue->hip_stream() >>>( m, dA, ldda );
    }
}
