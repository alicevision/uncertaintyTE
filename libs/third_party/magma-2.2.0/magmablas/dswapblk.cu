#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/zswapblk.cu, normal z -> d, Sun Nov 20 20:20:29 2016

*/
#include "magma_internal.h"

#define BLOCK_SIZE 64

typedef struct {
    double *A;
    double *B;
    int n, ldda, lddb, npivots;
    short ipiv[BLOCK_SIZE];
} magmagpu_dswapblk_params_t;


/******************************************************************************/
__global__ void magmagpu_dswapblkrm( magmagpu_dswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    if ( y < params.n )
    {
        double *A = params.A + y - params.ldda;
        double *B = params.B + y;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A += params.ldda;
            if ( params.ipiv[i] == -1 )
                continue;
            double  tmp1 = *A;
            double *tmp2 = B + params.ipiv[i]*params.lddb;
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
}


/******************************************************************************/
__global__ void magmagpu_dswapblkcm( magmagpu_dswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = y*params.ldda;
    unsigned int offset2 = y*params.lddb;
    if ( y < params.n )
    {
        double *A = params.A + offset1 - 1;
        double *B = params.B + offset2;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A++;
            if ( params.ipiv[i] == -1 )
                continue;
            double  tmp1 = *A;
            double *tmp2 = B + params.ipiv[i];
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
    __syncthreads();
}


/***************************************************************************//**
    Blocked version: swap several pairs of lines.
    Used in magma_dtstrf() and magma_dssssm().
    @ingroup magma_swapblk
*******************************************************************************/
extern "C" void 
magmablas_dswapblk(
    magma_order_t order, magma_int_t n, 
    magmaDouble_ptr dA, magma_int_t ldda,
    magmaDouble_ptr dB, magma_int_t lddb,
    magma_int_t i1, magma_int_t i2,
    const magma_int_t *ipiv, magma_int_t inci, magma_int_t offset,
    magma_queue_t queue )
{
    magma_int_t  blocksize = 64;
    dim3 blocks( magma_ceildiv( n, blocksize ) );
    magma_int_t  k, im;
    
    /* Quick return */
    if ( n == 0 )
        return;
    
    if ( order == MagmaColMajor ) {
        for( k=(i1-1); k < i2; k += BLOCK_SIZE )
        {
            magma_int_t sb = min(BLOCK_SIZE, i2-k);
            magmagpu_dswapblk_params_t params = { dA+k, dB, int(n), int(ldda), int(lddb), int(sb) };
            for( magma_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_dswapblkcm<<< blocks, blocksize, 0, queue->hip_stream() >>>( params );
        }
    }
    else {
        for( k=(i1-1); k < i2; k += BLOCK_SIZE )
        {
            magma_int_t sb = min(BLOCK_SIZE, i2-k);
            magmagpu_dswapblk_params_t params = { dA+k*ldda, dB, int(n), int(ldda), int(lddb), int(sb) };
            for( magma_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_dswapblkrm<<< blocks, blocksize, 0, queue->hip_stream() >>>( params );
        }
    }
}
