#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/clat2z.cu, mixed zc -> ds, Sun Nov 20 20:20:28 2016
       @author Mark Gates
*/
#include "magma_internal.h"

#define BLK_X 64
#define BLK_Y 32


/*
    Divides matrix into ceil( n/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.
    Updates only the diagonal and below.
    Blocks that are fully above the diagonal exit immediately.
    
    Code similar to dlag2s and zlaset.
*/
__global__
void slat2d_lower(
    int n,
    const float *SA, int ldsa,
    double      *A,  int lda )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column && (below diag) */
    bool full = (iby + BLK_Y <= n && (ind >= iby + BLK_Y));
    /* do only rows inside matrix, and blocks not above diag */
    if ( ind < n && ind + BLK_X > iby ) {
        A  += ind + iby*lda;
        SA += ind + iby*ldsa;
        if ( full ) {
            // full block-column, off-diagonal block
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                A[j*lda] = MAGMA_D_MAKE( MAGMA_S_REAL( SA[j*ldsa] ),
                                         MAGMA_S_IMAG( SA[j*ldsa] ) );
            }
        }
        else {
            // either partial block-column or diagonal block
            for( int j=0; j < BLK_Y && iby+j < n && ind >= iby+j; ++j ) {
                A[j*lda] = MAGMA_D_MAKE( MAGMA_S_REAL( SA[j*ldsa] ),
                                         MAGMA_S_IMAG( SA[j*ldsa] ) );
            }
        }
    }
}


/*
    Similar to slat2d_full, but updates only the diagonal and above.
    Blocks that are fully below the diagonal exit immediately.
    
    Code similar to dlag2s and zlaset.
*/
__global__
void slat2d_upper(
    int n,
    const float *SA, int ldsa,
    double      *A,  int lda )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column && (above diag) */
    bool full = (iby + BLK_Y <= n && (ind + BLK_X <= iby));
    /* do only rows inside matrix, and blocks not below diag */
    if ( ind < n && ind < iby + BLK_Y ) {
        A  += ind + iby*lda;
        SA += ind + iby*ldsa;
        if ( full ) {
            // full block-column, off-diagonal block
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                A[j*lda] = MAGMA_D_MAKE( MAGMA_S_REAL( SA[j*ldsa] ),
                                         MAGMA_S_IMAG( SA[j*ldsa] ) );
            }
        }
        else {
            // either partial block-column or diagonal block
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                if ( ind <= iby+j ) {
                    A[j*lda] = MAGMA_D_MAKE( MAGMA_S_REAL( SA[j*ldsa] ),
                                             MAGMA_S_IMAG( SA[j*ldsa] ) );
                }
            }
        }
    }
}


/***************************************************************************//**
    Purpose
    -------
    SLAT2D converts a single-real matrix, SA,
                 to a double-real matrix, A.

    Note that while it is possible to overflow while converting
    from double to single, it is not possible to overflow when
    converting from single to double.

    Arguments
    ---------
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of the matrix A to be converted.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  n >= 0.
    
    @param[in]
    A       DOUBLE PRECISION array, dimension (LDA,n)
            On entry, the n-by-n coefficient matrix A.
    
    @param[in]
    lda     INTEGER
            The leading dimension of the array A.  LDA >= max(1,n).
    
    @param[out]
    SA      SINGLE PRECISION array, dimension (LDSA,n)
            On exit, if INFO=0, the n-by-n coefficient matrix SA;
            if INFO > 0, the content of SA is unspecified.
    
    @param[in]
    ldsa    INTEGER
            The leading dimension of the array SA.  LDSA >= max(1,n).
    
    @param[out]
    info    INTEGER
      -     = 0:  successful exit.
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
    
    @ingroup magma_lat2
*******************************************************************************/
extern "C" void
magmablas_slat2d(
    magma_uplo_t uplo, magma_int_t n,
    magmaFloat_const_ptr SA, magma_int_t ldsa,
    magmaDouble_ptr      A,  magma_int_t lda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( uplo != MagmaLower && uplo != MagmaUpper )
        *info = -1;
    else if ( n < 0 )
        *info = -2;
    else if ( lda < max(1,n) )
        *info = -4;
    else if ( ldsa < max(1,n) )
        *info = -6;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return; //*info;
    }

    /* quick return */
    if ( n == 0 ) {
        return;
    }
    
    dim3 threads( BLK_X, 1 );
    dim3 grid( magma_ceildiv( n, BLK_X ), magma_ceildiv( n, BLK_Y ) );
    
    if (uplo == MagmaLower) {
        slat2d_lower<<< grid, threads, 0, queue->hip_stream() >>> (n, SA, ldsa, A, lda);
    }
    else if (uplo == MagmaUpper) {
        slat2d_upper<<< grid, threads, 0, queue->hip_stream() >>> (n, SA, ldsa, A, lda);
    }
}
