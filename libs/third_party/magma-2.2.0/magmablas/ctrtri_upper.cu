#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/ztrtri_upper.cu, normal z -> c, Sun Nov 20 20:20:30 2016

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah
       
       This file implements upper case, and is called by ctrtri_kernel.cu.
       It's convenient to have separate files for lower & upper, to diff the sources.
*/

#include "magma_internal.h"

#define TRTRI_NONBATCHED
#include "ctrtri.cuh"
#include "ctrtri_upper_device.cuh"


/******************************************************************************/
__global__ void
ctrtri_diag_upper_kernel(
    magma_diag_t diag, int n, const magmaFloatComplex *A, int lda, magmaFloatComplex *d_dinvA)
{
    ctrtri_diag_upper_device(diag, n, A, lda, d_dinvA);
}


/******************************************************************************/
__global__ void
triple_cgemm16_part1_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm16_part1_upper_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm16_part2_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm16_part2_upper_device( n,  Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm32_part1_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm32_part1_upper_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm32_part2_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm32_part2_upper_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm64_part1_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm64_part1_upper_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm64_part2_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm64_part2_upper_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part1_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part1_upper_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part2_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part2_upper_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part3_upper_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part3_upper_device( n, Ain, lda, d_dinvA, jb, npages);
}
