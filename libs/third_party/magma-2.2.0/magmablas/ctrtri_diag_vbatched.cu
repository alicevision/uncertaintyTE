#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/ztrtri_diag_vbatched.cu, normal z -> c, Sun Nov 20 20:20:32 2016

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       File named ctrtri_diag.cu to avoid name conflict with src/ctrtri.o
       in the library. The actual kernels are in ctrtri_lower.cu and ctrtri_upper.cu
*/

#include "magma_internal.h"

#define    TRTRI_BATCHED
#include "ctrtri.cuh"

/***************************************************************************//**
    Purpose
    -------
    ctrtri_diag inverts the NB x NB diagonal blocks of A.

    Arguments
    ----------
    @param[in]
    uplo    magma_uplo_t.
            On entry, uplo specifies whether the matrix A is an upper or
            lower triangular matrix as follows:
      -     = MagmaUpper:  A is an upper triangular matrix.
      -     = MagmaLower:  A is a  lower triangular matrix.

    @param[in]
    diag    magma_diag_t.
            On entry, diag specifies whether or not A is unit triangular
            as follows:
      -     = MagmaUnit:     A is assumed to be unit triangular.
      -     = MagmaNonUnit:  A is not assumed to be unit triangular.

    @param[in]
    n       INTEGER.
            On entry, n specifies the order of the matrix A. N >= 0.

    @param[in]
    dA_array      COMPLEX array of dimension ( ldda, n )
            The triangular matrix A.
    \n
            If UPLO = 'U', the leading N-by-N upper triangular part of A
            contains the upper triangular matrix, and the strictly lower
            triangular part of A is not referenced.
    \n
            If UPLO = 'L', the leading N-by-N lower triangular part of A
            contains the lower triangular matrix, and the strictly upper
            triangular part of A is not referenced.
    \n
            If DIAG = 'U', the diagonal elements of A are also not referenced
            and are assumed to be 1.

    @param[in]
    ldda    INTEGER.
            The leading dimension of the array A.  LDDA >= max(1,N).

    @param[out]
    dinvA_array COMPLEX array of dimension (NB, ceil(n/NB)*NB),
            where NB = 128.
            On exit, contains inverses of the NB-by-NB diagonal blocks of A.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_trtri_batched
*******************************************************************************/
extern "C" void
magmablas_ctrtri_diag_vbatched(
    magma_uplo_t uplo, magma_diag_t diag, magma_int_t nmax, magma_int_t *n,
    magmaFloatComplex const * const *dA_array, magma_int_t *ldda,
    magmaFloatComplex **dinvA_array, 
    magma_int_t resetozero, magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t info = 0;
    if (uplo != MagmaLower && uplo != MagmaUpper)
        info = -1;
    else if (diag != MagmaNonUnit && diag != MagmaUnit)
        info = -2;
    else if (nmax < 0)
        info = -3;
    //else if (ldda < n)
    //    info = -5;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info
    }
    
    // allocate temp buffers for dimensions
    magma_int_t *mm, *nn;
    magma_imalloc( &mm, batchCount );
    magma_imalloc( &nn, batchCount );
    
    int nblocks = magma_ceildiv( nmax, IB );

    if ( resetozero ) {
        // roundup dimensions in 'n' and write it to 'mm' : magma_roundup( n, NB ) 
        magma_ivec_roundup( batchCount, n, NB, mm, queue);
        // set vector 'nn' to NB
        magma_ivec_setc( batchCount, nn, NB, queue);
        magma_int_t max_m = magma_roundup( nmax, NB );
        magma_int_t max_n = NB;
        //magmablas_claset_batched (MagmaFull, magma_roundup( n, NB ), NB, MAGMA_C_ZERO, MAGMA_C_ZERO, dinvA_array, magma_roundup( n, NB ), batchCount, queue);
        magmablas_claset_vbatched(MagmaFull, max_m, max_n, mm, nn, MAGMA_C_ZERO, MAGMA_C_ZERO, dinvA_array, mm, batchCount, queue);
    }
    // if someone want to use cudamemset he need to set the whole vectors 
    // of initial size otherwise it is a bug and thus need to have dinvA_length 
    // in input parameter and has been tested and was slower.
    //was not the largest size computed by the high API getrf_batched then it is bug and need to use magmablas_claset_batched


    if ( uplo == MagmaLower ) {
        // invert diagonal IB x IB inner blocks
        dim3 diaggrid( nblocks, 1, batchCount );  // emulate 3D grid
        ctrtri_diag_lower_kernel_vbatched<<< diaggrid, IB, 0, queue->hip_stream() >>>( diag, n, dA_array, ldda, dinvA_array );

        // build up NB x NB blocks (assuming IB=16 here):
        // use   16 x 16  blocks to build  32 x 32  blocks,  1 x (1 x npages) grid,  4 x 4 threads;
        // then  32 x 32  blocks to build  64 x 64  blocks,  1 x (2 x npages) grid,  8 x 4 threads;
        // then  64 x 64  blocks to build 128 x 128 blocks,  1 x (4 x npages) grid, 16 x 4 threads;
        // then 128 x 128 blocks to build 256 x 256 blocks,  2 x (8 x npages) grid, 16 x 4 threads.
        for( int jb=IB; jb < NB; jb *= 2 ) {
            int kb = jb*2;
            int npages = magma_ceildiv( nmax, kb );
            dim3 threads( (jb <= 32 ? jb/4 : 16), 4 );
            dim3 grid( jb/(threads.x*threads.y), npages*(jb/16), batchCount );  // emulate 3D grid: NX * (NY*npages), for CUDA ARCH 1.x
            
            //printf( "n %d, jb %d, grid %d x %d (%d x %d)\n", n, jb, grid.x, grid.y, grid.y / npages, npages );
            switch (jb) {
                case 16:
                    triple_cgemm16_part1_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm16_part2_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    break;
                case 32:
                    triple_cgemm32_part1_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm32_part2_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    break;
                case 64:
                    triple_cgemm64_part1_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm64_part2_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    break;
                default:
                    triple_cgemm_above64_part1_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm_above64_part2_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm_above64_part3_lower_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    break;
            }
            if ( kb >= nmax ) break;
        }
    }
    else {
        dim3 diaggrid( nblocks, 1, batchCount );  // emulate 3D grid
        ctrtri_diag_upper_kernel_vbatched<<< diaggrid, IB, 0, queue->hip_stream() >>>( diag, n, dA_array, ldda, dinvA_array );

        // update the inverse up to the size of IB
        for( int jb=IB; jb < NB; jb*=2 ) {
            int kb = jb*2;
            int npages = magma_ceildiv( nmax, kb );
            dim3 threads( (jb <= 32 ? jb/4 : 16), 4 );
            dim3 grid( jb/(threads.x*threads.y), npages*(jb/16), batchCount );  // emulate 3D grid: NX * (NY*npages), for CUDA ARCH 1.x
            
            switch (jb) {
                case 16:
                    triple_cgemm16_part1_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm16_part2_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    break;
                case 32:
                    triple_cgemm32_part1_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm32_part2_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    break;
                case 64:
                    triple_cgemm64_part1_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm64_part2_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    break;
                default:
                    triple_cgemm_above64_part1_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm_above64_part2_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    triple_cgemm_above64_part3_upper_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>( n, dA_array, ldda, dinvA_array, jb, npages );
                    break;
            }
            if ( kb >= nmax ) break;
        }
    }
    
    // free allocated buffers
    magma_free(mm);
    magma_free(nn);
}
