#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @generated from magmablas/ztrtri_lower.cu, normal z -> s, Sun Nov 20 20:20:30 2016

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       This file implements lower case, and is called by strtri_kernel.cu.
       It's convenient to have separate files for lower & upper, to diff the sources.
*/

#include "magma_internal.h"

#define TRTRI_NONBATCHED
#include "strtri.cuh"
#include "strtri_lower_device.cuh"


/******************************************************************************/
__global__ void
strtri_diag_lower_kernel(
    magma_diag_t diag, int n, const float *A, int lda, float *d_dinvA)
{
    strtri_diag_lower_device(diag, n, A, lda, d_dinvA);
}


/******************************************************************************/
__global__ void
triple_sgemm16_part1_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm16_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm16_part2_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm16_part2_lower_device( n,  Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm32_part1_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm32_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm32_part2_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm32_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm64_part1_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm64_part2_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm_above64_part1_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm_above64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm_above64_part2_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm_above64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm_above64_part3_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm_above64_part3_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}
